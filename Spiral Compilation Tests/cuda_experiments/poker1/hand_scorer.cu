#pragma warning(disable: 4101 4065 4060)
// Add these as extra argument to the compiler to suppress the rest:
// --diag-suppress 186 --diag-suppress 177

#include <hip/hip_runtime.h>
#include <cstdint>
#include <array>
#include <random>
#include <iostream>
struct Card { uint8_t rank : 4; uint8_t suit : 2; };
#include <algorithm>
struct Tuple0;
struct Tuple1;
struct Tuple2;
struct Tuple3;
struct Tuple4;
uint32_t loop_ranks_3(uint64_t v0, int8_t v1, int8_t v2);
uint32_t try_suit_2(uint64_t v0, uint16_t v1, int8_t v2);
uint32_t loop_5(uint64_t v0, int8_t v1, int8_t v2, int8_t v3, uint32_t v4);
uint32_t loop_suits_7(uint64_t v0, int8_t v1, int8_t v2, int8_t v3, int8_t v4, uint32_t v5);
uint32_t loop_ranks_6(uint64_t v0, int8_t v1, int8_t v2, int8_t v3, uint32_t v4);
uint32_t loop_ranks_11(uint64_t v0, int8_t v1, int8_t v2, int8_t v3, uint32_t v4);
uint32_t try_suit_10(uint64_t v0, uint16_t v1, int8_t v2);
uint32_t loop_suits_17(uint64_t v0, int8_t v1, int8_t v2, int8_t v3, int8_t v4, int8_t v5, uint32_t v6);
uint32_t loop_ranks_16(uint64_t v0, int8_t v1, int8_t v2, int8_t v3, int8_t v4, uint32_t v5);
uint32_t loop_suits_20(uint64_t v0, int8_t v1, int8_t v2, int8_t v3, uint32_t v4);
uint32_t loop_ranks_19(uint64_t v0, int8_t v1, int8_t v2, uint32_t v3);
uint64_t loop_pair_18(uint64_t v0, uint64_t v1, int8_t v2);
uint64_t loop_pair_15(uint64_t v0, uint64_t v1, int8_t v2, uint32_t v3, int8_t v4);
uint64_t loop_pair__14(uint64_t v0, uint64_t v1, int8_t v2);
uint64_t loop_triple_13(uint64_t v0, uint64_t v1, int8_t v2);
uint64_t loop_12(uint64_t v0, uint64_t v1, int8_t v2);
uint64_t loop_pair_9(uint64_t v0, uint64_t v1, uint16_t v2, int8_t v3, uint32_t v4, int8_t v5);
uint64_t loop_triple_8(uint64_t v0, uint64_t v1, uint16_t v2, int8_t v3);
uint64_t loop_ranks_4(uint64_t v0, uint64_t v1, uint16_t v2, int8_t v3);
uint64_t score_1(uint64_t v0);
Tuple2 score__0(uint64_t v0);
struct Tuple5;
typedef bool (* Fun0)(Card, Card);
struct Tuple6;
struct US0;
struct US1;
struct US2;
struct US3;
struct Tuple7;
struct Tuple8;
struct US4;
struct Tuple9;
struct US5;
struct US6;
struct US7;
Tuple5 score_21(std::array<Card,7l> v0);
struct Tuple0 {
    uint64_t v0;
    int32_t v1;
};
struct Tuple1 {
    int8_t v0;
    int8_t v1;
};
struct Tuple2 {
    int8_t v0;
    int8_t v1;
    int8_t v2;
    int8_t v3;
    int8_t v4;
    int8_t v5;
};
struct Tuple3 {
    uint16_t v1;
    int8_t v0;
};
struct Tuple4 {
    uint64_t v1;
    int8_t v0;
};
struct Tuple5 {
    std::array<Card,5l> v0;
    int8_t v1;
};
struct Tuple6 {
    int32_t v0;
    int32_t v1;
    int32_t v2;
    uint8_t v3;
};
struct US0 {
    union U {
        struct {
            std::array<Card,2l> v0;
            std::array<Card,5l> v1;
        } case1; // Some
        U() {}
    } v;
    char tag : 2;
    US0() {}
    US0(const US0 & x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
    }
    US0(const US0 && x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
    }
    US0 & operator=(US0 & x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
        return *this;
    }
    US0 & operator=(US0 && x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
        return *this;
    }
};
struct US1 {
    union U {
        struct {
            std::array<Card,5l> v0;
        } case1; // Some
        U() {}
    } v;
    char tag : 2;
    US1() {}
    US1(const US1 & x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
    }
    US1(const US1 && x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
    }
    US1 & operator=(US1 & x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
        return *this;
    }
    US1 & operator=(US1 && x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
        return *this;
    }
};
struct US2 {
    union U {
        struct {
            std::array<Card,2l> v0;
            std::array<Card,3l> v1;
        } case1; // Some
        U() {}
    } v;
    char tag : 2;
    US2() {}
    US2(const US2 & x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
    }
    US2(const US2 && x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
    }
    US2 & operator=(US2 & x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
        return *this;
    }
    US2 & operator=(US2 && x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
        return *this;
    }
};
struct US3 {
    union U {
        struct {
            std::array<Card,3l> v0;
            std::array<Card,4l> v1;
        } case1; // Some
        U() {}
    } v;
    char tag : 2;
    US3() {}
    US3(const US3 & x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
    }
    US3(const US3 && x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
    }
    US3 & operator=(US3 & x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
        return *this;
    }
    US3 & operator=(US3 && x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
        return *this;
    }
};
struct Tuple7 {
    int32_t v0;
    int32_t v1;
    uint8_t v2;
};
struct Tuple8 {
    int32_t v0;
    int32_t v1;
};
struct US4 {
    union U {
        U() {}
    } v;
    char tag : 2;
    US4() {}
    US4(const US4 & x) {
        this->tag = x.tag;
        switch (x.tag) {
        }
    }
    US4(const US4 && x) {
        this->tag = x.tag;
        switch (x.tag) {
        }
    }
    US4 & operator=(US4 & x) {
        this->tag = x.tag;
        switch (x.tag) {
        }
        return *this;
    }
    US4 & operator=(US4 && x) {
        this->tag = x.tag;
        switch (x.tag) {
        }
        return *this;
    }
};
struct Tuple9 {
    US4 v1;
    int32_t v0;
};
struct US5 {
    union U {
        struct {
            std::array<Card,2l> v0;
            std::array<Card,2l> v1;
        } case1; // Some
        U() {}
    } v;
    char tag : 2;
    US5() {}
    US5(const US5 & x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
    }
    US5(const US5 && x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
    }
    US5 & operator=(US5 & x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
        return *this;
    }
    US5 & operator=(US5 && x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
        return *this;
    }
};
struct US6 {
    union U {
        struct {
            std::array<Card,4l> v0;
            std::array<Card,3l> v1;
        } case1; // Some
        U() {}
    } v;
    char tag : 2;
    US6() {}
    US6(const US6 & x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
    }
    US6(const US6 && x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
    }
    US6 & operator=(US6 & x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
        return *this;
    }
    US6 & operator=(US6 && x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
        return *this;
    }
};
struct US7 {
    union U {
        struct {
            std::array<Card,5l> v0;
            int8_t v1;
        } case1; // Some
        U() {}
    } v;
    char tag : 2;
    US7() {}
    US7(const US7 & x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
    }
    US7(const US7 && x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
    }
    US7 & operator=(US7 & x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
        return *this;
    }
    US7 & operator=(US7 && x) {
        this->tag = x.tag;
        switch (x.tag) {
            case 1: { this->v.case1 = x.v.case1; break; }
        }
        return *this;
    }
};
inline Tuple0 TupleCreate0(uint64_t v0, int32_t v1){
    Tuple0 x;
    x.v0 = v0; x.v1 = v1;
    return x;
}
inline bool while_method_0(uint64_t v0){
    bool v1;
    v1 = v0 < 100000ull;
    return v1;
}
inline Tuple1 TupleCreate1(int8_t v0, int8_t v1){
    Tuple1 x;
    x.v0 = v0; x.v1 = v1;
    return x;
}
inline bool while_method_1(int32_t v0){
    bool v1;
    v1 = v0 > 0l;
    return v1;
}
inline Tuple2 TupleCreate2(int8_t v0, int8_t v1, int8_t v2, int8_t v3, int8_t v4, int8_t v5){
    Tuple2 x;
    x.v0 = v0; x.v1 = v1; x.v2 = v2; x.v3 = v3; x.v4 = v4; x.v5 = v5;
    return x;
}
inline Tuple3 TupleCreate3(int8_t v0, uint16_t v1){
    Tuple3 x;
    x.v0 = v0; x.v1 = v1;
    return x;
}
inline bool while_method_2(int8_t v0){
    bool v1;
    v1 = v0 < 13;
    return v1;
}
inline bool while_method_3(int8_t v0){
    bool v1;
    v1 = v0 < 4;
    return v1;
}
inline Tuple4 TupleCreate4(int8_t v0, uint64_t v1){
    Tuple4 x;
    x.v0 = v0; x.v1 = v1;
    return x;
}
uint32_t loop_ranks_3(uint64_t v0, int8_t v1, int8_t v2){
    bool v3;
    v3 = -1 <= v2;
    if (v3){
        int8_t v4;
        v4 = v2 + 4;
        int8_t v5;
        v5 = v1 * 13;
        int8_t v6;
        v6 = v5 + v4;
        int32_t v7;
        v7 = (int32_t)v6;
        uint64_t v8;
        v8 = v0 >> v7;
        uint8_t v9;
        v9 = (uint8_t)v8;
        uint8_t v10;
        v10 = v9 & 1u;
        int8_t v11;
        v11 = v2 + 3;
        int8_t v12;
        v12 = v5 + v11;
        int32_t v13;
        v13 = (int32_t)v12;
        uint64_t v14;
        v14 = v0 >> v13;
        uint8_t v15;
        v15 = (uint8_t)v14;
        uint8_t v16;
        v16 = v15 & 1u;
        uint8_t v17;
        v17 = v10 & v16;
        int8_t v18;
        v18 = v2 + 2;
        int8_t v19;
        v19 = v5 + v18;
        int32_t v20;
        v20 = (int32_t)v19;
        uint64_t v21;
        v21 = v0 >> v20;
        uint8_t v22;
        v22 = (uint8_t)v21;
        uint8_t v23;
        v23 = v22 & 1u;
        uint8_t v24;
        v24 = v17 & v23;
        int8_t v25;
        v25 = v2 + 1;
        int8_t v26;
        v26 = v5 + v25;
        int32_t v27;
        v27 = (int32_t)v26;
        uint64_t v28;
        v28 = v0 >> v27;
        uint8_t v29;
        v29 = (uint8_t)v28;
        uint8_t v30;
        v30 = v29 & 1u;
        uint8_t v31;
        v31 = v24 & v30;
        bool v32;
        v32 = v2 < 0;
        int8_t v34;
        if (v32){
            int8_t v33;
            v33 = v2 + 13;
            v34 = v33;
        } else {
            v34 = v2;
        }
        int8_t v35;
        v35 = v5 + v34;
        int32_t v36;
        v36 = (int32_t)v35;
        uint64_t v37;
        v37 = v0 >> v36;
        uint8_t v38;
        v38 = (uint8_t)v37;
        uint8_t v39;
        v39 = v38 & 1u;
        uint8_t v40;
        v40 = v31 & v39;
        bool v41;
        v41 = (bool)v40;
        if (v41){
            uint32_t v42;
            v42 = (uint32_t)v6;
            uint32_t v43;
            v43 = v42 << 6l;
            uint32_t v44;
            v44 = (uint32_t)v12;
            uint32_t v45;
            v45 = v43 + v44;
            uint32_t v46;
            v46 = v45 << 6l;
            uint32_t v47;
            v47 = (uint32_t)v19;
            uint32_t v48;
            v48 = v46 + v47;
            uint32_t v49;
            v49 = v48 << 6l;
            uint32_t v50;
            v50 = (uint32_t)v26;
            uint32_t v51;
            v51 = v49 + v50;
            int8_t v53;
            if (v32){
                int8_t v52;
                v52 = v2 + 13;
                v53 = v52;
            } else {
                v53 = v2;
            }
            uint32_t v54;
            v54 = v51 << 6l;
            int8_t v55;
            v55 = v5 + v53;
            uint32_t v56;
            v56 = (uint32_t)v55;
            uint32_t v57;
            v57 = v54 + v56;
            return v57;
        } else {
            int8_t v58;
            v58 = v2 - 1;
            return loop_ranks_3(v0, v1, v58);
        }
    } else {
        return 0ul;
    }
}
uint32_t try_suit_2(uint64_t v0, uint16_t v1, int8_t v2){
    int8_t v3;
    v3 = 4 * v2;
    int32_t v4;
    v4 = (int32_t)v3;
    uint16_t v5;
    v5 = v1 >> v4;
    int8_t v6;
    v6 = (int8_t)v5;
    int8_t v7;
    v7 = v6 & 15;
    bool v8;
    v8 = 5 <= v7;
    if (v8){
        int8_t v9;
        v9 = 8;
        return loop_ranks_3(v0, v2, v9);
    } else {
        return 0ul;
    }
}
uint32_t loop_5(uint64_t v0, int8_t v1, int8_t v2, int8_t v3, uint32_t v4){
    bool v5;
    v5 = v2 < 4;
    bool v7;
    if (v5){
        bool v6;
        v6 = 0 < v3;
        v7 = v6;
    } else {
        v7 = false;
    }
    if (v7){
        int8_t v8;
        v8 = v2 * 13;
        int8_t v9;
        v9 = v8 + v1;
        int32_t v10;
        v10 = (int32_t)v9;
        uint64_t v11;
        v11 = v0 >> v10;
        uint8_t v12;
        v12 = (uint8_t)v11;
        uint8_t v13;
        v13 = v12 & 1u;
        bool v14;
        v14 = (bool)v13;
        if (v14){
            int8_t v15;
            v15 = v2 + 1;
            int8_t v16;
            v16 = v3 - 1;
            uint32_t v17;
            v17 = v4 << 6l;
            uint32_t v18;
            v18 = (uint32_t)v9;
            uint32_t v19;
            v19 = v17 + v18;
            return loop_5(v0, v1, v15, v16, v19);
        } else {
            int8_t v21;
            v21 = v2 + 1;
            return loop_5(v0, v1, v21, v3, v4);
        }
    } else {
        return v4;
    }
}
uint32_t loop_suits_7(uint64_t v0, int8_t v1, int8_t v2, int8_t v3, int8_t v4, uint32_t v5){
    bool v6;
    v6 = 0 < v4;
    if (v6){
        bool v7;
        v7 = v3 < 4;
        if (v7){
            int8_t v8;
            v8 = v3 * 13;
            int8_t v9;
            v9 = v8 + v2;
            int32_t v10;
            v10 = (int32_t)v9;
            uint64_t v11;
            v11 = v0 >> v10;
            uint8_t v12;
            v12 = (uint8_t)v11;
            uint8_t v13;
            v13 = v12 & 1u;
            bool v14;
            v14 = (bool)v13;
            if (v14){
                int8_t v15;
                v15 = v3 + 1;
                int8_t v16;
                v16 = v4 - 1;
                uint32_t v17;
                v17 = v5 << 6l;
                uint32_t v18;
                v18 = (uint32_t)v9;
                uint32_t v19;
                v19 = v17 + v18;
                return loop_suits_7(v0, v1, v2, v15, v16, v19);
            } else {
                int8_t v21;
                v21 = v3 + 1;
                return loop_suits_7(v0, v1, v2, v21, v4, v5);
            }
        } else {
            int8_t v24;
            v24 = v2 - 1;
            return loop_ranks_6(v0, v1, v24, v4, v5);
        }
    } else {
        return v5;
    }
}
uint32_t loop_ranks_6(uint64_t v0, int8_t v1, int8_t v2, int8_t v3, uint32_t v4){
    bool v5;
    v5 = 0 <= v2;
    if (v5){
        bool v6;
        v6 = v1 == v2;
        if (v6){
            int8_t v7;
            v7 = v2 - 1;
            return loop_ranks_6(v0, v1, v7, v3, v4);
        } else {
            int8_t v9;
            v9 = 0;
            return loop_suits_7(v0, v1, v2, v9, v3, v4);
        }
    } else {
        return v4;
    }
}
uint32_t loop_ranks_11(uint64_t v0, int8_t v1, int8_t v2, int8_t v3, uint32_t v4){
    bool v5;
    v5 = 0 <= v2;
    bool v7;
    if (v5){
        bool v6;
        v6 = 0 < v3;
        v7 = v6;
    } else {
        v7 = false;
    }
    if (v7){
        int8_t v8;
        v8 = v1 * 13;
        int8_t v9;
        v9 = v8 + v2;
        int32_t v10;
        v10 = (int32_t)v9;
        uint64_t v11;
        v11 = v0 >> v10;
        uint8_t v12;
        v12 = (uint8_t)v11;
        uint8_t v13;
        v13 = v12 & 1u;
        bool v14;
        v14 = (bool)v13;
        if (v14){
            int8_t v15;
            v15 = v2 - 1;
            int8_t v16;
            v16 = v3 - 1;
            uint32_t v17;
            v17 = v4 << 6l;
            uint32_t v18;
            v18 = (uint32_t)v9;
            uint32_t v19;
            v19 = v17 + v18;
            return loop_ranks_11(v0, v1, v15, v16, v19);
        } else {
            int8_t v21;
            v21 = v2 - 1;
            return loop_ranks_11(v0, v1, v21, v3, v4);
        }
    } else {
        return v4;
    }
}
uint32_t try_suit_10(uint64_t v0, uint16_t v1, int8_t v2){
    int8_t v3;
    v3 = 4 * v2;
    int32_t v4;
    v4 = (int32_t)v3;
    uint16_t v5;
    v5 = v1 >> v4;
    int8_t v6;
    v6 = (int8_t)v5;
    int8_t v7;
    v7 = v6 & 15;
    bool v8;
    v8 = 5 <= v7;
    if (v8){
        int8_t v9;
        v9 = 12;
        int8_t v10;
        v10 = 5;
        uint32_t v11;
        v11 = 0ul;
        return loop_ranks_11(v0, v2, v9, v10, v11);
    } else {
        return 0ul;
    }
}
uint32_t loop_suits_17(uint64_t v0, int8_t v1, int8_t v2, int8_t v3, int8_t v4, int8_t v5, uint32_t v6){
    bool v7;
    v7 = 0 < v5;
    if (v7){
        bool v8;
        v8 = v4 < 4;
        if (v8){
            int8_t v9;
            v9 = v4 * 13;
            int8_t v10;
            v10 = v9 + v3;
            int32_t v11;
            v11 = (int32_t)v10;
            uint64_t v12;
            v12 = v0 >> v11;
            uint8_t v13;
            v13 = (uint8_t)v12;
            uint8_t v14;
            v14 = v13 & 1u;
            bool v15;
            v15 = (bool)v14;
            if (v15){
                int8_t v16;
                v16 = v4 + 1;
                int8_t v17;
                v17 = v5 - 1;
                uint32_t v18;
                v18 = v6 << 6l;
                uint32_t v19;
                v19 = (uint32_t)v10;
                uint32_t v20;
                v20 = v18 + v19;
                return loop_suits_17(v0, v1, v2, v3, v16, v17, v20);
            } else {
                int8_t v22;
                v22 = v4 + 1;
                return loop_suits_17(v0, v1, v2, v3, v22, v5, v6);
            }
        } else {
            int8_t v25;
            v25 = v3 - 1;
            return loop_ranks_16(v0, v1, v2, v25, v5, v6);
        }
    } else {
        return v6;
    }
}
uint32_t loop_ranks_16(uint64_t v0, int8_t v1, int8_t v2, int8_t v3, int8_t v4, uint32_t v5){
    bool v6;
    v6 = 0 <= v3;
    if (v6){
        bool v7;
        v7 = v1 == v3;
        bool v9;
        if (v7){
            v9 = true;
        } else {
            bool v8;
            v8 = v2 == v3;
            v9 = v8;
        }
        if (v9){
            int8_t v10;
            v10 = v3 - 1;
            return loop_ranks_16(v0, v1, v2, v10, v4, v5);
        } else {
            int8_t v12;
            v12 = 0;
            return loop_suits_17(v0, v1, v2, v3, v12, v4, v5);
        }
    } else {
        return v5;
    }
}
uint32_t loop_suits_20(uint64_t v0, int8_t v1, int8_t v2, int8_t v3, uint32_t v4){
    bool v5;
    v5 = 0 < v3;
    if (v5){
        bool v6;
        v6 = v2 < 4;
        if (v6){
            int8_t v7;
            v7 = v2 * 13;
            int8_t v8;
            v8 = v7 + v1;
            int32_t v9;
            v9 = (int32_t)v8;
            uint64_t v10;
            v10 = v0 >> v9;
            uint8_t v11;
            v11 = (uint8_t)v10;
            uint8_t v12;
            v12 = v11 & 1u;
            bool v13;
            v13 = (bool)v12;
            if (v13){
                int8_t v14;
                v14 = v2 + 1;
                int8_t v15;
                v15 = v3 - 1;
                uint32_t v16;
                v16 = v4 << 6l;
                uint32_t v17;
                v17 = (uint32_t)v8;
                uint32_t v18;
                v18 = v16 + v17;
                return loop_suits_20(v0, v1, v14, v15, v18);
            } else {
                int8_t v20;
                v20 = v2 + 1;
                return loop_suits_20(v0, v1, v20, v3, v4);
            }
        } else {
            int8_t v23;
            v23 = v1 - 1;
            return loop_ranks_19(v0, v23, v3, v4);
        }
    } else {
        return v4;
    }
}
uint32_t loop_ranks_19(uint64_t v0, int8_t v1, int8_t v2, uint32_t v3){
    bool v4;
    v4 = 0 <= v1;
    if (v4){
        int8_t v5;
        v5 = 0;
        return loop_suits_20(v0, v1, v5, v2, v3);
    } else {
        return v3;
    }
}
uint64_t loop_pair_18(uint64_t v0, uint64_t v1, int8_t v2){
    bool v3;
    v3 = 0 <= v2;
    if (v3){
        int8_t v4;
        v4 = 3 * v2;
        int32_t v5;
        v5 = (int32_t)v4;
        uint64_t v6;
        v6 = v0 >> v5;
        int8_t v7;
        v7 = (int8_t)v6;
        int8_t v8;
        v8 = v7 & 7;
        bool v9;
        v9 = v8 == 2;
        if (v9){
            int8_t v10;
            v10 = 0;
            int8_t v11;
            v11 = 2;
            uint32_t v12;
            v12 = 0ul;
            uint32_t v13;
            v13 = loop_5(v1, v2, v10, v11, v12);
            int8_t v14;
            v14 = 12;
            int8_t v15;
            v15 = 3;
            uint32_t v16;
            v16 = loop_ranks_6(v1, v2, v14, v15, v13);
            uint64_t v17;
            v17 = (uint64_t)v16;
            uint64_t v18;
            v18 = 4294967296ull | v17;
            return v18;
        } else {
            int8_t v19;
            v19 = v2 - 1;
            return loop_pair_18(v0, v1, v19);
        }
    } else {
        int8_t v22;
        v22 = 12;
        int8_t v23;
        v23 = 5;
        uint32_t v24;
        v24 = 0ul;
        uint32_t v25;
        v25 = loop_ranks_19(v1, v22, v23, v24);
        uint64_t v26;
        v26 = (uint64_t)v25;
        uint64_t v27;
        v27 = 0ull | v26;
        return v27;
    }
}
uint64_t loop_pair_15(uint64_t v0, uint64_t v1, int8_t v2, uint32_t v3, int8_t v4){
    bool v5;
    v5 = v2 == v4;
    if (v5){
        int8_t v6;
        v6 = v4 - 1;
        return loop_pair_15(v0, v1, v2, v3, v6);
    } else {
        bool v8;
        v8 = 0 <= v4;
        if (v8){
            int8_t v9;
            v9 = 3 * v4;
            int32_t v10;
            v10 = (int32_t)v9;
            uint64_t v11;
            v11 = v0 >> v10;
            int8_t v12;
            v12 = (int8_t)v11;
            int8_t v13;
            v13 = v12 & 7;
            bool v14;
            v14 = v13 == 2;
            if (v14){
                int8_t v15;
                v15 = 0;
                int8_t v16;
                v16 = 2;
                uint32_t v17;
                v17 = loop_5(v1, v4, v15, v16, v3);
                int8_t v18;
                v18 = 12;
                int8_t v19;
                v19 = 1;
                uint32_t v20;
                v20 = loop_ranks_16(v1, v2, v4, v18, v19, v17);
                uint64_t v21;
                v21 = (uint64_t)v20;
                uint64_t v22;
                v22 = 8589934592ull | v21;
                return v22;
            } else {
                int8_t v23;
                v23 = v4 - 1;
                return loop_pair_15(v0, v1, v2, v3, v23);
            }
        } else {
            int8_t v26;
            v26 = 12;
            return loop_pair_18(v0, v1, v26);
        }
    }
}
uint64_t loop_pair__14(uint64_t v0, uint64_t v1, int8_t v2){
    bool v3;
    v3 = 0 <= v2;
    if (v3){
        int8_t v4;
        v4 = 3 * v2;
        int32_t v5;
        v5 = (int32_t)v4;
        uint64_t v6;
        v6 = v0 >> v5;
        int8_t v7;
        v7 = (int8_t)v6;
        int8_t v8;
        v8 = v7 & 7;
        bool v9;
        v9 = v8 == 2;
        if (v9){
            int8_t v10;
            v10 = 0;
            int8_t v11;
            v11 = 2;
            uint32_t v12;
            v12 = 0ul;
            uint32_t v13;
            v13 = loop_5(v1, v2, v10, v11, v12);
            int8_t v14;
            v14 = 12;
            return loop_pair_15(v0, v1, v2, v13, v14);
        } else {
            int8_t v16;
            v16 = v2 - 1;
            return loop_pair__14(v0, v1, v16);
        }
    } else {
        int8_t v19;
        v19 = 12;
        return loop_pair_18(v0, v1, v19);
    }
}
uint64_t loop_triple_13(uint64_t v0, uint64_t v1, int8_t v2){
    bool v3;
    v3 = 0 <= v2;
    if (v3){
        int8_t v4;
        v4 = 3 * v2;
        int32_t v5;
        v5 = (int32_t)v4;
        uint64_t v6;
        v6 = v0 >> v5;
        int8_t v7;
        v7 = (int8_t)v6;
        int8_t v8;
        v8 = v7 & 7;
        bool v9;
        v9 = v8 == 3;
        if (v9){
            int8_t v10;
            v10 = 0;
            int8_t v11;
            v11 = 3;
            uint32_t v12;
            v12 = 0ul;
            uint32_t v13;
            v13 = loop_5(v1, v2, v10, v11, v12);
            int8_t v14;
            v14 = 12;
            int8_t v15;
            v15 = 2;
            uint32_t v16;
            v16 = loop_ranks_6(v1, v2, v14, v15, v13);
            uint64_t v17;
            v17 = (uint64_t)v16;
            uint64_t v18;
            v18 = 12884901888ull | v17;
            return v18;
        } else {
            int8_t v19;
            v19 = v2 - 1;
            return loop_triple_13(v0, v1, v19);
        }
    } else {
        int8_t v22;
        v22 = 12;
        return loop_pair__14(v0, v1, v22);
    }
}
uint64_t loop_12(uint64_t v0, uint64_t v1, int8_t v2){
    bool v3;
    v3 = -1 <= v2;
    if (v3){
        int8_t v4;
        v4 = v2 + 4;
        int8_t v5;
        v5 = 3 * v4;
        int32_t v6;
        v6 = (int32_t)v5;
        uint64_t v7;
        v7 = v0 >> v6;
        int8_t v8;
        v8 = (int8_t)v7;
        int8_t v9;
        v9 = v8 & 7;
        bool v10;
        v10 = 0 < v9;
        int8_t v11;
        if (v10){
            v11 = 1;
        } else {
            v11 = 0;
        }
        int8_t v12;
        v12 = v2 + 3;
        int8_t v13;
        v13 = 3 * v12;
        int32_t v14;
        v14 = (int32_t)v13;
        uint64_t v15;
        v15 = v0 >> v14;
        int8_t v16;
        v16 = (int8_t)v15;
        int8_t v17;
        v17 = v16 & 7;
        bool v18;
        v18 = 0 < v17;
        int8_t v19;
        if (v18){
            v19 = 1;
        } else {
            v19 = 0;
        }
        int8_t v20;
        v20 = v11 & v19;
        int8_t v21;
        v21 = v2 + 2;
        int8_t v22;
        v22 = 3 * v21;
        int32_t v23;
        v23 = (int32_t)v22;
        uint64_t v24;
        v24 = v0 >> v23;
        int8_t v25;
        v25 = (int8_t)v24;
        int8_t v26;
        v26 = v25 & 7;
        bool v27;
        v27 = 0 < v26;
        int8_t v28;
        if (v27){
            v28 = 1;
        } else {
            v28 = 0;
        }
        int8_t v29;
        v29 = v20 & v28;
        int8_t v30;
        v30 = v2 + 1;
        int8_t v31;
        v31 = 3 * v30;
        int32_t v32;
        v32 = (int32_t)v31;
        uint64_t v33;
        v33 = v0 >> v32;
        int8_t v34;
        v34 = (int8_t)v33;
        int8_t v35;
        v35 = v34 & 7;
        bool v36;
        v36 = 0 < v35;
        int8_t v37;
        if (v36){
            v37 = 1;
        } else {
            v37 = 0;
        }
        int8_t v38;
        v38 = v29 & v37;
        bool v39;
        v39 = v2 < 0;
        int8_t v41;
        if (v39){
            int8_t v40;
            v40 = v2 + 13;
            v41 = v40;
        } else {
            v41 = v2;
        }
        int8_t v42;
        v42 = 3 * v41;
        int32_t v43;
        v43 = (int32_t)v42;
        uint64_t v44;
        v44 = v0 >> v43;
        int8_t v45;
        v45 = (int8_t)v44;
        int8_t v46;
        v46 = v45 & 7;
        bool v47;
        v47 = 0 < v46;
        int8_t v48;
        if (v47){
            v48 = 1;
        } else {
            v48 = 0;
        }
        int8_t v49;
        v49 = v38 & v48;
        bool v50;
        v50 = (bool)v49;
        if (v50){
            int8_t v51;
            v51 = 0;
            int8_t v52;
            v52 = 1;
            uint32_t v53;
            v53 = 0ul;
            uint32_t v54;
            v54 = loop_5(v1, v4, v51, v52, v53);
            int8_t v55;
            v55 = 0;
            int8_t v56;
            v56 = 1;
            uint32_t v57;
            v57 = loop_5(v1, v12, v55, v56, v54);
            int8_t v58;
            v58 = 0;
            int8_t v59;
            v59 = 1;
            uint32_t v60;
            v60 = loop_5(v1, v21, v58, v59, v57);
            int8_t v61;
            v61 = 0;
            int8_t v62;
            v62 = 1;
            uint32_t v63;
            v63 = loop_5(v1, v30, v61, v62, v60);
            int8_t v65;
            if (v39){
                int8_t v64;
                v64 = v2 + 13;
                v65 = v64;
            } else {
                v65 = v2;
            }
            int8_t v66;
            v66 = 0;
            int8_t v67;
            v67 = 1;
            uint32_t v68;
            v68 = loop_5(v1, v65, v66, v67, v63);
            uint64_t v69;
            v69 = (uint64_t)v68;
            uint64_t v70;
            v70 = 17179869184ull | v69;
            return v70;
        } else {
            int8_t v71;
            v71 = v2 - 1;
            return loop_12(v0, v1, v71);
        }
    } else {
        int8_t v74;
        v74 = 12;
        return loop_triple_13(v0, v1, v74);
    }
}
uint64_t loop_pair_9(uint64_t v0, uint64_t v1, uint16_t v2, int8_t v3, uint32_t v4, int8_t v5){
    bool v6;
    v6 = 0 <= v5;
    if (v6){
        bool v7;
        v7 = v3 == v5;
        if (v7){
            int8_t v8;
            v8 = v5 - 1;
            return loop_pair_9(v0, v1, v2, v3, v4, v8);
        } else {
            int8_t v10;
            v10 = 3 * v5;
            int32_t v11;
            v11 = (int32_t)v10;
            uint64_t v12;
            v12 = v0 >> v11;
            int8_t v13;
            v13 = (int8_t)v12;
            int8_t v14;
            v14 = v13 & 7;
            bool v15;
            v15 = 2 <= v14;
            if (v15){
                int8_t v16;
                v16 = 0;
                int8_t v17;
                v17 = 2;
                uint32_t v18;
                v18 = loop_5(v1, v5, v16, v17, v4);
                uint64_t v19;
                v19 = (uint64_t)v18;
                uint64_t v20;
                v20 = 25769803776ull | v19;
                return v20;
            } else {
                int8_t v21;
                v21 = v5 - 1;
                return loop_pair_9(v0, v1, v2, v3, v4, v21);
            }
        }
    } else {
        int8_t v25;
        v25 = 0;
        uint32_t v26;
        v26 = try_suit_10(v1, v2, v25);
        bool v27;
        v27 = v26 >= 0ul;
        uint32_t v28;
        if (v27){
            v28 = v26;
        } else {
            v28 = 0ul;
        }
        int8_t v29;
        v29 = 1;
        uint32_t v30;
        v30 = try_suit_10(v1, v2, v29);
        bool v31;
        v31 = v30 >= v28;
        uint32_t v32;
        if (v31){
            v32 = v30;
        } else {
            v32 = v28;
        }
        int8_t v33;
        v33 = 2;
        uint32_t v34;
        v34 = try_suit_10(v1, v2, v33);
        bool v35;
        v35 = v34 >= v32;
        uint32_t v36;
        if (v35){
            v36 = v34;
        } else {
            v36 = v32;
        }
        int8_t v37;
        v37 = 3;
        uint32_t v38;
        v38 = try_suit_10(v1, v2, v37);
        bool v39;
        v39 = v38 >= v36;
        uint32_t v40;
        if (v39){
            v40 = v38;
        } else {
            v40 = v36;
        }
        bool v41;
        v41 = 0ul < v40;
        if (v41){
            uint64_t v42;
            v42 = (uint64_t)v40;
            uint64_t v43;
            v43 = 21474836480ull | v42;
            return v43;
        } else {
            int8_t v44;
            v44 = 8;
            return loop_12(v0, v1, v44);
        }
    }
}
uint64_t loop_triple_8(uint64_t v0, uint64_t v1, uint16_t v2, int8_t v3){
    bool v4;
    v4 = 0 <= v3;
    if (v4){
        int8_t v5;
        v5 = 3 * v3;
        int32_t v6;
        v6 = (int32_t)v5;
        uint64_t v7;
        v7 = v0 >> v6;
        int8_t v8;
        v8 = (int8_t)v7;
        int8_t v9;
        v9 = v8 & 7;
        bool v10;
        v10 = v9 == 3;
        if (v10){
            int8_t v11;
            v11 = 0;
            int8_t v12;
            v12 = 3;
            uint32_t v13;
            v13 = 0ul;
            uint32_t v14;
            v14 = loop_5(v1, v3, v11, v12, v13);
            int8_t v15;
            v15 = 12;
            return loop_pair_9(v0, v1, v2, v3, v14, v15);
        } else {
            int8_t v17;
            v17 = v3 - 1;
            return loop_triple_8(v0, v1, v2, v17);
        }
    } else {
        int8_t v20;
        v20 = 0;
        uint32_t v21;
        v21 = try_suit_10(v1, v2, v20);
        bool v22;
        v22 = v21 >= 0ul;
        uint32_t v23;
        if (v22){
            v23 = v21;
        } else {
            v23 = 0ul;
        }
        int8_t v24;
        v24 = 1;
        uint32_t v25;
        v25 = try_suit_10(v1, v2, v24);
        bool v26;
        v26 = v25 >= v23;
        uint32_t v27;
        if (v26){
            v27 = v25;
        } else {
            v27 = v23;
        }
        int8_t v28;
        v28 = 2;
        uint32_t v29;
        v29 = try_suit_10(v1, v2, v28);
        bool v30;
        v30 = v29 >= v27;
        uint32_t v31;
        if (v30){
            v31 = v29;
        } else {
            v31 = v27;
        }
        int8_t v32;
        v32 = 3;
        uint32_t v33;
        v33 = try_suit_10(v1, v2, v32);
        bool v34;
        v34 = v33 >= v31;
        uint32_t v35;
        if (v34){
            v35 = v33;
        } else {
            v35 = v31;
        }
        bool v36;
        v36 = 0ul < v35;
        if (v36){
            uint64_t v37;
            v37 = (uint64_t)v35;
            uint64_t v38;
            v38 = 21474836480ull | v37;
            return v38;
        } else {
            int8_t v39;
            v39 = 8;
            return loop_12(v0, v1, v39);
        }
    }
}
uint64_t loop_ranks_4(uint64_t v0, uint64_t v1, uint16_t v2, int8_t v3){
    bool v4;
    v4 = 0 <= v3;
    if (v4){
        int8_t v5;
        v5 = 3 * v3;
        int32_t v6;
        v6 = (int32_t)v5;
        uint64_t v7;
        v7 = v0 >> v6;
        int8_t v8;
        v8 = (int8_t)v7;
        int8_t v9;
        v9 = v8 & 7;
        bool v10;
        v10 = v9 == 4;
        if (v10){
            int8_t v11;
            v11 = 0;
            int8_t v12;
            v12 = 4;
            uint32_t v13;
            v13 = 0ul;
            uint32_t v14;
            v14 = loop_5(v1, v3, v11, v12, v13);
            int8_t v15;
            v15 = 12;
            int8_t v16;
            v16 = 1;
            uint32_t v17;
            v17 = loop_ranks_6(v1, v3, v15, v16, v14);
            uint64_t v18;
            v18 = (uint64_t)v17;
            uint64_t v19;
            v19 = 30064771072ull | v18;
            return v19;
        } else {
            int8_t v20;
            v20 = v3 - 1;
            return loop_ranks_4(v0, v1, v2, v20);
        }
    } else {
        int8_t v23;
        v23 = 12;
        return loop_triple_8(v0, v1, v2, v23);
    }
}
uint64_t score_1(uint64_t v0){
    int8_t v1; uint16_t v2;
    Tuple3 tmp8 = TupleCreate3(0, 0u);
    v1 = tmp8.v0; v2 = tmp8.v1;
    while (while_method_2(v1)){
        int8_t v4; uint16_t v5;
        Tuple3 tmp9 = TupleCreate3(0, v2);
        v4 = tmp9.v0; v5 = tmp9.v1;
        while (while_method_3(v4)){
            int8_t v7;
            v7 = v4 * 13;
            int8_t v8;
            v8 = v7 + v1;
            int32_t v9;
            v9 = (int32_t)v8;
            uint64_t v10;
            v10 = v0 >> v9;
            uint8_t v11;
            v11 = (uint8_t)v10;
            uint8_t v12;
            v12 = v11 & 1u;
            uint8_t v13;
            v13 = (uint8_t)v4;
            uint16_t v14;
            v14 = (uint16_t)v12;
            uint8_t v15;
            v15 = 4u * v13;
            int32_t v16;
            v16 = (int32_t)v15;
            uint16_t v17;
            v17 = v14 << v16;
            uint16_t v18;
            v18 = v5 + v17;
            v5 = v18;
            v4++;
        }
        v2 = v5;
        v1++;
    }
    int8_t v19; uint64_t v20;
    Tuple4 tmp10 = TupleCreate4(0, 0ull);
    v19 = tmp10.v0; v20 = tmp10.v1;
    while (while_method_2(v19)){
        int8_t v22; uint64_t v23;
        Tuple4 tmp11 = TupleCreate4(0, v20);
        v22 = tmp11.v0; v23 = tmp11.v1;
        while (while_method_3(v22)){
            int8_t v25;
            v25 = v22 * 13;
            int8_t v26;
            v26 = v25 + v19;
            int32_t v27;
            v27 = (int32_t)v26;
            uint64_t v28;
            v28 = v0 >> v27;
            uint8_t v29;
            v29 = (uint8_t)v28;
            uint8_t v30;
            v30 = v29 & 1u;
            uint8_t v31;
            v31 = (uint8_t)v19;
            uint64_t v32;
            v32 = (uint64_t)v30;
            uint8_t v33;
            v33 = 3u * v31;
            int32_t v34;
            v34 = (int32_t)v33;
            uint64_t v35;
            v35 = v32 << v34;
            uint64_t v36;
            v36 = v23 + v35;
            v23 = v36;
            v22++;
        }
        v20 = v23;
        v19++;
    }
    int8_t v37;
    v37 = 0;
    uint32_t v38;
    v38 = try_suit_2(v0, v2, v37);
    bool v39;
    v39 = v38 >= 0ul;
    uint32_t v40;
    if (v39){
        v40 = v38;
    } else {
        v40 = 0ul;
    }
    int8_t v41;
    v41 = 1;
    uint32_t v42;
    v42 = try_suit_2(v0, v2, v41);
    bool v43;
    v43 = v42 >= v40;
    uint32_t v44;
    if (v43){
        v44 = v42;
    } else {
        v44 = v40;
    }
    int8_t v45;
    v45 = 2;
    uint32_t v46;
    v46 = try_suit_2(v0, v2, v45);
    bool v47;
    v47 = v46 >= v44;
    uint32_t v48;
    if (v47){
        v48 = v46;
    } else {
        v48 = v44;
    }
    int8_t v49;
    v49 = 3;
    uint32_t v50;
    v50 = try_suit_2(v0, v2, v49);
    bool v51;
    v51 = v50 >= v48;
    uint32_t v52;
    if (v51){
        v52 = v50;
    } else {
        v52 = v48;
    }
    bool v53;
    v53 = 0ul < v52;
    if (v53){
        uint64_t v54;
        v54 = (uint64_t)v52;
        uint64_t v55;
        v55 = 34359738368ull | v54;
        return v55;
    } else {
        int8_t v56;
        v56 = 12;
        return loop_ranks_4(v20, v0, v2, v56);
    }
}
Tuple2 score__0(uint64_t v0){
    uint64_t v1;
    v1 = score_1(v0);
    uint64_t v2;
    v2 = v1 >> 32l;
    int8_t v3;
    v3 = (int8_t)v2;
    uint32_t v4;
    v4 = (uint32_t)v1;
    uint32_t v5;
    v5 = v4 >> 6l;
    uint32_t v6;
    v6 = v4 & 63ul;
    int8_t v7;
    v7 = (int8_t)v6;
    uint32_t v8;
    v8 = v5 >> 6l;
    uint32_t v9;
    v9 = v5 & 63ul;
    int8_t v10;
    v10 = (int8_t)v9;
    uint32_t v11;
    v11 = v8 >> 6l;
    uint32_t v12;
    v12 = v8 & 63ul;
    int8_t v13;
    v13 = (int8_t)v12;
    uint32_t v14;
    v14 = v11 >> 6l;
    uint32_t v15;
    v15 = v11 & 63ul;
    int8_t v16;
    v16 = (int8_t)v15;
    int8_t v17;
    v17 = (int8_t)v14;
    return TupleCreate2(v17, v16, v13, v10, v7, v3);
}
inline Tuple5 TupleCreate5(std::array<Card,5l> v0, int8_t v1){
    Tuple5 x;
    x.v0 = v0; x.v1 = v1;
    return x;
}
inline bool while_method_4(int32_t v0){
    bool v1;
    v1 = v0 < 7l;
    return v1;
}
bool ClosureMethod0(Card tup0, Card tup1){
    Card v0 = tup0; Card v1 = tup1;
    uint8_t v2;
    v2 = v0.rank;
    uint8_t v3;
    v3 = v1.rank;
    bool v4;
    v4 = v2 > v3;
    if (v4){
        return true;
    } else {
        uint8_t v5;
        v5 = v0.rank;
        uint8_t v6;
        v6 = v1.rank;
        bool v7;
        v7 = v5 == v6;
        if (v7){
            uint8_t v8;
            v8 = v0.suit;
            uint8_t v9;
            v9 = v1.suit;
            bool v10;
            v10 = v8 < v9;
            return v10;
        } else {
            return false;
        }
    }
}
inline bool while_method_5(int32_t v0){
    bool v1;
    v1 = v0 < 5l;
    return v1;
}
inline Tuple6 TupleCreate6(int32_t v0, int32_t v1, int32_t v2, uint8_t v3){
    Tuple6 x;
    x.v0 = v0; x.v1 = v1; x.v2 = v2; x.v3 = v3;
    return x;
}
US0 US0_0() { // None
    US0 x;
    x.tag = 0;
    return x;
}
US0 US0_1(std::array<Card,2l> v0, std::array<Card,5l> v1) { // Some
    US0 x;
    x.tag = 1;
    x.v.case1.v0 = v0; x.v.case1.v1 = v1;
    return x;
}
US1 US1_0() { // None
    US1 x;
    x.tag = 0;
    return x;
}
US1 US1_1(std::array<Card,5l> v0) { // Some
    US1 x;
    x.tag = 1;
    x.v.case1.v0 = v0;
    return x;
}
inline bool while_method_6(int32_t v0){
    bool v1;
    v1 = v0 < 3l;
    return v1;
}
inline bool while_method_7(int32_t v0){
    bool v1;
    v1 = v0 < 2l;
    return v1;
}
US2 US2_0() { // None
    US2 x;
    x.tag = 0;
    return x;
}
US2 US2_1(std::array<Card,2l> v0, std::array<Card,3l> v1) { // Some
    US2 x;
    x.tag = 1;
    x.v.case1.v0 = v0; x.v.case1.v1 = v1;
    return x;
}
inline bool while_method_8(int32_t v0){
    bool v1;
    v1 = v0 < 1l;
    return v1;
}
US3 US3_0() { // None
    US3 x;
    x.tag = 0;
    return x;
}
US3 US3_1(std::array<Card,3l> v0, std::array<Card,4l> v1) { // Some
    US3 x;
    x.tag = 1;
    x.v.case1.v0 = v0; x.v.case1.v1 = v1;
    return x;
}
inline bool while_method_9(int32_t v0){
    bool v1;
    v1 = v0 < 4l;
    return v1;
}
inline Tuple7 TupleCreate7(int32_t v0, int32_t v1, uint8_t v2){
    Tuple7 x;
    x.v0 = v0; x.v1 = v1; x.v2 = v2;
    return x;
}
inline Tuple8 TupleCreate8(int32_t v0, int32_t v1){
    Tuple8 x;
    x.v0 = v0; x.v1 = v1;
    return x;
}
US4 US4_0() { // Eq
    US4 x;
    x.tag = 0;
    return x;
}
US4 US4_1() { // Gt
    US4 x;
    x.tag = 1;
    return x;
}
US4 US4_2() { // Lt
    US4 x;
    x.tag = 2;
    return x;
}
inline Tuple9 TupleCreate9(int32_t v0, US4 v1){
    Tuple9 x;
    x.v0 = v0; x.v1 = v1;
    return x;
}
US5 US5_0() { // None
    US5 x;
    x.tag = 0;
    return x;
}
US5 US5_1(std::array<Card,2l> v0, std::array<Card,2l> v1) { // Some
    US5 x;
    x.tag = 1;
    x.v.case1.v0 = v0; x.v.case1.v1 = v1;
    return x;
}
US6 US6_0() { // None
    US6 x;
    x.tag = 0;
    return x;
}
US6 US6_1(std::array<Card,4l> v0, std::array<Card,3l> v1) { // Some
    US6 x;
    x.tag = 1;
    x.v.case1.v0 = v0; x.v.case1.v1 = v1;
    return x;
}
US7 US7_0() { // None
    US7 x;
    x.tag = 0;
    return x;
}
US7 US7_1(std::array<Card,5l> v0, int8_t v1) { // Some
    US7 x;
    x.tag = 1;
    x.v.case1.v0 = v0; x.v.case1.v1 = v1;
    return x;
}
Tuple5 score_21(std::array<Card,7l> v0){
    std::array<Card,7l> v1;
    int32_t v2;
    v2 = 0l;
    while (while_method_4(v2)){
        Card v4;
        v4 = v0[v2];
        v1[v2] = v4;
        v2++;
    }
    Fun0 v5;
    v5 = ClosureMethod0;
    std::sort(v1.begin(),v1.end(),v5);
    std::array<Card,5l> v6;
    int32_t v7;
    v7 = 0l;
    while (while_method_5(v7)){
        Card v9;
        v9 = v1[v7];
        v6[v7] = v9;
        v7++;
    }
    std::array<Card,2l> v10;
    std::array<Card,5l> v11;
    int32_t v12; int32_t v13; int32_t v14; uint8_t v15;
    Tuple6 tmp13 = TupleCreate6(0l, 0l, 0l, 12u);
    v12 = tmp13.v0; v13 = tmp13.v1; v14 = tmp13.v2; v15 = tmp13.v3;
    while (while_method_4(v12)){
        Card v17;
        v17 = v1[v12];
        bool v18;
        v18 = v14 < 2l;
        int32_t v27; int32_t v28; uint8_t v29;
        if (v18){
            uint8_t v19;
            v19 = v17.rank;
            bool v20;
            v20 = v15 == v19;
            int32_t v21;
            if (v20){
                v21 = v14;
            } else {
                v21 = 0l;
            }
            v10[v21] = v17;
            int32_t v22;
            v22 = v21 + 1l;
            uint8_t v23;
            v23 = v17.rank;
            v27 = v12; v28 = v22; v29 = v23;
        } else {
            break;
        }
        v13 = v27;
        v14 = v28;
        v15 = v29;
        v12++;
    }
    bool v30;
    v30 = v14 == 2l;
    US0 v40;
    if (v30){
        int32_t v31;
        v31 = 0l;
        while (while_method_5(v31)){
            int32_t v33;
            v33 = v13 + -1l;
            bool v34;
            v34 = v31 < v33;
            int32_t v35;
            if (v34){
                v35 = 0l;
            } else {
                v35 = 2l;
            }
            int32_t v36;
            v36 = v35 + v31;
            Card v37;
            v37 = v1[v36];
            v11[v31] = v37;
            v31++;
        }
        v40 = US0_1(v10, v11);
    } else {
        v40 = US0_0();
    }
    US1 v59;
    switch (v40.tag) {
        case 0: { // None
            v59 = US1_0();
            break;
        }
        default: { // Some
            std::array<Card,2l> v41 = v40.v.case1.v0; std::array<Card,5l> v42 = v40.v.case1.v1;
            std::array<Card,3l> v43;
            int32_t v44;
            v44 = 0l;
            while (while_method_6(v44)){
                Card v46;
                v46 = v42[v44];
                v43[v44] = v46;
                v44++;
            }
            std::array<Card,0l> v47;
            std::array<Card,5l> v48;
            int32_t v49;
            v49 = 0l;
            while (while_method_7(v49)){
                Card v51;
                v51 = v41[v49];
                v48[v49] = v51;
                v49++;
            }
            int32_t v52;
            v52 = 0l;
            while (while_method_6(v52)){
                Card v54;
                v54 = v43[v52];
                int32_t v55;
                v55 = 2l + v52;
                v48[v55] = v54;
                v52++;
            }
            v59 = US1_1(v48);
        }
    }
    std::array<Card,2l> v60;
    std::array<Card,5l> v61;
    int32_t v62; int32_t v63; int32_t v64; uint8_t v65;
    Tuple6 tmp14 = TupleCreate6(0l, 0l, 0l, 12u);
    v62 = tmp14.v0; v63 = tmp14.v1; v64 = tmp14.v2; v65 = tmp14.v3;
    while (while_method_4(v62)){
        Card v67;
        v67 = v1[v62];
        bool v68;
        v68 = v64 < 2l;
        int32_t v77; int32_t v78; uint8_t v79;
        if (v68){
            uint8_t v69;
            v69 = v67.rank;
            bool v70;
            v70 = v65 == v69;
            int32_t v71;
            if (v70){
                v71 = v64;
            } else {
                v71 = 0l;
            }
            v60[v71] = v67;
            int32_t v72;
            v72 = v71 + 1l;
            uint8_t v73;
            v73 = v67.rank;
            v77 = v62; v78 = v72; v79 = v73;
        } else {
            break;
        }
        v63 = v77;
        v64 = v78;
        v65 = v79;
        v62++;
    }
    bool v80;
    v80 = v64 == 2l;
    US0 v90;
    if (v80){
        int32_t v81;
        v81 = 0l;
        while (while_method_5(v81)){
            int32_t v83;
            v83 = v63 + -1l;
            bool v84;
            v84 = v81 < v83;
            int32_t v85;
            if (v84){
                v85 = 0l;
            } else {
                v85 = 2l;
            }
            int32_t v86;
            v86 = v85 + v81;
            Card v87;
            v87 = v1[v86];
            v61[v81] = v87;
            v81++;
        }
        v90 = US0_1(v60, v61);
    } else {
        v90 = US0_0();
    }
    US1 v148;
    switch (v90.tag) {
        case 0: { // None
            v148 = US1_0();
            break;
        }
        default: { // Some
            std::array<Card,2l> v91 = v90.v.case1.v0; std::array<Card,5l> v92 = v90.v.case1.v1;
            std::array<Card,2l> v93;
            std::array<Card,3l> v94;
            int32_t v95; int32_t v96; int32_t v97; uint8_t v98;
            Tuple6 tmp15 = TupleCreate6(0l, 0l, 0l, 12u);
            v95 = tmp15.v0; v96 = tmp15.v1; v97 = tmp15.v2; v98 = tmp15.v3;
            while (while_method_5(v95)){
                Card v100;
                v100 = v92[v95];
                bool v101;
                v101 = v97 < 2l;
                int32_t v110; int32_t v111; uint8_t v112;
                if (v101){
                    uint8_t v102;
                    v102 = v100.rank;
                    bool v103;
                    v103 = v98 == v102;
                    int32_t v104;
                    if (v103){
                        v104 = v97;
                    } else {
                        v104 = 0l;
                    }
                    v93[v104] = v100;
                    int32_t v105;
                    v105 = v104 + 1l;
                    uint8_t v106;
                    v106 = v100.rank;
                    v110 = v95; v111 = v105; v112 = v106;
                } else {
                    break;
                }
                v96 = v110;
                v97 = v111;
                v98 = v112;
                v95++;
            }
            bool v113;
            v113 = v97 == 2l;
            US2 v123;
            if (v113){
                int32_t v114;
                v114 = 0l;
                while (while_method_6(v114)){
                    int32_t v116;
                    v116 = v96 + -1l;
                    bool v117;
                    v117 = v114 < v116;
                    int32_t v118;
                    if (v117){
                        v118 = 0l;
                    } else {
                        v118 = 2l;
                    }
                    int32_t v119;
                    v119 = v118 + v114;
                    Card v120;
                    v120 = v92[v119];
                    v94[v114] = v120;
                    v114++;
                }
                v123 = US2_1(v93, v94);
            } else {
                v123 = US2_0();
            }
            switch (v123.tag) {
                case 0: { // None
                    v148 = US1_0();
                    break;
                }
                default: { // Some
                    std::array<Card,2l> v124 = v123.v.case1.v0; std::array<Card,3l> v125 = v123.v.case1.v1;
                    std::array<Card,1l> v126;
                    int32_t v127;
                    v127 = 0l;
                    while (while_method_8(v127)){
                        Card v129;
                        v129 = v125[v127];
                        v126[v127] = v129;
                        v127++;
                    }
                    std::array<Card,5l> v130;
                    int32_t v131;
                    v131 = 0l;
                    while (while_method_7(v131)){
                        Card v133;
                        v133 = v91[v131];
                        v130[v131] = v133;
                        v131++;
                    }
                    int32_t v134;
                    v134 = 0l;
                    while (while_method_7(v134)){
                        Card v136;
                        v136 = v124[v134];
                        int32_t v137;
                        v137 = 2l + v134;
                        v130[v137] = v136;
                        v134++;
                    }
                    int32_t v138;
                    v138 = 0l;
                    while (while_method_8(v138)){
                        Card v140;
                        v140 = v126[v138];
                        int32_t v141;
                        v141 = 4l + v138;
                        v130[v141] = v140;
                        v138++;
                    }
                    v148 = US1_1(v130);
                }
            }
        }
    }
    std::array<Card,3l> v149;
    std::array<Card,4l> v150;
    int32_t v151; int32_t v152; int32_t v153; uint8_t v154;
    Tuple6 tmp16 = TupleCreate6(0l, 0l, 0l, 12u);
    v151 = tmp16.v0; v152 = tmp16.v1; v153 = tmp16.v2; v154 = tmp16.v3;
    while (while_method_4(v151)){
        Card v156;
        v156 = v1[v151];
        bool v157;
        v157 = v153 < 3l;
        int32_t v166; int32_t v167; uint8_t v168;
        if (v157){
            uint8_t v158;
            v158 = v156.rank;
            bool v159;
            v159 = v154 == v158;
            int32_t v160;
            if (v159){
                v160 = v153;
            } else {
                v160 = 0l;
            }
            v149[v160] = v156;
            int32_t v161;
            v161 = v160 + 1l;
            uint8_t v162;
            v162 = v156.rank;
            v166 = v151; v167 = v161; v168 = v162;
        } else {
            break;
        }
        v152 = v166;
        v153 = v167;
        v154 = v168;
        v151++;
    }
    bool v169;
    v169 = v153 == 3l;
    US3 v179;
    if (v169){
        int32_t v170;
        v170 = 0l;
        while (while_method_9(v170)){
            int32_t v172;
            v172 = v152 + -2l;
            bool v173;
            v173 = v170 < v172;
            int32_t v174;
            if (v173){
                v174 = 0l;
            } else {
                v174 = 3l;
            }
            int32_t v175;
            v175 = v174 + v170;
            Card v176;
            v176 = v1[v175];
            v150[v170] = v176;
            v170++;
        }
        v179 = US3_1(v149, v150);
    } else {
        v179 = US3_0();
    }
    US1 v198;
    switch (v179.tag) {
        case 0: { // None
            v198 = US1_0();
            break;
        }
        default: { // Some
            std::array<Card,3l> v180 = v179.v.case1.v0; std::array<Card,4l> v181 = v179.v.case1.v1;
            std::array<Card,2l> v182;
            int32_t v183;
            v183 = 0l;
            while (while_method_7(v183)){
                Card v185;
                v185 = v181[v183];
                v182[v183] = v185;
                v183++;
            }
            std::array<Card,0l> v186;
            std::array<Card,5l> v187;
            int32_t v188;
            v188 = 0l;
            while (while_method_6(v188)){
                Card v190;
                v190 = v180[v188];
                v187[v188] = v190;
                v188++;
            }
            int32_t v191;
            v191 = 0l;
            while (while_method_7(v191)){
                Card v193;
                v193 = v182[v191];
                int32_t v194;
                v194 = 3l + v191;
                v187[v194] = v193;
                v191++;
            }
            v198 = US1_1(v187);
        }
    }
    std::array<Card,5l> v199;
    int32_t v200; int32_t v201; uint8_t v202;
    Tuple7 tmp17 = TupleCreate7(0l, 0l, 12u);
    v200 = tmp17.v0; v201 = tmp17.v1; v202 = tmp17.v2;
    while (while_method_4(v200)){
        Card v204;
        v204 = v1[v200];
        bool v205;
        v205 = v201 < 5l;
        int32_t v220; uint8_t v221;
        if (v205){
            uint8_t v206;
            v206 = v204.rank;
            uint8_t v207;
            v207 = v206 - 1u;
            bool v208;
            v208 = v202 == v207;
            bool v209;
            v209 = v208 != true;
            if (v209){
                uint8_t v210;
                v210 = v204.rank;
                bool v211;
                v211 = v202 == v210;
                int32_t v212;
                if (v211){
                    v212 = v201;
                } else {
                    v212 = 0l;
                }
                v199[v212] = v204;
                int32_t v213;
                v213 = v212 + 1l;
                uint8_t v214;
                v214 = v204.rank;
                uint8_t v215;
                v215 = v214 - 1u;
                v220 = v213; v221 = v215;
            } else {
                v220 = v201; v221 = v202;
            }
        } else {
            break;
        }
        v201 = v220;
        v202 = v221;
        v200++;
    }
    bool v222;
    v222 = v201 == 4l;
    bool v230;
    if (v222){
        uint8_t v223;
        v223 = v202 + 1u;
        bool v224;
        v224 = v223 == 0u;
        if (v224){
            Card v225;
            v225 = v1[0l];
            uint8_t v226;
            v226 = v225.rank;
            bool v227;
            v227 = v226 == 12u;
            if (v227){
                v199[4l] = v225;
                v230 = true;
            } else {
                v230 = false;
            }
        } else {
            v230 = false;
        }
    } else {
        v230 = false;
    }
    US1 v236;
    if (v230){
        v236 = US1_1(v199);
    } else {
        bool v232;
        v232 = v201 == 5l;
        if (v232){
            v236 = US1_1(v199);
        } else {
            v236 = US1_0();
        }
    }
    std::array<Card,5l> v237;
    int32_t v238; int32_t v239;
    Tuple8 tmp18 = TupleCreate8(0l, 0l);
    v238 = tmp18.v0; v239 = tmp18.v1;
    while (while_method_4(v238)){
        Card v241;
        v241 = v1[v238];
        uint8_t v242;
        v242 = v241.suit;
        bool v243;
        v243 = v242 == 3u;
        bool v245;
        if (v243){
            bool v244;
            v244 = v239 < 5l;
            v245 = v244;
        } else {
            v245 = false;
        }
        int32_t v247;
        if (v245){
            v237[v239] = v241;
            int32_t v246;
            v246 = v239 + 1l;
            v247 = v246;
        } else {
            v247 = v239;
        }
        v239 = v247;
        v238++;
    }
    bool v248;
    v248 = v239 == 5l;
    US1 v251;
    if (v248){
        v251 = US1_1(v237);
    } else {
        v251 = US1_0();
    }
    std::array<Card,5l> v252;
    int32_t v253; int32_t v254;
    Tuple8 tmp19 = TupleCreate8(0l, 0l);
    v253 = tmp19.v0; v254 = tmp19.v1;
    while (while_method_4(v253)){
        Card v256;
        v256 = v1[v253];
        uint8_t v257;
        v257 = v256.suit;
        bool v258;
        v258 = v257 == 2u;
        bool v260;
        if (v258){
            bool v259;
            v259 = v254 < 5l;
            v260 = v259;
        } else {
            v260 = false;
        }
        int32_t v262;
        if (v260){
            v252[v254] = v256;
            int32_t v261;
            v261 = v254 + 1l;
            v262 = v261;
        } else {
            v262 = v254;
        }
        v254 = v262;
        v253++;
    }
    bool v263;
    v263 = v254 == 5l;
    US1 v266;
    if (v263){
        v266 = US1_1(v252);
    } else {
        v266 = US1_0();
    }
    std::array<Card,5l> v267;
    int32_t v268; int32_t v269;
    Tuple8 tmp20 = TupleCreate8(0l, 0l);
    v268 = tmp20.v0; v269 = tmp20.v1;
    while (while_method_4(v268)){
        Card v271;
        v271 = v1[v268];
        uint8_t v272;
        v272 = v271.suit;
        bool v273;
        v273 = v272 == 1u;
        bool v275;
        if (v273){
            bool v274;
            v274 = v269 < 5l;
            v275 = v274;
        } else {
            v275 = false;
        }
        int32_t v277;
        if (v275){
            v267[v269] = v271;
            int32_t v276;
            v276 = v269 + 1l;
            v277 = v276;
        } else {
            v277 = v269;
        }
        v269 = v277;
        v268++;
    }
    bool v278;
    v278 = v269 == 5l;
    US1 v281;
    if (v278){
        v281 = US1_1(v267);
    } else {
        v281 = US1_0();
    }
    std::array<Card,5l> v282;
    int32_t v283; int32_t v284;
    Tuple8 tmp21 = TupleCreate8(0l, 0l);
    v283 = tmp21.v0; v284 = tmp21.v1;
    while (while_method_4(v283)){
        Card v286;
        v286 = v1[v283];
        uint8_t v287;
        v287 = v286.suit;
        bool v288;
        v288 = v287 == 0u;
        bool v290;
        if (v288){
            bool v289;
            v289 = v284 < 5l;
            v290 = v289;
        } else {
            v290 = false;
        }
        int32_t v292;
        if (v290){
            v282[v284] = v286;
            int32_t v291;
            v291 = v284 + 1l;
            v292 = v291;
        } else {
            v292 = v284;
        }
        v284 = v292;
        v283++;
    }
    bool v293;
    v293 = v284 == 5l;
    US1 v296;
    if (v293){
        v296 = US1_1(v282);
    } else {
        v296 = US1_0();
    }
    US1 v322;
    switch (v296.tag) {
        case 0: { // None
            v322 = v281;
            break;
        }
        default: { // Some
            std::array<Card,5l> v297 = v296.v.case1.v0;
            switch (v281.tag) {
                case 0: { // None
                    v322 = v296;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v298 = v281.v.case1.v0;
                    US4 v299;
                    v299 = US4_0();
                    int32_t v300; US4 v301;
                    Tuple9 tmp22 = TupleCreate9(0l, v299);
                    v300 = tmp22.v0; v301 = tmp22.v1;
                    while (while_method_5(v300)){
                        Card v303;
                        v303 = v297[v300];
                        Card v304;
                        v304 = v298[v300];
                        US4 v315;
                        switch (v301.tag) {
                            case 0: { // Eq
                                uint8_t v305;
                                v305 = v303.rank;
                                uint8_t v306;
                                v306 = v304.rank;
                                bool v307;
                                v307 = v305 < v306;
                                if (v307){
                                    v315 = US4_2();
                                } else {
                                    bool v309;
                                    v309 = v305 > v306;
                                    if (v309){
                                        v315 = US4_1();
                                    } else {
                                        v315 = US4_0();
                                    }
                                }
                                break;
                            }
                            default: {
                                break;
                            }
                        }
                        v301 = v315;
                        v300++;
                    }
                    bool v316;
                    switch (v301.tag) {
                        case 1: { // Gt
                            v316 = true;
                            break;
                        }
                        default: {
                            v316 = false;
                        }
                    }
                    std::array<Card,5l> v317;
                    if (v316){
                        v317 = v297;
                    } else {
                        v317 = v298;
                    }
                    v322 = US1_1(v317);
                }
            }
        }
    }
    US1 v348;
    switch (v322.tag) {
        case 0: { // None
            v348 = v266;
            break;
        }
        default: { // Some
            std::array<Card,5l> v323 = v322.v.case1.v0;
            switch (v266.tag) {
                case 0: { // None
                    v348 = v322;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v324 = v266.v.case1.v0;
                    US4 v325;
                    v325 = US4_0();
                    int32_t v326; US4 v327;
                    Tuple9 tmp23 = TupleCreate9(0l, v325);
                    v326 = tmp23.v0; v327 = tmp23.v1;
                    while (while_method_5(v326)){
                        Card v329;
                        v329 = v323[v326];
                        Card v330;
                        v330 = v324[v326];
                        US4 v341;
                        switch (v327.tag) {
                            case 0: { // Eq
                                uint8_t v331;
                                v331 = v329.rank;
                                uint8_t v332;
                                v332 = v330.rank;
                                bool v333;
                                v333 = v331 < v332;
                                if (v333){
                                    v341 = US4_2();
                                } else {
                                    bool v335;
                                    v335 = v331 > v332;
                                    if (v335){
                                        v341 = US4_1();
                                    } else {
                                        v341 = US4_0();
                                    }
                                }
                                break;
                            }
                            default: {
                                break;
                            }
                        }
                        v327 = v341;
                        v326++;
                    }
                    bool v342;
                    switch (v327.tag) {
                        case 1: { // Gt
                            v342 = true;
                            break;
                        }
                        default: {
                            v342 = false;
                        }
                    }
                    std::array<Card,5l> v343;
                    if (v342){
                        v343 = v323;
                    } else {
                        v343 = v324;
                    }
                    v348 = US1_1(v343);
                }
            }
        }
    }
    US1 v374;
    switch (v348.tag) {
        case 0: { // None
            v374 = v251;
            break;
        }
        default: { // Some
            std::array<Card,5l> v349 = v348.v.case1.v0;
            switch (v251.tag) {
                case 0: { // None
                    v374 = v348;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v350 = v251.v.case1.v0;
                    US4 v351;
                    v351 = US4_0();
                    int32_t v352; US4 v353;
                    Tuple9 tmp24 = TupleCreate9(0l, v351);
                    v352 = tmp24.v0; v353 = tmp24.v1;
                    while (while_method_5(v352)){
                        Card v355;
                        v355 = v349[v352];
                        Card v356;
                        v356 = v350[v352];
                        US4 v367;
                        switch (v353.tag) {
                            case 0: { // Eq
                                uint8_t v357;
                                v357 = v355.rank;
                                uint8_t v358;
                                v358 = v356.rank;
                                bool v359;
                                v359 = v357 < v358;
                                if (v359){
                                    v367 = US4_2();
                                } else {
                                    bool v361;
                                    v361 = v357 > v358;
                                    if (v361){
                                        v367 = US4_1();
                                    } else {
                                        v367 = US4_0();
                                    }
                                }
                                break;
                            }
                            default: {
                                break;
                            }
                        }
                        v353 = v367;
                        v352++;
                    }
                    bool v368;
                    switch (v353.tag) {
                        case 1: { // Gt
                            v368 = true;
                            break;
                        }
                        default: {
                            v368 = false;
                        }
                    }
                    std::array<Card,5l> v369;
                    if (v368){
                        v369 = v349;
                    } else {
                        v369 = v350;
                    }
                    v374 = US1_1(v369);
                }
            }
        }
    }
    std::array<Card,3l> v375;
    std::array<Card,4l> v376;
    int32_t v377; int32_t v378; int32_t v379; uint8_t v380;
    Tuple6 tmp25 = TupleCreate6(0l, 0l, 0l, 12u);
    v377 = tmp25.v0; v378 = tmp25.v1; v379 = tmp25.v2; v380 = tmp25.v3;
    while (while_method_4(v377)){
        Card v382;
        v382 = v1[v377];
        bool v383;
        v383 = v379 < 3l;
        int32_t v392; int32_t v393; uint8_t v394;
        if (v383){
            uint8_t v384;
            v384 = v382.rank;
            bool v385;
            v385 = v380 == v384;
            int32_t v386;
            if (v385){
                v386 = v379;
            } else {
                v386 = 0l;
            }
            v375[v386] = v382;
            int32_t v387;
            v387 = v386 + 1l;
            uint8_t v388;
            v388 = v382.rank;
            v392 = v377; v393 = v387; v394 = v388;
        } else {
            break;
        }
        v378 = v392;
        v379 = v393;
        v380 = v394;
        v377++;
    }
    bool v395;
    v395 = v379 == 3l;
    US3 v405;
    if (v395){
        int32_t v396;
        v396 = 0l;
        while (while_method_9(v396)){
            int32_t v398;
            v398 = v378 + -2l;
            bool v399;
            v399 = v396 < v398;
            int32_t v400;
            if (v399){
                v400 = 0l;
            } else {
                v400 = 3l;
            }
            int32_t v401;
            v401 = v400 + v396;
            Card v402;
            v402 = v1[v401];
            v376[v396] = v402;
            v396++;
        }
        v405 = US3_1(v375, v376);
    } else {
        v405 = US3_0();
    }
    US1 v456;
    switch (v405.tag) {
        case 0: { // None
            v456 = US1_0();
            break;
        }
        default: { // Some
            std::array<Card,3l> v406 = v405.v.case1.v0; std::array<Card,4l> v407 = v405.v.case1.v1;
            std::array<Card,2l> v408;
            std::array<Card,2l> v409;
            int32_t v410; int32_t v411; int32_t v412; uint8_t v413;
            Tuple6 tmp26 = TupleCreate6(0l, 0l, 0l, 12u);
            v410 = tmp26.v0; v411 = tmp26.v1; v412 = tmp26.v2; v413 = tmp26.v3;
            while (while_method_9(v410)){
                Card v415;
                v415 = v407[v410];
                bool v416;
                v416 = v412 < 2l;
                int32_t v425; int32_t v426; uint8_t v427;
                if (v416){
                    uint8_t v417;
                    v417 = v415.rank;
                    bool v418;
                    v418 = v413 == v417;
                    int32_t v419;
                    if (v418){
                        v419 = v412;
                    } else {
                        v419 = 0l;
                    }
                    v408[v419] = v415;
                    int32_t v420;
                    v420 = v419 + 1l;
                    uint8_t v421;
                    v421 = v415.rank;
                    v425 = v410; v426 = v420; v427 = v421;
                } else {
                    break;
                }
                v411 = v425;
                v412 = v426;
                v413 = v427;
                v410++;
            }
            bool v428;
            v428 = v412 == 2l;
            US5 v438;
            if (v428){
                int32_t v429;
                v429 = 0l;
                while (while_method_7(v429)){
                    int32_t v431;
                    v431 = v411 + -1l;
                    bool v432;
                    v432 = v429 < v431;
                    int32_t v433;
                    if (v432){
                        v433 = 0l;
                    } else {
                        v433 = 2l;
                    }
                    int32_t v434;
                    v434 = v433 + v429;
                    Card v435;
                    v435 = v407[v434];
                    v409[v429] = v435;
                    v429++;
                }
                v438 = US5_1(v408, v409);
            } else {
                v438 = US5_0();
            }
            switch (v438.tag) {
                case 0: { // None
                    v456 = US1_0();
                    break;
                }
                default: { // Some
                    std::array<Card,2l> v439 = v438.v.case1.v0; std::array<Card,2l> v440 = v438.v.case1.v1;
                    std::array<Card,0l> v441;
                    std::array<Card,5l> v442;
                    int32_t v443;
                    v443 = 0l;
                    while (while_method_6(v443)){
                        Card v445;
                        v445 = v406[v443];
                        v442[v443] = v445;
                        v443++;
                    }
                    int32_t v446;
                    v446 = 0l;
                    while (while_method_7(v446)){
                        Card v448;
                        v448 = v439[v446];
                        int32_t v449;
                        v449 = 3l + v446;
                        v442[v449] = v448;
                        v446++;
                    }
                    v456 = US1_1(v442);
                }
            }
        }
    }
    std::array<Card,4l> v457;
    std::array<Card,3l> v458;
    int32_t v459; int32_t v460; int32_t v461; uint8_t v462;
    Tuple6 tmp27 = TupleCreate6(0l, 0l, 0l, 12u);
    v459 = tmp27.v0; v460 = tmp27.v1; v461 = tmp27.v2; v462 = tmp27.v3;
    while (while_method_4(v459)){
        Card v464;
        v464 = v1[v459];
        bool v465;
        v465 = v461 < 4l;
        int32_t v474; int32_t v475; uint8_t v476;
        if (v465){
            uint8_t v466;
            v466 = v464.rank;
            bool v467;
            v467 = v462 == v466;
            int32_t v468;
            if (v467){
                v468 = v461;
            } else {
                v468 = 0l;
            }
            v457[v468] = v464;
            int32_t v469;
            v469 = v468 + 1l;
            uint8_t v470;
            v470 = v464.rank;
            v474 = v459; v475 = v469; v476 = v470;
        } else {
            break;
        }
        v460 = v474;
        v461 = v475;
        v462 = v476;
        v459++;
    }
    bool v477;
    v477 = v461 == 4l;
    US6 v487;
    if (v477){
        int32_t v478;
        v478 = 0l;
        while (while_method_6(v478)){
            int32_t v480;
            v480 = v460 + -3l;
            bool v481;
            v481 = v478 < v480;
            int32_t v482;
            if (v481){
                v482 = 0l;
            } else {
                v482 = 4l;
            }
            int32_t v483;
            v483 = v482 + v478;
            Card v484;
            v484 = v1[v483];
            v458[v478] = v484;
            v478++;
        }
        v487 = US6_1(v457, v458);
    } else {
        v487 = US6_0();
    }
    US1 v506;
    switch (v487.tag) {
        case 0: { // None
            v506 = US1_0();
            break;
        }
        default: { // Some
            std::array<Card,4l> v488 = v487.v.case1.v0; std::array<Card,3l> v489 = v487.v.case1.v1;
            std::array<Card,1l> v490;
            int32_t v491;
            v491 = 0l;
            while (while_method_8(v491)){
                Card v493;
                v493 = v489[v491];
                v490[v491] = v493;
                v491++;
            }
            std::array<Card,0l> v494;
            std::array<Card,5l> v495;
            int32_t v496;
            v496 = 0l;
            while (while_method_9(v496)){
                Card v498;
                v498 = v488[v496];
                v495[v496] = v498;
                v496++;
            }
            int32_t v499;
            v499 = 0l;
            while (while_method_8(v499)){
                Card v501;
                v501 = v490[v499];
                int32_t v502;
                v502 = 4l + v499;
                v495[v502] = v501;
                v499++;
            }
            v506 = US1_1(v495);
        }
    }
    std::array<Card,5l> v507;
    int32_t v508; int32_t v509; uint8_t v510;
    Tuple7 tmp28 = TupleCreate7(0l, 0l, 12u);
    v508 = tmp28.v0; v509 = tmp28.v1; v510 = tmp28.v2;
    while (while_method_4(v508)){
        Card v512;
        v512 = v1[v508];
        bool v513;
        v513 = v509 < 5l;
        int32_t v526; uint8_t v527;
        if (v513){
            uint8_t v514;
            v514 = v512.suit;
            bool v515;
            v515 = 3u == v514;
            if (v515){
                uint8_t v516;
                v516 = v512.rank;
                bool v517;
                v517 = v510 == v516;
                int32_t v518;
                if (v517){
                    v518 = v509;
                } else {
                    v518 = 0l;
                }
                v507[v518] = v512;
                int32_t v519;
                v519 = v518 + 1l;
                uint8_t v520;
                v520 = v512.rank;
                uint8_t v521;
                v521 = v520 - 1u;
                v526 = v519; v527 = v521;
            } else {
                v526 = v509; v527 = v510;
            }
        } else {
            break;
        }
        v509 = v526;
        v510 = v527;
        v508++;
    }
    bool v528;
    v528 = v509 == 4l;
    bool v563;
    if (v528){
        uint8_t v529;
        v529 = v510 + 1u;
        bool v530;
        v530 = v529 == 0u;
        if (v530){
            Card v531;
            v531 = v1[0l];
            uint8_t v532;
            v532 = v531.suit;
            bool v533;
            v533 = 3u == v532;
            bool v537;
            if (v533){
                uint8_t v534;
                v534 = v531.rank;
                bool v535;
                v535 = v534 == 12u;
                if (v535){
                    v507[4l] = v531;
                    v537 = true;
                } else {
                    v537 = false;
                }
            } else {
                v537 = false;
            }
            if (v537){
                v563 = true;
            } else {
                Card v538;
                v538 = v1[1l];
                uint8_t v539;
                v539 = v538.suit;
                bool v540;
                v540 = 3u == v539;
                bool v544;
                if (v540){
                    uint8_t v541;
                    v541 = v538.rank;
                    bool v542;
                    v542 = v541 == 12u;
                    if (v542){
                        v507[4l] = v538;
                        v544 = true;
                    } else {
                        v544 = false;
                    }
                } else {
                    v544 = false;
                }
                if (v544){
                    v563 = true;
                } else {
                    Card v545;
                    v545 = v1[2l];
                    uint8_t v546;
                    v546 = v545.suit;
                    bool v547;
                    v547 = 3u == v546;
                    bool v551;
                    if (v547){
                        uint8_t v548;
                        v548 = v545.rank;
                        bool v549;
                        v549 = v548 == 12u;
                        if (v549){
                            v507[4l] = v545;
                            v551 = true;
                        } else {
                            v551 = false;
                        }
                    } else {
                        v551 = false;
                    }
                    if (v551){
                        v563 = true;
                    } else {
                        Card v552;
                        v552 = v1[3l];
                        uint8_t v553;
                        v553 = v552.suit;
                        bool v554;
                        v554 = 3u == v553;
                        if (v554){
                            uint8_t v555;
                            v555 = v552.rank;
                            bool v556;
                            v556 = v555 == 12u;
                            if (v556){
                                v507[4l] = v552;
                                v563 = true;
                            } else {
                                v563 = false;
                            }
                        } else {
                            v563 = false;
                        }
                    }
                }
            }
        } else {
            v563 = false;
        }
    } else {
        v563 = false;
    }
    US1 v569;
    if (v563){
        v569 = US1_1(v507);
    } else {
        bool v565;
        v565 = v509 == 5l;
        if (v565){
            v569 = US1_1(v507);
        } else {
            v569 = US1_0();
        }
    }
    std::array<Card,5l> v570;
    int32_t v571; int32_t v572; uint8_t v573;
    Tuple7 tmp29 = TupleCreate7(0l, 0l, 12u);
    v571 = tmp29.v0; v572 = tmp29.v1; v573 = tmp29.v2;
    while (while_method_4(v571)){
        Card v575;
        v575 = v1[v571];
        bool v576;
        v576 = v572 < 5l;
        int32_t v589; uint8_t v590;
        if (v576){
            uint8_t v577;
            v577 = v575.suit;
            bool v578;
            v578 = 2u == v577;
            if (v578){
                uint8_t v579;
                v579 = v575.rank;
                bool v580;
                v580 = v573 == v579;
                int32_t v581;
                if (v580){
                    v581 = v572;
                } else {
                    v581 = 0l;
                }
                v570[v581] = v575;
                int32_t v582;
                v582 = v581 + 1l;
                uint8_t v583;
                v583 = v575.rank;
                uint8_t v584;
                v584 = v583 - 1u;
                v589 = v582; v590 = v584;
            } else {
                v589 = v572; v590 = v573;
            }
        } else {
            break;
        }
        v572 = v589;
        v573 = v590;
        v571++;
    }
    bool v591;
    v591 = v572 == 4l;
    bool v626;
    if (v591){
        uint8_t v592;
        v592 = v573 + 1u;
        bool v593;
        v593 = v592 == 0u;
        if (v593){
            Card v594;
            v594 = v1[0l];
            uint8_t v595;
            v595 = v594.suit;
            bool v596;
            v596 = 2u == v595;
            bool v600;
            if (v596){
                uint8_t v597;
                v597 = v594.rank;
                bool v598;
                v598 = v597 == 12u;
                if (v598){
                    v570[4l] = v594;
                    v600 = true;
                } else {
                    v600 = false;
                }
            } else {
                v600 = false;
            }
            if (v600){
                v626 = true;
            } else {
                Card v601;
                v601 = v1[1l];
                uint8_t v602;
                v602 = v601.suit;
                bool v603;
                v603 = 2u == v602;
                bool v607;
                if (v603){
                    uint8_t v604;
                    v604 = v601.rank;
                    bool v605;
                    v605 = v604 == 12u;
                    if (v605){
                        v570[4l] = v601;
                        v607 = true;
                    } else {
                        v607 = false;
                    }
                } else {
                    v607 = false;
                }
                if (v607){
                    v626 = true;
                } else {
                    Card v608;
                    v608 = v1[2l];
                    uint8_t v609;
                    v609 = v608.suit;
                    bool v610;
                    v610 = 2u == v609;
                    bool v614;
                    if (v610){
                        uint8_t v611;
                        v611 = v608.rank;
                        bool v612;
                        v612 = v611 == 12u;
                        if (v612){
                            v570[4l] = v608;
                            v614 = true;
                        } else {
                            v614 = false;
                        }
                    } else {
                        v614 = false;
                    }
                    if (v614){
                        v626 = true;
                    } else {
                        Card v615;
                        v615 = v1[3l];
                        uint8_t v616;
                        v616 = v615.suit;
                        bool v617;
                        v617 = 2u == v616;
                        if (v617){
                            uint8_t v618;
                            v618 = v615.rank;
                            bool v619;
                            v619 = v618 == 12u;
                            if (v619){
                                v570[4l] = v615;
                                v626 = true;
                            } else {
                                v626 = false;
                            }
                        } else {
                            v626 = false;
                        }
                    }
                }
            }
        } else {
            v626 = false;
        }
    } else {
        v626 = false;
    }
    US1 v632;
    if (v626){
        v632 = US1_1(v570);
    } else {
        bool v628;
        v628 = v572 == 5l;
        if (v628){
            v632 = US1_1(v570);
        } else {
            v632 = US1_0();
        }
    }
    std::array<Card,5l> v633;
    int32_t v634; int32_t v635; uint8_t v636;
    Tuple7 tmp30 = TupleCreate7(0l, 0l, 12u);
    v634 = tmp30.v0; v635 = tmp30.v1; v636 = tmp30.v2;
    while (while_method_4(v634)){
        Card v638;
        v638 = v1[v634];
        bool v639;
        v639 = v635 < 5l;
        int32_t v652; uint8_t v653;
        if (v639){
            uint8_t v640;
            v640 = v638.suit;
            bool v641;
            v641 = 1u == v640;
            if (v641){
                uint8_t v642;
                v642 = v638.rank;
                bool v643;
                v643 = v636 == v642;
                int32_t v644;
                if (v643){
                    v644 = v635;
                } else {
                    v644 = 0l;
                }
                v633[v644] = v638;
                int32_t v645;
                v645 = v644 + 1l;
                uint8_t v646;
                v646 = v638.rank;
                uint8_t v647;
                v647 = v646 - 1u;
                v652 = v645; v653 = v647;
            } else {
                v652 = v635; v653 = v636;
            }
        } else {
            break;
        }
        v635 = v652;
        v636 = v653;
        v634++;
    }
    bool v654;
    v654 = v635 == 4l;
    bool v689;
    if (v654){
        uint8_t v655;
        v655 = v636 + 1u;
        bool v656;
        v656 = v655 == 0u;
        if (v656){
            Card v657;
            v657 = v1[0l];
            uint8_t v658;
            v658 = v657.suit;
            bool v659;
            v659 = 1u == v658;
            bool v663;
            if (v659){
                uint8_t v660;
                v660 = v657.rank;
                bool v661;
                v661 = v660 == 12u;
                if (v661){
                    v633[4l] = v657;
                    v663 = true;
                } else {
                    v663 = false;
                }
            } else {
                v663 = false;
            }
            if (v663){
                v689 = true;
            } else {
                Card v664;
                v664 = v1[1l];
                uint8_t v665;
                v665 = v664.suit;
                bool v666;
                v666 = 1u == v665;
                bool v670;
                if (v666){
                    uint8_t v667;
                    v667 = v664.rank;
                    bool v668;
                    v668 = v667 == 12u;
                    if (v668){
                        v633[4l] = v664;
                        v670 = true;
                    } else {
                        v670 = false;
                    }
                } else {
                    v670 = false;
                }
                if (v670){
                    v689 = true;
                } else {
                    Card v671;
                    v671 = v1[2l];
                    uint8_t v672;
                    v672 = v671.suit;
                    bool v673;
                    v673 = 1u == v672;
                    bool v677;
                    if (v673){
                        uint8_t v674;
                        v674 = v671.rank;
                        bool v675;
                        v675 = v674 == 12u;
                        if (v675){
                            v633[4l] = v671;
                            v677 = true;
                        } else {
                            v677 = false;
                        }
                    } else {
                        v677 = false;
                    }
                    if (v677){
                        v689 = true;
                    } else {
                        Card v678;
                        v678 = v1[3l];
                        uint8_t v679;
                        v679 = v678.suit;
                        bool v680;
                        v680 = 1u == v679;
                        if (v680){
                            uint8_t v681;
                            v681 = v678.rank;
                            bool v682;
                            v682 = v681 == 12u;
                            if (v682){
                                v633[4l] = v678;
                                v689 = true;
                            } else {
                                v689 = false;
                            }
                        } else {
                            v689 = false;
                        }
                    }
                }
            }
        } else {
            v689 = false;
        }
    } else {
        v689 = false;
    }
    US1 v695;
    if (v689){
        v695 = US1_1(v633);
    } else {
        bool v691;
        v691 = v635 == 5l;
        if (v691){
            v695 = US1_1(v633);
        } else {
            v695 = US1_0();
        }
    }
    std::array<Card,5l> v696;
    int32_t v697; int32_t v698; uint8_t v699;
    Tuple7 tmp31 = TupleCreate7(0l, 0l, 12u);
    v697 = tmp31.v0; v698 = tmp31.v1; v699 = tmp31.v2;
    while (while_method_4(v697)){
        Card v701;
        v701 = v1[v697];
        bool v702;
        v702 = v698 < 5l;
        int32_t v715; uint8_t v716;
        if (v702){
            uint8_t v703;
            v703 = v701.suit;
            bool v704;
            v704 = 0u == v703;
            if (v704){
                uint8_t v705;
                v705 = v701.rank;
                bool v706;
                v706 = v699 == v705;
                int32_t v707;
                if (v706){
                    v707 = v698;
                } else {
                    v707 = 0l;
                }
                v696[v707] = v701;
                int32_t v708;
                v708 = v707 + 1l;
                uint8_t v709;
                v709 = v701.rank;
                uint8_t v710;
                v710 = v709 - 1u;
                v715 = v708; v716 = v710;
            } else {
                v715 = v698; v716 = v699;
            }
        } else {
            break;
        }
        v698 = v715;
        v699 = v716;
        v697++;
    }
    bool v717;
    v717 = v698 == 4l;
    bool v752;
    if (v717){
        uint8_t v718;
        v718 = v699 + 1u;
        bool v719;
        v719 = v718 == 0u;
        if (v719){
            Card v720;
            v720 = v1[0l];
            uint8_t v721;
            v721 = v720.suit;
            bool v722;
            v722 = 0u == v721;
            bool v726;
            if (v722){
                uint8_t v723;
                v723 = v720.rank;
                bool v724;
                v724 = v723 == 12u;
                if (v724){
                    v696[4l] = v720;
                    v726 = true;
                } else {
                    v726 = false;
                }
            } else {
                v726 = false;
            }
            if (v726){
                v752 = true;
            } else {
                Card v727;
                v727 = v1[1l];
                uint8_t v728;
                v728 = v727.suit;
                bool v729;
                v729 = 0u == v728;
                bool v733;
                if (v729){
                    uint8_t v730;
                    v730 = v727.rank;
                    bool v731;
                    v731 = v730 == 12u;
                    if (v731){
                        v696[4l] = v727;
                        v733 = true;
                    } else {
                        v733 = false;
                    }
                } else {
                    v733 = false;
                }
                if (v733){
                    v752 = true;
                } else {
                    Card v734;
                    v734 = v1[2l];
                    uint8_t v735;
                    v735 = v734.suit;
                    bool v736;
                    v736 = 0u == v735;
                    bool v740;
                    if (v736){
                        uint8_t v737;
                        v737 = v734.rank;
                        bool v738;
                        v738 = v737 == 12u;
                        if (v738){
                            v696[4l] = v734;
                            v740 = true;
                        } else {
                            v740 = false;
                        }
                    } else {
                        v740 = false;
                    }
                    if (v740){
                        v752 = true;
                    } else {
                        Card v741;
                        v741 = v1[3l];
                        uint8_t v742;
                        v742 = v741.suit;
                        bool v743;
                        v743 = 0u == v742;
                        if (v743){
                            uint8_t v744;
                            v744 = v741.rank;
                            bool v745;
                            v745 = v744 == 12u;
                            if (v745){
                                v696[4l] = v741;
                                v752 = true;
                            } else {
                                v752 = false;
                            }
                        } else {
                            v752 = false;
                        }
                    }
                }
            }
        } else {
            v752 = false;
        }
    } else {
        v752 = false;
    }
    US1 v758;
    if (v752){
        v758 = US1_1(v696);
    } else {
        bool v754;
        v754 = v698 == 5l;
        if (v754){
            v758 = US1_1(v696);
        } else {
            v758 = US1_0();
        }
    }
    US1 v784;
    switch (v758.tag) {
        case 0: { // None
            v784 = v695;
            break;
        }
        default: { // Some
            std::array<Card,5l> v759 = v758.v.case1.v0;
            switch (v695.tag) {
                case 0: { // None
                    v784 = v758;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v760 = v695.v.case1.v0;
                    US4 v761;
                    v761 = US4_0();
                    int32_t v762; US4 v763;
                    Tuple9 tmp32 = TupleCreate9(0l, v761);
                    v762 = tmp32.v0; v763 = tmp32.v1;
                    while (while_method_5(v762)){
                        Card v765;
                        v765 = v759[v762];
                        Card v766;
                        v766 = v760[v762];
                        US4 v777;
                        switch (v763.tag) {
                            case 0: { // Eq
                                uint8_t v767;
                                v767 = v765.rank;
                                uint8_t v768;
                                v768 = v766.rank;
                                bool v769;
                                v769 = v767 < v768;
                                if (v769){
                                    v777 = US4_2();
                                } else {
                                    bool v771;
                                    v771 = v767 > v768;
                                    if (v771){
                                        v777 = US4_1();
                                    } else {
                                        v777 = US4_0();
                                    }
                                }
                                break;
                            }
                            default: {
                                break;
                            }
                        }
                        v763 = v777;
                        v762++;
                    }
                    bool v778;
                    switch (v763.tag) {
                        case 1: { // Gt
                            v778 = true;
                            break;
                        }
                        default: {
                            v778 = false;
                        }
                    }
                    std::array<Card,5l> v779;
                    if (v778){
                        v779 = v759;
                    } else {
                        v779 = v760;
                    }
                    v784 = US1_1(v779);
                }
            }
        }
    }
    US1 v810;
    switch (v784.tag) {
        case 0: { // None
            v810 = v632;
            break;
        }
        default: { // Some
            std::array<Card,5l> v785 = v784.v.case1.v0;
            switch (v632.tag) {
                case 0: { // None
                    v810 = v784;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v786 = v632.v.case1.v0;
                    US4 v787;
                    v787 = US4_0();
                    int32_t v788; US4 v789;
                    Tuple9 tmp33 = TupleCreate9(0l, v787);
                    v788 = tmp33.v0; v789 = tmp33.v1;
                    while (while_method_5(v788)){
                        Card v791;
                        v791 = v785[v788];
                        Card v792;
                        v792 = v786[v788];
                        US4 v803;
                        switch (v789.tag) {
                            case 0: { // Eq
                                uint8_t v793;
                                v793 = v791.rank;
                                uint8_t v794;
                                v794 = v792.rank;
                                bool v795;
                                v795 = v793 < v794;
                                if (v795){
                                    v803 = US4_2();
                                } else {
                                    bool v797;
                                    v797 = v793 > v794;
                                    if (v797){
                                        v803 = US4_1();
                                    } else {
                                        v803 = US4_0();
                                    }
                                }
                                break;
                            }
                            default: {
                                break;
                            }
                        }
                        v789 = v803;
                        v788++;
                    }
                    bool v804;
                    switch (v789.tag) {
                        case 1: { // Gt
                            v804 = true;
                            break;
                        }
                        default: {
                            v804 = false;
                        }
                    }
                    std::array<Card,5l> v805;
                    if (v804){
                        v805 = v785;
                    } else {
                        v805 = v786;
                    }
                    v810 = US1_1(v805);
                }
            }
        }
    }
    US1 v836;
    switch (v810.tag) {
        case 0: { // None
            v836 = v569;
            break;
        }
        default: { // Some
            std::array<Card,5l> v811 = v810.v.case1.v0;
            switch (v569.tag) {
                case 0: { // None
                    v836 = v810;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v812 = v569.v.case1.v0;
                    US4 v813;
                    v813 = US4_0();
                    int32_t v814; US4 v815;
                    Tuple9 tmp34 = TupleCreate9(0l, v813);
                    v814 = tmp34.v0; v815 = tmp34.v1;
                    while (while_method_5(v814)){
                        Card v817;
                        v817 = v811[v814];
                        Card v818;
                        v818 = v812[v814];
                        US4 v829;
                        switch (v815.tag) {
                            case 0: { // Eq
                                uint8_t v819;
                                v819 = v817.rank;
                                uint8_t v820;
                                v820 = v818.rank;
                                bool v821;
                                v821 = v819 < v820;
                                if (v821){
                                    v829 = US4_2();
                                } else {
                                    bool v823;
                                    v823 = v819 > v820;
                                    if (v823){
                                        v829 = US4_1();
                                    } else {
                                        v829 = US4_0();
                                    }
                                }
                                break;
                            }
                            default: {
                                break;
                            }
                        }
                        v815 = v829;
                        v814++;
                    }
                    bool v830;
                    switch (v815.tag) {
                        case 1: { // Gt
                            v830 = true;
                            break;
                        }
                        default: {
                            v830 = false;
                        }
                    }
                    std::array<Card,5l> v831;
                    if (v830){
                        v831 = v811;
                    } else {
                        v831 = v812;
                    }
                    v836 = US1_1(v831);
                }
            }
        }
    }
    US7 v841;
    switch (v59.tag) {
        case 0: { // None
            v841 = US7_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v837 = v59.v.case1.v0;
            v841 = US7_1(v837, 1);
        }
    }
    US7 v846;
    switch (v148.tag) {
        case 0: { // None
            v846 = US7_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v842 = v148.v.case1.v0;
            v846 = US7_1(v842, 2);
        }
    }
    US7 v851;
    switch (v198.tag) {
        case 0: { // None
            v851 = US7_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v847 = v198.v.case1.v0;
            v851 = US7_1(v847, 3);
        }
    }
    US7 v856;
    switch (v236.tag) {
        case 0: { // None
            v856 = US7_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v852 = v236.v.case1.v0;
            v856 = US7_1(v852, 4);
        }
    }
    US7 v861;
    switch (v374.tag) {
        case 0: { // None
            v861 = US7_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v857 = v374.v.case1.v0;
            v861 = US7_1(v857, 5);
        }
    }
    US7 v866;
    switch (v456.tag) {
        case 0: { // None
            v866 = US7_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v862 = v456.v.case1.v0;
            v866 = US7_1(v862, 6);
        }
    }
    US7 v871;
    switch (v506.tag) {
        case 0: { // None
            v871 = US7_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v867 = v506.v.case1.v0;
            v871 = US7_1(v867, 7);
        }
    }
    US7 v876;
    switch (v836.tag) {
        case 0: { // None
            v876 = US7_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v872 = v836.v.case1.v0;
            v876 = US7_1(v872, 8);
        }
    }
    US7 v878;
    switch (v876.tag) {
        case 0: { // None
            v878 = US7_0();
            break;
        }
        default: {
            v878 = v876;
        }
    }
    US7 v888;
    switch (v878.tag) {
        case 1: { // Some
            std::array<Card,5l> v879 = v878.v.case1.v0; int8_t v880 = v878.v.case1.v1;
            switch (v871.tag) {
                case 0: { // None
                    v888 = v878;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v881 = v871.v.case1.v0; int8_t v882 = v871.v.case1.v1;
                    v888 = US7_1(v879, v880);
                }
            }
            break;
        }
        default: {
            switch (v871.tag) {
                case 0: { // None
                    v888 = v878;
                    break;
                }
                default: {
                    switch (v878.tag) {
                        default: { // None
                            v888 = v871;
                        }
                    }
                }
            }
        }
    }
    US7 v898;
    switch (v888.tag) {
        case 1: { // Some
            std::array<Card,5l> v889 = v888.v.case1.v0; int8_t v890 = v888.v.case1.v1;
            switch (v866.tag) {
                case 0: { // None
                    v898 = v888;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v891 = v866.v.case1.v0; int8_t v892 = v866.v.case1.v1;
                    v898 = US7_1(v889, v890);
                }
            }
            break;
        }
        default: {
            switch (v866.tag) {
                case 0: { // None
                    v898 = v888;
                    break;
                }
                default: {
                    switch (v888.tag) {
                        default: { // None
                            v898 = v866;
                        }
                    }
                }
            }
        }
    }
    US7 v908;
    switch (v898.tag) {
        case 1: { // Some
            std::array<Card,5l> v899 = v898.v.case1.v0; int8_t v900 = v898.v.case1.v1;
            switch (v861.tag) {
                case 0: { // None
                    v908 = v898;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v901 = v861.v.case1.v0; int8_t v902 = v861.v.case1.v1;
                    v908 = US7_1(v899, v900);
                }
            }
            break;
        }
        default: {
            switch (v861.tag) {
                case 0: { // None
                    v908 = v898;
                    break;
                }
                default: {
                    switch (v898.tag) {
                        default: { // None
                            v908 = v861;
                        }
                    }
                }
            }
        }
    }
    US7 v918;
    switch (v908.tag) {
        case 1: { // Some
            std::array<Card,5l> v909 = v908.v.case1.v0; int8_t v910 = v908.v.case1.v1;
            switch (v856.tag) {
                case 0: { // None
                    v918 = v908;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v911 = v856.v.case1.v0; int8_t v912 = v856.v.case1.v1;
                    v918 = US7_1(v909, v910);
                }
            }
            break;
        }
        default: {
            switch (v856.tag) {
                case 0: { // None
                    v918 = v908;
                    break;
                }
                default: {
                    switch (v908.tag) {
                        default: { // None
                            v918 = v856;
                        }
                    }
                }
            }
        }
    }
    US7 v928;
    switch (v918.tag) {
        case 1: { // Some
            std::array<Card,5l> v919 = v918.v.case1.v0; int8_t v920 = v918.v.case1.v1;
            switch (v851.tag) {
                case 0: { // None
                    v928 = v918;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v921 = v851.v.case1.v0; int8_t v922 = v851.v.case1.v1;
                    v928 = US7_1(v919, v920);
                }
            }
            break;
        }
        default: {
            switch (v851.tag) {
                case 0: { // None
                    v928 = v918;
                    break;
                }
                default: {
                    switch (v918.tag) {
                        default: { // None
                            v928 = v851;
                        }
                    }
                }
            }
        }
    }
    US7 v938;
    switch (v928.tag) {
        case 1: { // Some
            std::array<Card,5l> v929 = v928.v.case1.v0; int8_t v930 = v928.v.case1.v1;
            switch (v846.tag) {
                case 0: { // None
                    v938 = v928;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v931 = v846.v.case1.v0; int8_t v932 = v846.v.case1.v1;
                    v938 = US7_1(v929, v930);
                }
            }
            break;
        }
        default: {
            switch (v846.tag) {
                case 0: { // None
                    v938 = v928;
                    break;
                }
                default: {
                    switch (v928.tag) {
                        default: { // None
                            v938 = v846;
                        }
                    }
                }
            }
        }
    }
    US7 v948;
    switch (v938.tag) {
        case 1: { // Some
            std::array<Card,5l> v939 = v938.v.case1.v0; int8_t v940 = v938.v.case1.v1;
            switch (v841.tag) {
                case 0: { // None
                    v948 = v938;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v941 = v841.v.case1.v0; int8_t v942 = v841.v.case1.v1;
                    v948 = US7_1(v939, v940);
                }
            }
            break;
        }
        default: {
            switch (v841.tag) {
                case 0: { // None
                    v948 = v938;
                    break;
                }
                default: {
                    switch (v938.tag) {
                        default: { // None
                            v948 = v841;
                        }
                    }
                }
            }
        }
    }
    std::array<Card,5l> v953; int8_t v954;
    switch (v948.tag) {
        case 0: { // None
            v953 = v6; v954 = 0;
            break;
        }
        default: { // Some
            std::array<Card,5l> v949 = v948.v.case1.v0; int8_t v950 = v948.v.case1.v1;
            v953 = v949; v954 = v950;
        }
    }
    return TupleCreate5(v953, v954);
}
int32_t main() {
    std::random_device v0;
    std::mt19937 v1(v0());
    std::mt19937 & v2 = v1;
    std::uniform_int_distribution<int32_t> v3(0l, 51l);
    uint64_t v4; int32_t v5;
    Tuple0 tmp0 = TupleCreate0(0ull, 0l);
    v4 = tmp0.v0; v5 = tmp0.v1;
    while (while_method_0(v4)){
        std::array<Tuple1,7l> v7;
        uint64_t v8;
        v8 = 0ull;
        int32_t v9;
        v9 = 7l;
        while (while_method_1(v9)){
            int32_t v11;
            v11 = v3(v2);
            int8_t v12;
            v12 = (int8_t)v11;
            int32_t v13;
            v13 = (int32_t)v12;
            uint64_t v14;
            v14 = 1ull << v13;
            uint64_t v15;
            v15 = v8 & v14;
            bool v16;
            v16 = v15 == 0ull;
            bool v17;
            v17 = v16 != true;
            int32_t v22;
            if (v17){
                v22 = v9;
            } else {
                int32_t v18;
                v18 = v9 - 1l;
                int8_t v19;
                v19 = v12 % 13;
                int8_t v20;
                v20 = v12 / 13;
                v7[v18] = TupleCreate1(v19, v20);
                uint64_t v21;
                v21 = v8 | v14;
                v8 = v21;
                v22 = v18;
            }
            v9 = v22;
        }
        int8_t v23; int8_t v24;
        Tuple1 tmp1 = v7[0l];
        v23 = tmp1.v0; v24 = tmp1.v1;
        int8_t v25; int8_t v26;
        Tuple1 tmp2 = v7[1l];
        v25 = tmp2.v0; v26 = tmp2.v1;
        int8_t v27; int8_t v28;
        Tuple1 tmp3 = v7[2l];
        v27 = tmp3.v0; v28 = tmp3.v1;
        int8_t v29; int8_t v30;
        Tuple1 tmp4 = v7[3l];
        v29 = tmp4.v0; v30 = tmp4.v1;
        int8_t v31; int8_t v32;
        Tuple1 tmp5 = v7[4l];
        v31 = tmp5.v0; v32 = tmp5.v1;
        int8_t v33; int8_t v34;
        Tuple1 tmp6 = v7[5l];
        v33 = tmp6.v0; v34 = tmp6.v1;
        int8_t v35; int8_t v36;
        Tuple1 tmp7 = v7[6l];
        v35 = tmp7.v0; v36 = tmp7.v1;
        int8_t v37;
        v37 = v36 * 13;
        int8_t v38;
        v38 = v37 + v35;
        int32_t v39;
        v39 = (int32_t)v38;
        uint64_t v40;
        v40 = 1ull << v39;
        int8_t v41;
        v41 = v34 * 13;
        int8_t v42;
        v42 = v41 + v33;
        int32_t v43;
        v43 = (int32_t)v42;
        uint64_t v44;
        v44 = 1ull << v43;
        int8_t v45;
        v45 = v32 * 13;
        int8_t v46;
        v46 = v45 + v31;
        int32_t v47;
        v47 = (int32_t)v46;
        uint64_t v48;
        v48 = 1ull << v47;
        int8_t v49;
        v49 = v30 * 13;
        int8_t v50;
        v50 = v49 + v29;
        int32_t v51;
        v51 = (int32_t)v50;
        uint64_t v52;
        v52 = 1ull << v51;
        int8_t v53;
        v53 = v28 * 13;
        int8_t v54;
        v54 = v53 + v27;
        int32_t v55;
        v55 = (int32_t)v54;
        uint64_t v56;
        v56 = 1ull << v55;
        int8_t v57;
        v57 = v26 * 13;
        int8_t v58;
        v58 = v57 + v25;
        int32_t v59;
        v59 = (int32_t)v58;
        uint64_t v60;
        v60 = 1ull << v59;
        int8_t v61;
        v61 = v24 * 13;
        int8_t v62;
        v62 = v61 + v23;
        int32_t v63;
        v63 = (int32_t)v62;
        uint64_t v64;
        v64 = 1ull << v63;
        uint64_t v65;
        v65 = 0ull | v64;
        uint64_t v66;
        v66 = v65 | v60;
        uint64_t v67;
        v67 = v66 | v56;
        uint64_t v68;
        v68 = v67 | v52;
        uint64_t v69;
        v69 = v68 | v48;
        uint64_t v70;
        v70 = v69 | v44;
        uint64_t v71;
        v71 = v70 | v40;
        int8_t v72; int8_t v73; int8_t v74; int8_t v75; int8_t v76; int8_t v77;
        Tuple2 tmp12 = score__0(v71);
        v72 = tmp12.v0; v73 = tmp12.v1; v74 = tmp12.v2; v75 = tmp12.v3; v76 = tmp12.v4; v77 = tmp12.v5;
        int8_t v78;
        v78 = v72 / 13;
        int8_t v79;
        v79 = v72 % 13;
        int8_t v80;
        v80 = v73 / 13;
        int8_t v81;
        v81 = v73 % 13;
        int8_t v82;
        v82 = v74 / 13;
        int8_t v83;
        v83 = v74 % 13;
        int8_t v84;
        v84 = v75 / 13;
        int8_t v85;
        v85 = v75 % 13;
        int8_t v86;
        v86 = v76 / 13;
        int8_t v87;
        v87 = v76 % 13;
        std::array<Card,7l> v88;
        uint8_t v89;
        v89 = (uint8_t)v23;
        uint8_t v90;
        v90 = (uint8_t)v24;
        Card v91;
        v91 = {v89, v90};
        v88[0l] = v91;
        uint8_t v92;
        v92 = (uint8_t)v25;
        uint8_t v93;
        v93 = (uint8_t)v26;
        Card v94;
        v94 = {v92, v93};
        v88[1l] = v94;
        uint8_t v95;
        v95 = (uint8_t)v27;
        uint8_t v96;
        v96 = (uint8_t)v28;
        Card v97;
        v97 = {v95, v96};
        v88[2l] = v97;
        uint8_t v98;
        v98 = (uint8_t)v29;
        uint8_t v99;
        v99 = (uint8_t)v30;
        Card v100;
        v100 = {v98, v99};
        v88[3l] = v100;
        uint8_t v101;
        v101 = (uint8_t)v31;
        uint8_t v102;
        v102 = (uint8_t)v32;
        Card v103;
        v103 = {v101, v102};
        v88[4l] = v103;
        uint8_t v104;
        v104 = (uint8_t)v33;
        uint8_t v105;
        v105 = (uint8_t)v34;
        Card v106;
        v106 = {v104, v105};
        v88[5l] = v106;
        uint8_t v107;
        v107 = (uint8_t)v35;
        uint8_t v108;
        v108 = (uint8_t)v36;
        Card v109;
        v109 = {v107, v108};
        v88[6l] = v109;
        std::array<Card,5l> v110; int8_t v111;
        Tuple5 tmp35 = score_21(v88);
        v110 = tmp35.v0; v111 = tmp35.v1;
        Card v112;
        v112 = v110[0l];
        uint8_t v113;
        v113 = v112.suit;
        int8_t v114;
        v114 = (int8_t)v113;
        uint8_t v115;
        v115 = v112.rank;
        int8_t v116;
        v116 = (int8_t)v115;
        Card v117;
        v117 = v110[1l];
        uint8_t v118;
        v118 = v117.suit;
        int8_t v119;
        v119 = (int8_t)v118;
        uint8_t v120;
        v120 = v117.rank;
        int8_t v121;
        v121 = (int8_t)v120;
        Card v122;
        v122 = v110[2l];
        uint8_t v123;
        v123 = v122.suit;
        int8_t v124;
        v124 = (int8_t)v123;
        uint8_t v125;
        v125 = v122.rank;
        int8_t v126;
        v126 = (int8_t)v125;
        Card v127;
        v127 = v110[3l];
        uint8_t v128;
        v128 = v127.suit;
        int8_t v129;
        v129 = (int8_t)v128;
        uint8_t v130;
        v130 = v127.rank;
        int8_t v131;
        v131 = (int8_t)v130;
        Card v132;
        v132 = v110[4l];
        uint8_t v133;
        v133 = v132.suit;
        int8_t v134;
        v134 = (int8_t)v133;
        uint8_t v135;
        v135 = v132.rank;
        int8_t v136;
        v136 = (int8_t)v135;
        bool v137;
        v137 = v79 == v116;
        bool v139;
        if (v137){
            bool v138;
            v138 = v78 == v114;
            v139 = v138;
        } else {
            v139 = false;
        }
        bool v155;
        if (v139){
            bool v140;
            v140 = v81 == v121;
            bool v142;
            if (v140){
                bool v141;
                v141 = v80 == v119;
                v142 = v141;
            } else {
                v142 = false;
            }
            if (v142){
                bool v143;
                v143 = v83 == v126;
                bool v145;
                if (v143){
                    bool v144;
                    v144 = v82 == v124;
                    v145 = v144;
                } else {
                    v145 = false;
                }
                if (v145){
                    bool v146;
                    v146 = v85 == v131;
                    bool v148;
                    if (v146){
                        bool v147;
                        v147 = v84 == v129;
                        v148 = v147;
                    } else {
                        v148 = false;
                    }
                    if (v148){
                        bool v149;
                        v149 = v87 == v136;
                        if (v149){
                            bool v150;
                            v150 = v86 == v134;
                            v155 = v150;
                        } else {
                            v155 = false;
                        }
                    } else {
                        v155 = false;
                    }
                } else {
                    v155 = false;
                }
            } else {
                v155 = false;
            }
        } else {
            v155 = false;
        }
        bool v157;
        if (v155){
            bool v156;
            v156 = v77 == v111;
            v157 = v156;
        } else {
            v157 = false;
        }
        bool v158;
        v158 = v157 != true;
        int32_t v160;
        if (v158){
            std::cout << "{rank = " << (int) v23 << "; suit = " << (int) v24 << "}; " ;
            std::cout << "{rank = " << (int) v25 << "; suit = " << (int) v26 << "}; " ;
            std::cout << "{rank = " << (int) v27 << "; suit = " << (int) v28 << "}; " ;
            std::cout << "{rank = " << (int) v29 << "; suit = " << (int) v30 << "}; " ;
            std::cout << "{rank = " << (int) v31 << "; suit = " << (int) v32 << "}; " ;
            std::cout << "{rank = " << (int) v33 << "; suit = " << (int) v34 << "}; " ;
            std::cout << "{rank = " << (int) v35 << "; suit = " << (int) v36 << "}; " ;
            std::cout << std::endl;
            std::cout << "Score: " << (int) v77 << " " ;
            std::cout << "Card: ";
            std::cout << "(" << (int) v79 << "," << (int) v78 << ") " ;
            std::cout << "(" << (int) v81 << "," << (int) v80 << ") " ;
            std::cout << "(" << (int) v83 << "," << (int) v82 << ") " ;
            std::cout << "(" << (int) v85 << "," << (int) v84 << ") " ;
            std::cout << "(" << (int) v87 << "," << (int) v86 << ") " ;
            std::cout << std::endl;
            std::cout << "Score: " << (int) v111 << " " ;
            std::cout << "Card: ";
            std::cout << "(" << (int) v116 << "," << (int) v114 << ") " ;
            std::cout << "(" << (int) v121 << "," << (int) v119 << ") " ;
            std::cout << "(" << (int) v126 << "," << (int) v124 << ") " ;
            std::cout << "(" << (int) v131 << "," << (int) v129 << ") " ;
            std::cout << "(" << (int) v136 << "," << (int) v134 << ") " ;
            std::cout << std::endl;
            int32_t v159;
            v159 = v5 + 1l;
            v160 = v159;
        } else {
            v160 = v5;
        }
        v5 = v160;
        v4++;
    }
    std::cout << "The number of errors is: " << v5 << std::endl;
    return v5;
}
