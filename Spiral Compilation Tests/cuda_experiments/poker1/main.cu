#include "hip/hip_runtime.h"
#pragma warning(disable: 4101 4065 4060)
// Add these as extra argument to the compiler to suppress the rest:
// --diag-suppress 186 --diag-suppress 177 --diag-suppress 550
#include <cstdint>
#include <array>
#include <iostream>
#include <random>
#include <bitset>
struct Card { uint8_t rank : 4; uint8_t suit : 2; };
#include <cmath>
#include <limits>
#include <algorithm>
struct Tuple0;
int32_t random_int_4(int32_t v0, int32_t v1, std::mt19937 & v2);
struct Tuple1;
int32_t sample_without_3(std::bitset<52l> & v0, std::mt19937 & v1);
Card draw_card_2(std::bitset<52l> & v0, std::mt19937 & v1);
struct US0;
struct US1;
struct Tuple2;
struct US2;
struct Tuple3;
float random_f32_template_7(bool v0, std::mt19937 & v1);
struct US3;
struct Tuple4;
int32_t sample_discrete__6(std::array<float,8l> v0, std::mt19937 & v1);
US2 sample_discrete_5(std::array<Tuple3,8l> v0, std::mt19937 & v1);
struct Tuple5;
typedef bool (* Fun0)(Card, Card);
struct Tuple6;
struct US4;
struct US5;
struct US6;
struct US7;
struct Tuple7;
struct Tuple8;
struct US8;
struct Tuple9;
struct US9;
struct US10;
struct US11;
Tuple5 score_9(std::array<Card,7l> v0);
Tuple5 score_8(Card v0, Card v1, Card v2, Card v3, Card v4, Card v5, Card v6);
struct Tuple10;
US8 method_10(std::array<Card,5l> v0, std::array<Card,5l> v1);
int16_t game_1(std::bitset<52l> & v0, std::mt19937 & v1);
int16_t game_loop_0();
struct Tuple0 {
    int32_t v0;
    int16_t v1;
    Tuple0(int32_t t0, int16_t t1) : v0(t0), v1(t1) {}
    Tuple0() = default;
};
struct Tuple1 {
    int32_t v0;
    int32_t v1;
    int32_t v2;
    Tuple1(int32_t t0, int32_t t1, int32_t t2) : v0(t0), v1(t1), v2(t2) {}
    Tuple1() = default;
};
struct US0 {
    union {
        struct {
            Card v0;
            Card v1;
        } case1; // Some
    } v;
    char tag : 2;
};
struct US1 {
    union {
        struct {
            int8_t v0;
            int8_t v1;
        } case1; // TurnOf
    } v;
    char tag : 2;
};
struct Tuple2 {
    US0 v1;
    US0 v3;
    US1 v5;
    int16_t v2;
    int16_t v4;
    uint8_t v0;
    Tuple2(uint8_t t0, US0 t1, int16_t t2, US0 t3, int16_t t4, US1 t5) : v0(t0), v1(t1), v2(t2), v3(t3), v4(t4), v5(t5) {}
    Tuple2() = default;
};
struct US2 {
    union {
        struct {
            int16_t v0;
        } case2; // RaiseTo
    } v;
    char tag : 2;
};
struct Tuple3 {
    US2 v0;
    float v1;
    Tuple3(US2 t0, float t1) : v0(t0), v1(t1) {}
    Tuple3() = default;
};
struct US3 {
    union {
        struct {
            int32_t v0;
        } case1; // Some
    } v;
    char tag : 2;
};
struct Tuple4 {
    US3 v1;
    int32_t v0;
    Tuple4(int32_t t0, US3 t1) : v0(t0), v1(t1) {}
    Tuple4() = default;
};
struct Tuple5 {
    std::array<Card,5l> v0;
    int8_t v1;
    Tuple5(std::array<Card,5l> t0, int8_t t1) : v0(t0), v1(t1) {}
    Tuple5() = default;
};
struct Tuple6 {
    int32_t v0;
    int32_t v1;
    int32_t v2;
    uint8_t v3;
    Tuple6(int32_t t0, int32_t t1, int32_t t2, uint8_t t3) : v0(t0), v1(t1), v2(t2), v3(t3) {}
    Tuple6() = default;
};
struct US4 {
    union {
        struct {
            std::array<Card,2l> v0;
            std::array<Card,5l> v1;
        } case1; // Some
    } v;
    char tag : 2;
};
struct US5 {
    union {
        struct {
            std::array<Card,5l> v0;
        } case1; // Some
    } v;
    char tag : 2;
};
struct US6 {
    union {
        struct {
            std::array<Card,2l> v0;
            std::array<Card,3l> v1;
        } case1; // Some
    } v;
    char tag : 2;
};
struct US7 {
    union {
        struct {
            std::array<Card,3l> v0;
            std::array<Card,4l> v1;
        } case1; // Some
    } v;
    char tag : 2;
};
struct Tuple7 {
    int32_t v0;
    int32_t v1;
    uint8_t v2;
    Tuple7(int32_t t0, int32_t t1, uint8_t t2) : v0(t0), v1(t1), v2(t2) {}
    Tuple7() = default;
};
struct Tuple8 {
    int32_t v0;
    int32_t v1;
    Tuple8(int32_t t0, int32_t t1) : v0(t0), v1(t1) {}
    Tuple8() = default;
};
struct US8 {
    union {
    } v;
    char tag : 2;
};
struct Tuple9 {
    US8 v1;
    int32_t v0;
    Tuple9(int32_t t0, US8 t1) : v0(t0), v1(t1) {}
    Tuple9() = default;
};
struct US9 {
    union {
        struct {
            std::array<Card,2l> v0;
            std::array<Card,2l> v1;
        } case1; // Some
    } v;
    char tag : 2;
};
struct US10 {
    union {
        struct {
            std::array<Card,4l> v0;
            std::array<Card,3l> v1;
        } case1; // Some
    } v;
    char tag : 2;
};
struct US11 {
    union {
        struct {
            std::array<Card,5l> v0;
            int8_t v1;
        } case1; // Some
    } v;
    char tag : 2;
};
struct Tuple10 {
    US8 v0;
    int32_t v1;
    Tuple10(US8 t0, int32_t t1) : v0(t0), v1(t1) {}
    Tuple10() = default;
};
inline bool while_method_0(int32_t v0){
    bool v1;
    v1 = v0 < 10000000l;
    return v1;
}
int32_t random_int_4(int32_t v0, int32_t v1, std::mt19937 & v2){
    std::uniform_int_distribution<int32_t> v3(v0, v1);
    int32_t v4;
    v4 = v3(v2);
    return v4;
}
inline bool while_method_1(int32_t v0){
    bool v1;
    v1 = v0 < 52l;
    return v1;
}
int32_t sample_without_3(std::bitset<52l> & v0, std::mt19937 & v1){
    int32_t v2;
    v2 = v0.count();
    int32_t v3;
    v3 = 52l - v2;
    int32_t v4;
    v4 = v3 - 1l;
    int32_t v5;
    v5 = 0l;
    int32_t v6;
    v6 = random_int_4(v5, v4, v1);
    int32_t v7;
    v7 = v6 + 1l;
    int32_t v8; int32_t v9; int32_t v10;
    Tuple1 tmp1 = Tuple1(0l, 0l, v7);
    v8 = tmp1.v0; v9 = tmp1.v1; v10 = tmp1.v2;
    while (while_method_1(v8)){
        bool v12;
        v12 = v10 > 0l;
        int32_t v18; int32_t v19;
        if (v12){
            bool v13;
            v13 = v0[v8];
            int32_t v15;
            if (v13){
                v15 = v10;
            } else {
                int32_t v14;
                v14 = v10 - 1l;
                v15 = v14;
            }
            v18 = v8; v19 = v15;
        } else {
            break;
        }
        v9 = v18;
        v10 = v19;
        v8++;
    }
    v0.set(v9,true);
    return v9;
}
Card draw_card_2(std::bitset<52l> & v0, std::mt19937 & v1){
    int32_t v2;
    v2 = sample_without_3(v0, v1);
    int32_t v3;
    v3 = v2 % 13l;
    uint8_t v4;
    v4 = (uint8_t)v3;
    int32_t v5;
    v5 = v2 / 13l;
    uint8_t v6;
    v6 = (uint8_t)v5;
    Card v7;
    v7 = {v4, v6};
    return v7;
}
US0 US0_0() { // None
    US0 x;
    x.tag = 0;
    return x;
}
US0 US0_1(Card v0, Card v1) { // Some
    US0 x;
    x.tag = 1;
    x.v.case1.v0 = v0; x.v.case1.v1 = v1;
    return x;
}
US1 US1_0() { // Done
    US1 x;
    x.tag = 0;
    return x;
}
US1 US1_1(int8_t v0, int8_t v1) { // TurnOf
    US1 x;
    x.tag = 1;
    x.v.case1.v0 = v0; x.v.case1.v1 = v1;
    return x;
}
inline bool while_method_2(uint8_t v0, US0 v1, int16_t v2, US0 v3, int16_t v4, US1 v5){
    bool v6;
    v6 = v0 > 0u;
    if (v6){
        bool v8;
        switch (v5.tag) {
            case 0: { // Done
                v8 = true;
                break;
            }
            default: {
                v8 = false;
            }
        }
        bool v9;
        v9 = v8 != true;
        return v9;
    } else {
        return false;
    }
}
US2 US2_0() { // Call
    US2 x;
    x.tag = 0;
    return x;
}
US2 US2_1() { // Fold
    US2 x;
    x.tag = 1;
    return x;
}
US2 US2_2(int16_t v0) { // RaiseTo
    US2 x;
    x.tag = 2;
    x.v.case2.v0 = v0;
    return x;
}
inline bool while_method_3(int32_t v0){
    bool v1;
    v1 = v0 < 8l;
    return v1;
}
inline bool while_method_4(std::array<float,8l> v0, int32_t v1){
    bool v2;
    v2 = v1 < 8l;
    return v2;
}
inline bool while_method_5(int32_t v0, int32_t v1){
    bool v2;
    v2 = v1 > v0;
    return v2;
}
float random_f32_template_7(bool v0, std::mt19937 & v1){
    float v3;
    if (v0){
        float v2;
        v2 = std::nextafter(0.0f, std::numeric_limits<float>::max());
        v3 = v2;
    } else {
        v3 = 0.0f;
    }
    std::uniform_real_distribution<float> v4(v3, 1.0f);
    float v5;
    v5 = v4(v1);
    return v5;
}
US3 US3_0() { // None
    US3 x;
    x.tag = 0;
    return x;
}
US3 US3_1(int32_t v0) { // Some
    US3 x;
    x.tag = 1;
    x.v.case1.v0 = v0;
    return x;
}
int32_t sample_discrete__6(std::array<float,8l> v0, std::mt19937 & v1){
    std::array<float,8l> v2;
    int32_t v3;
    v3 = 0l;
    while (while_method_3(v3)){
        float v5;
        v5 = v0[v3];
        bool v6;
        v6 = 0.0f >= v5;
        float v7;
        if (v6){
            v7 = 0.0f;
        } else {
            v7 = v5;
        }
        v2[v3] = v7;
        v3++;
    }
    std::array<float,8l> v8;
    int32_t v9;
    v9 = 0l;
    while (while_method_3(v9)){
        float v11;
        v11 = v2[v9];
        v8[v9] = v11;
        v9++;
    }
    int32_t v12;
    v12 = 1l;
    while (while_method_4(v8, v12)){
        int32_t v14;
        v14 = 8l;
        while (while_method_5(v12, v14)){
            v14--;
            int32_t v16;
            v16 = v14 - v12;
            float v17;
            v17 = v8[v16];
            float v18;
            v18 = v8[v14];
            float v19;
            v19 = v17 + v18;
            v8[v14] = v19;
        }
        int32_t v20;
        v20 = v12 * 2l;
        v12 = v20;
    }
    float v21;
    v21 = v8[7l];
    std::array<float,8l> v22;
    int32_t v23;
    v23 = 0l;
    while (while_method_3(v23)){
        float v25;
        v25 = v8[v23];
        float v26;
        v26 = v25 / v21;
        v22[v23] = v26;
        v23++;
    }
    bool v27;
    v27 = false;
    float v28;
    v28 = random_f32_template_7(v27, v1);
    US3 v29;
    v29 = US3_0();
    int32_t v30; US3 v31;
    Tuple4 tmp4 = Tuple4(0l, v29);
    v30 = tmp4.v0; v31 = tmp4.v1;
    while (while_method_3(v30)){
        float v33;
        v33 = v22[v30];
        US3 v37;
        switch (v31.tag) {
            case 0: { // None
                bool v34;
                v34 = v28 < v33;
                if (v34){
                    v37 = US3_1(v30);
                } else {
                    v37 = v31;
                }
                break;
            }
            default: {
                v37 = v31;
            }
        }
        v31 = v37;
        v30++;
    }
    switch (v31.tag) {
        case 0: { // None
            return 0l;
            break;
        }
        default: { // Some
            int32_t v38 = v31.v.case1.v0;
            return v38;
        }
    }
}
US2 sample_discrete_5(std::array<Tuple3,8l> v0, std::mt19937 & v1){
    std::array<float,8l> v2;
    int32_t v3;
    v3 = 0l;
    while (while_method_3(v3)){
        US2 v5; float v6;
        Tuple3 tmp3 = v0[v3];
        v5 = tmp3.v0; v6 = tmp3.v1;
        v2[v3] = v6;
        v3++;
    }
    int32_t v7;
    v7 = sample_discrete__6(v2, v1);
    US2 v8; float v9;
    Tuple3 tmp5 = v0[v7];
    v8 = tmp5.v0; v9 = tmp5.v1;
    return v8;
}
inline bool while_method_6(int32_t v0){
    bool v1;
    v1 = v0 < 7l;
    return v1;
}
bool ClosureMethod0(Card tup0, Card tup1){
    Card v0 = tup0; Card v1 = tup1;
    uint8_t v2;
    v2 = v0.rank;
    uint8_t v3;
    v3 = v1.rank;
    bool v4;
    v4 = v2 > v3;
    if (v4){
        return true;
    } else {
        uint8_t v5;
        v5 = v0.rank;
        uint8_t v6;
        v6 = v1.rank;
        bool v7;
        v7 = v5 == v6;
        if (v7){
            uint8_t v8;
            v8 = v0.suit;
            uint8_t v9;
            v9 = v1.suit;
            bool v10;
            v10 = v8 < v9;
            return v10;
        } else {
            return false;
        }
    }
}
inline bool while_method_7(int32_t v0){
    bool v1;
    v1 = v0 < 5l;
    return v1;
}
US4 US4_0() { // None
    US4 x;
    x.tag = 0;
    return x;
}
US4 US4_1(std::array<Card,2l> v0, std::array<Card,5l> v1) { // Some
    US4 x;
    x.tag = 1;
    x.v.case1.v0 = v0; x.v.case1.v1 = v1;
    return x;
}
US5 US5_0() { // None
    US5 x;
    x.tag = 0;
    return x;
}
US5 US5_1(std::array<Card,5l> v0) { // Some
    US5 x;
    x.tag = 1;
    x.v.case1.v0 = v0;
    return x;
}
inline bool while_method_8(int32_t v0){
    bool v1;
    v1 = v0 < 3l;
    return v1;
}
inline bool while_method_9(int32_t v0){
    bool v1;
    v1 = v0 < 2l;
    return v1;
}
US6 US6_0() { // None
    US6 x;
    x.tag = 0;
    return x;
}
US6 US6_1(std::array<Card,2l> v0, std::array<Card,3l> v1) { // Some
    US6 x;
    x.tag = 1;
    x.v.case1.v0 = v0; x.v.case1.v1 = v1;
    return x;
}
inline bool while_method_10(int32_t v0){
    bool v1;
    v1 = v0 < 1l;
    return v1;
}
US7 US7_0() { // None
    US7 x;
    x.tag = 0;
    return x;
}
US7 US7_1(std::array<Card,3l> v0, std::array<Card,4l> v1) { // Some
    US7 x;
    x.tag = 1;
    x.v.case1.v0 = v0; x.v.case1.v1 = v1;
    return x;
}
inline bool while_method_11(int32_t v0){
    bool v1;
    v1 = v0 < 4l;
    return v1;
}
US8 US8_0() { // Eq
    US8 x;
    x.tag = 0;
    return x;
}
US8 US8_1() { // Gt
    US8 x;
    x.tag = 1;
    return x;
}
US8 US8_2() { // Lt
    US8 x;
    x.tag = 2;
    return x;
}
US9 US9_0() { // None
    US9 x;
    x.tag = 0;
    return x;
}
US9 US9_1(std::array<Card,2l> v0, std::array<Card,2l> v1) { // Some
    US9 x;
    x.tag = 1;
    x.v.case1.v0 = v0; x.v.case1.v1 = v1;
    return x;
}
US10 US10_0() { // None
    US10 x;
    x.tag = 0;
    return x;
}
US10 US10_1(std::array<Card,4l> v0, std::array<Card,3l> v1) { // Some
    US10 x;
    x.tag = 1;
    x.v.case1.v0 = v0; x.v.case1.v1 = v1;
    return x;
}
US11 US11_0() { // None
    US11 x;
    x.tag = 0;
    return x;
}
US11 US11_1(std::array<Card,5l> v0, int8_t v1) { // Some
    US11 x;
    x.tag = 1;
    x.v.case1.v0 = v0; x.v.case1.v1 = v1;
    return x;
}
Tuple5 score_9(std::array<Card,7l> v0){
    std::array<Card,7l> v1;
    int32_t v2;
    v2 = 0l;
    while (while_method_6(v2)){
        Card v4;
        v4 = v0[v2];
        v1[v2] = v4;
        v2++;
    }
    Fun0 v5;
    v5 = ClosureMethod0;
    std::sort(v1.begin(),v1.end(),v5);
    std::array<Card,5l> v6;
    int32_t v7;
    v7 = 0l;
    while (while_method_7(v7)){
        Card v9;
        v9 = v1[v7];
        v6[v7] = v9;
        v7++;
    }
    std::array<Card,2l> v10;
    std::array<Card,5l> v11;
    int32_t v12; int32_t v13; int32_t v14; uint8_t v15;
    Tuple6 tmp9 = Tuple6(0l, 0l, 0l, 12u);
    v12 = tmp9.v0; v13 = tmp9.v1; v14 = tmp9.v2; v15 = tmp9.v3;
    while (while_method_6(v12)){
        Card v17;
        v17 = v1[v12];
        bool v18;
        v18 = v14 < 2l;
        int32_t v27; int32_t v28; uint8_t v29;
        if (v18){
            uint8_t v19;
            v19 = v17.rank;
            bool v20;
            v20 = v15 == v19;
            int32_t v21;
            if (v20){
                v21 = v14;
            } else {
                v21 = 0l;
            }
            v10[v21] = v17;
            int32_t v22;
            v22 = v21 + 1l;
            uint8_t v23;
            v23 = v17.rank;
            v27 = v12; v28 = v22; v29 = v23;
        } else {
            break;
        }
        v13 = v27;
        v14 = v28;
        v15 = v29;
        v12++;
    }
    bool v30;
    v30 = v14 == 2l;
    US4 v40;
    if (v30){
        int32_t v31;
        v31 = 0l;
        while (while_method_7(v31)){
            int32_t v33;
            v33 = v13 + -1l;
            bool v34;
            v34 = v31 < v33;
            int32_t v35;
            if (v34){
                v35 = 0l;
            } else {
                v35 = 2l;
            }
            int32_t v36;
            v36 = v35 + v31;
            Card v37;
            v37 = v1[v36];
            v11[v31] = v37;
            v31++;
        }
        v40 = US4_1(v10, v11);
    } else {
        v40 = US4_0();
    }
    US5 v59;
    switch (v40.tag) {
        case 0: { // None
            v59 = US5_0();
            break;
        }
        default: { // Some
            std::array<Card,2l> v41 = v40.v.case1.v0; std::array<Card,5l> v42 = v40.v.case1.v1;
            std::array<Card,3l> v43;
            int32_t v44;
            v44 = 0l;
            while (while_method_8(v44)){
                Card v46;
                v46 = v42[v44];
                v43[v44] = v46;
                v44++;
            }
            std::array<Card,0l> v47;
            std::array<Card,5l> v48;
            int32_t v49;
            v49 = 0l;
            while (while_method_9(v49)){
                Card v51;
                v51 = v41[v49];
                v48[v49] = v51;
                v49++;
            }
            int32_t v52;
            v52 = 0l;
            while (while_method_8(v52)){
                Card v54;
                v54 = v43[v52];
                int32_t v55;
                v55 = 2l + v52;
                v48[v55] = v54;
                v52++;
            }
            v59 = US5_1(v48);
        }
    }
    std::array<Card,2l> v60;
    std::array<Card,5l> v61;
    int32_t v62; int32_t v63; int32_t v64; uint8_t v65;
    Tuple6 tmp10 = Tuple6(0l, 0l, 0l, 12u);
    v62 = tmp10.v0; v63 = tmp10.v1; v64 = tmp10.v2; v65 = tmp10.v3;
    while (while_method_6(v62)){
        Card v67;
        v67 = v1[v62];
        bool v68;
        v68 = v64 < 2l;
        int32_t v77; int32_t v78; uint8_t v79;
        if (v68){
            uint8_t v69;
            v69 = v67.rank;
            bool v70;
            v70 = v65 == v69;
            int32_t v71;
            if (v70){
                v71 = v64;
            } else {
                v71 = 0l;
            }
            v60[v71] = v67;
            int32_t v72;
            v72 = v71 + 1l;
            uint8_t v73;
            v73 = v67.rank;
            v77 = v62; v78 = v72; v79 = v73;
        } else {
            break;
        }
        v63 = v77;
        v64 = v78;
        v65 = v79;
        v62++;
    }
    bool v80;
    v80 = v64 == 2l;
    US4 v90;
    if (v80){
        int32_t v81;
        v81 = 0l;
        while (while_method_7(v81)){
            int32_t v83;
            v83 = v63 + -1l;
            bool v84;
            v84 = v81 < v83;
            int32_t v85;
            if (v84){
                v85 = 0l;
            } else {
                v85 = 2l;
            }
            int32_t v86;
            v86 = v85 + v81;
            Card v87;
            v87 = v1[v86];
            v61[v81] = v87;
            v81++;
        }
        v90 = US4_1(v60, v61);
    } else {
        v90 = US4_0();
    }
    US5 v148;
    switch (v90.tag) {
        case 0: { // None
            v148 = US5_0();
            break;
        }
        default: { // Some
            std::array<Card,2l> v91 = v90.v.case1.v0; std::array<Card,5l> v92 = v90.v.case1.v1;
            std::array<Card,2l> v93;
            std::array<Card,3l> v94;
            int32_t v95; int32_t v96; int32_t v97; uint8_t v98;
            Tuple6 tmp11 = Tuple6(0l, 0l, 0l, 12u);
            v95 = tmp11.v0; v96 = tmp11.v1; v97 = tmp11.v2; v98 = tmp11.v3;
            while (while_method_7(v95)){
                Card v100;
                v100 = v92[v95];
                bool v101;
                v101 = v97 < 2l;
                int32_t v110; int32_t v111; uint8_t v112;
                if (v101){
                    uint8_t v102;
                    v102 = v100.rank;
                    bool v103;
                    v103 = v98 == v102;
                    int32_t v104;
                    if (v103){
                        v104 = v97;
                    } else {
                        v104 = 0l;
                    }
                    v93[v104] = v100;
                    int32_t v105;
                    v105 = v104 + 1l;
                    uint8_t v106;
                    v106 = v100.rank;
                    v110 = v95; v111 = v105; v112 = v106;
                } else {
                    break;
                }
                v96 = v110;
                v97 = v111;
                v98 = v112;
                v95++;
            }
            bool v113;
            v113 = v97 == 2l;
            US6 v123;
            if (v113){
                int32_t v114;
                v114 = 0l;
                while (while_method_8(v114)){
                    int32_t v116;
                    v116 = v96 + -1l;
                    bool v117;
                    v117 = v114 < v116;
                    int32_t v118;
                    if (v117){
                        v118 = 0l;
                    } else {
                        v118 = 2l;
                    }
                    int32_t v119;
                    v119 = v118 + v114;
                    Card v120;
                    v120 = v92[v119];
                    v94[v114] = v120;
                    v114++;
                }
                v123 = US6_1(v93, v94);
            } else {
                v123 = US6_0();
            }
            switch (v123.tag) {
                case 0: { // None
                    v148 = US5_0();
                    break;
                }
                default: { // Some
                    std::array<Card,2l> v124 = v123.v.case1.v0; std::array<Card,3l> v125 = v123.v.case1.v1;
                    std::array<Card,1l> v126;
                    int32_t v127;
                    v127 = 0l;
                    while (while_method_10(v127)){
                        Card v129;
                        v129 = v125[v127];
                        v126[v127] = v129;
                        v127++;
                    }
                    std::array<Card,5l> v130;
                    int32_t v131;
                    v131 = 0l;
                    while (while_method_9(v131)){
                        Card v133;
                        v133 = v91[v131];
                        v130[v131] = v133;
                        v131++;
                    }
                    int32_t v134;
                    v134 = 0l;
                    while (while_method_9(v134)){
                        Card v136;
                        v136 = v124[v134];
                        int32_t v137;
                        v137 = 2l + v134;
                        v130[v137] = v136;
                        v134++;
                    }
                    int32_t v138;
                    v138 = 0l;
                    while (while_method_10(v138)){
                        Card v140;
                        v140 = v126[v138];
                        int32_t v141;
                        v141 = 4l + v138;
                        v130[v141] = v140;
                        v138++;
                    }
                    v148 = US5_1(v130);
                }
            }
        }
    }
    std::array<Card,3l> v149;
    std::array<Card,4l> v150;
    int32_t v151; int32_t v152; int32_t v153; uint8_t v154;
    Tuple6 tmp12 = Tuple6(0l, 0l, 0l, 12u);
    v151 = tmp12.v0; v152 = tmp12.v1; v153 = tmp12.v2; v154 = tmp12.v3;
    while (while_method_6(v151)){
        Card v156;
        v156 = v1[v151];
        bool v157;
        v157 = v153 < 3l;
        int32_t v166; int32_t v167; uint8_t v168;
        if (v157){
            uint8_t v158;
            v158 = v156.rank;
            bool v159;
            v159 = v154 == v158;
            int32_t v160;
            if (v159){
                v160 = v153;
            } else {
                v160 = 0l;
            }
            v149[v160] = v156;
            int32_t v161;
            v161 = v160 + 1l;
            uint8_t v162;
            v162 = v156.rank;
            v166 = v151; v167 = v161; v168 = v162;
        } else {
            break;
        }
        v152 = v166;
        v153 = v167;
        v154 = v168;
        v151++;
    }
    bool v169;
    v169 = v153 == 3l;
    US7 v179;
    if (v169){
        int32_t v170;
        v170 = 0l;
        while (while_method_11(v170)){
            int32_t v172;
            v172 = v152 + -2l;
            bool v173;
            v173 = v170 < v172;
            int32_t v174;
            if (v173){
                v174 = 0l;
            } else {
                v174 = 3l;
            }
            int32_t v175;
            v175 = v174 + v170;
            Card v176;
            v176 = v1[v175];
            v150[v170] = v176;
            v170++;
        }
        v179 = US7_1(v149, v150);
    } else {
        v179 = US7_0();
    }
    US5 v198;
    switch (v179.tag) {
        case 0: { // None
            v198 = US5_0();
            break;
        }
        default: { // Some
            std::array<Card,3l> v180 = v179.v.case1.v0; std::array<Card,4l> v181 = v179.v.case1.v1;
            std::array<Card,2l> v182;
            int32_t v183;
            v183 = 0l;
            while (while_method_9(v183)){
                Card v185;
                v185 = v181[v183];
                v182[v183] = v185;
                v183++;
            }
            std::array<Card,0l> v186;
            std::array<Card,5l> v187;
            int32_t v188;
            v188 = 0l;
            while (while_method_8(v188)){
                Card v190;
                v190 = v180[v188];
                v187[v188] = v190;
                v188++;
            }
            int32_t v191;
            v191 = 0l;
            while (while_method_9(v191)){
                Card v193;
                v193 = v182[v191];
                int32_t v194;
                v194 = 3l + v191;
                v187[v194] = v193;
                v191++;
            }
            v198 = US5_1(v187);
        }
    }
    std::array<Card,5l> v199;
    int32_t v200; int32_t v201; uint8_t v202;
    Tuple7 tmp13 = Tuple7(0l, 0l, 12u);
    v200 = tmp13.v0; v201 = tmp13.v1; v202 = tmp13.v2;
    while (while_method_6(v200)){
        Card v204;
        v204 = v1[v200];
        bool v205;
        v205 = v201 < 5l;
        int32_t v220; uint8_t v221;
        if (v205){
            uint8_t v206;
            v206 = v204.rank;
            uint8_t v207;
            v207 = v206 - 1u;
            bool v208;
            v208 = v202 == v207;
            bool v209;
            v209 = v208 != true;
            if (v209){
                uint8_t v210;
                v210 = v204.rank;
                bool v211;
                v211 = v202 == v210;
                int32_t v212;
                if (v211){
                    v212 = v201;
                } else {
                    v212 = 0l;
                }
                v199[v212] = v204;
                int32_t v213;
                v213 = v212 + 1l;
                uint8_t v214;
                v214 = v204.rank;
                uint8_t v215;
                v215 = v214 - 1u;
                v220 = v213; v221 = v215;
            } else {
                v220 = v201; v221 = v202;
            }
        } else {
            break;
        }
        v201 = v220;
        v202 = v221;
        v200++;
    }
    bool v222;
    v222 = v201 == 4l;
    bool v230;
    if (v222){
        uint8_t v223;
        v223 = v202 + 1u;
        bool v224;
        v224 = v223 == 0u;
        if (v224){
            Card v225;
            v225 = v1[0l];
            uint8_t v226;
            v226 = v225.rank;
            bool v227;
            v227 = v226 == 12u;
            if (v227){
                v199[4l] = v225;
                v230 = true;
            } else {
                v230 = false;
            }
        } else {
            v230 = false;
        }
    } else {
        v230 = false;
    }
    US5 v236;
    if (v230){
        v236 = US5_1(v199);
    } else {
        bool v232;
        v232 = v201 == 5l;
        if (v232){
            v236 = US5_1(v199);
        } else {
            v236 = US5_0();
        }
    }
    std::array<Card,5l> v237;
    int32_t v238; int32_t v239;
    Tuple8 tmp14 = Tuple8(0l, 0l);
    v238 = tmp14.v0; v239 = tmp14.v1;
    while (while_method_6(v238)){
        Card v241;
        v241 = v1[v238];
        uint8_t v242;
        v242 = v241.suit;
        bool v243;
        v243 = v242 == 3u;
        bool v245;
        if (v243){
            bool v244;
            v244 = v239 < 5l;
            v245 = v244;
        } else {
            v245 = false;
        }
        int32_t v247;
        if (v245){
            v237[v239] = v241;
            int32_t v246;
            v246 = v239 + 1l;
            v247 = v246;
        } else {
            v247 = v239;
        }
        v239 = v247;
        v238++;
    }
    bool v248;
    v248 = v239 == 5l;
    US5 v251;
    if (v248){
        v251 = US5_1(v237);
    } else {
        v251 = US5_0();
    }
    std::array<Card,5l> v252;
    int32_t v253; int32_t v254;
    Tuple8 tmp15 = Tuple8(0l, 0l);
    v253 = tmp15.v0; v254 = tmp15.v1;
    while (while_method_6(v253)){
        Card v256;
        v256 = v1[v253];
        uint8_t v257;
        v257 = v256.suit;
        bool v258;
        v258 = v257 == 2u;
        bool v260;
        if (v258){
            bool v259;
            v259 = v254 < 5l;
            v260 = v259;
        } else {
            v260 = false;
        }
        int32_t v262;
        if (v260){
            v252[v254] = v256;
            int32_t v261;
            v261 = v254 + 1l;
            v262 = v261;
        } else {
            v262 = v254;
        }
        v254 = v262;
        v253++;
    }
    bool v263;
    v263 = v254 == 5l;
    US5 v266;
    if (v263){
        v266 = US5_1(v252);
    } else {
        v266 = US5_0();
    }
    std::array<Card,5l> v267;
    int32_t v268; int32_t v269;
    Tuple8 tmp16 = Tuple8(0l, 0l);
    v268 = tmp16.v0; v269 = tmp16.v1;
    while (while_method_6(v268)){
        Card v271;
        v271 = v1[v268];
        uint8_t v272;
        v272 = v271.suit;
        bool v273;
        v273 = v272 == 1u;
        bool v275;
        if (v273){
            bool v274;
            v274 = v269 < 5l;
            v275 = v274;
        } else {
            v275 = false;
        }
        int32_t v277;
        if (v275){
            v267[v269] = v271;
            int32_t v276;
            v276 = v269 + 1l;
            v277 = v276;
        } else {
            v277 = v269;
        }
        v269 = v277;
        v268++;
    }
    bool v278;
    v278 = v269 == 5l;
    US5 v281;
    if (v278){
        v281 = US5_1(v267);
    } else {
        v281 = US5_0();
    }
    std::array<Card,5l> v282;
    int32_t v283; int32_t v284;
    Tuple8 tmp17 = Tuple8(0l, 0l);
    v283 = tmp17.v0; v284 = tmp17.v1;
    while (while_method_6(v283)){
        Card v286;
        v286 = v1[v283];
        uint8_t v287;
        v287 = v286.suit;
        bool v288;
        v288 = v287 == 0u;
        bool v290;
        if (v288){
            bool v289;
            v289 = v284 < 5l;
            v290 = v289;
        } else {
            v290 = false;
        }
        int32_t v292;
        if (v290){
            v282[v284] = v286;
            int32_t v291;
            v291 = v284 + 1l;
            v292 = v291;
        } else {
            v292 = v284;
        }
        v284 = v292;
        v283++;
    }
    bool v293;
    v293 = v284 == 5l;
    US5 v296;
    if (v293){
        v296 = US5_1(v282);
    } else {
        v296 = US5_0();
    }
    US5 v322;
    switch (v296.tag) {
        case 0: { // None
            v322 = v281;
            break;
        }
        default: { // Some
            std::array<Card,5l> v297 = v296.v.case1.v0;
            switch (v281.tag) {
                case 0: { // None
                    v322 = v296;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v298 = v281.v.case1.v0;
                    US8 v299;
                    v299 = US8_0();
                    int32_t v300; US8 v301;
                    Tuple9 tmp18 = Tuple9(0l, v299);
                    v300 = tmp18.v0; v301 = tmp18.v1;
                    while (while_method_7(v300)){
                        Card v303;
                        v303 = v297[v300];
                        Card v304;
                        v304 = v298[v300];
                        US8 v315;
                        switch (v301.tag) {
                            case 0: { // Eq
                                uint8_t v305;
                                v305 = v303.rank;
                                uint8_t v306;
                                v306 = v304.rank;
                                bool v307;
                                v307 = v305 < v306;
                                if (v307){
                                    v315 = US8_2();
                                } else {
                                    bool v309;
                                    v309 = v305 > v306;
                                    if (v309){
                                        v315 = US8_1();
                                    } else {
                                        v315 = US8_0();
                                    }
                                }
                                break;
                            }
                            default: {
                                break;
                            }
                        }
                        v301 = v315;
                        v300++;
                    }
                    bool v316;
                    switch (v301.tag) {
                        case 1: { // Gt
                            v316 = true;
                            break;
                        }
                        default: {
                            v316 = false;
                        }
                    }
                    std::array<Card,5l> v317;
                    if (v316){
                        v317 = v297;
                    } else {
                        v317 = v298;
                    }
                    v322 = US5_1(v317);
                }
            }
        }
    }
    US5 v348;
    switch (v322.tag) {
        case 0: { // None
            v348 = v266;
            break;
        }
        default: { // Some
            std::array<Card,5l> v323 = v322.v.case1.v0;
            switch (v266.tag) {
                case 0: { // None
                    v348 = v322;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v324 = v266.v.case1.v0;
                    US8 v325;
                    v325 = US8_0();
                    int32_t v326; US8 v327;
                    Tuple9 tmp19 = Tuple9(0l, v325);
                    v326 = tmp19.v0; v327 = tmp19.v1;
                    while (while_method_7(v326)){
                        Card v329;
                        v329 = v323[v326];
                        Card v330;
                        v330 = v324[v326];
                        US8 v341;
                        switch (v327.tag) {
                            case 0: { // Eq
                                uint8_t v331;
                                v331 = v329.rank;
                                uint8_t v332;
                                v332 = v330.rank;
                                bool v333;
                                v333 = v331 < v332;
                                if (v333){
                                    v341 = US8_2();
                                } else {
                                    bool v335;
                                    v335 = v331 > v332;
                                    if (v335){
                                        v341 = US8_1();
                                    } else {
                                        v341 = US8_0();
                                    }
                                }
                                break;
                            }
                            default: {
                                break;
                            }
                        }
                        v327 = v341;
                        v326++;
                    }
                    bool v342;
                    switch (v327.tag) {
                        case 1: { // Gt
                            v342 = true;
                            break;
                        }
                        default: {
                            v342 = false;
                        }
                    }
                    std::array<Card,5l> v343;
                    if (v342){
                        v343 = v323;
                    } else {
                        v343 = v324;
                    }
                    v348 = US5_1(v343);
                }
            }
        }
    }
    US5 v374;
    switch (v348.tag) {
        case 0: { // None
            v374 = v251;
            break;
        }
        default: { // Some
            std::array<Card,5l> v349 = v348.v.case1.v0;
            switch (v251.tag) {
                case 0: { // None
                    v374 = v348;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v350 = v251.v.case1.v0;
                    US8 v351;
                    v351 = US8_0();
                    int32_t v352; US8 v353;
                    Tuple9 tmp20 = Tuple9(0l, v351);
                    v352 = tmp20.v0; v353 = tmp20.v1;
                    while (while_method_7(v352)){
                        Card v355;
                        v355 = v349[v352];
                        Card v356;
                        v356 = v350[v352];
                        US8 v367;
                        switch (v353.tag) {
                            case 0: { // Eq
                                uint8_t v357;
                                v357 = v355.rank;
                                uint8_t v358;
                                v358 = v356.rank;
                                bool v359;
                                v359 = v357 < v358;
                                if (v359){
                                    v367 = US8_2();
                                } else {
                                    bool v361;
                                    v361 = v357 > v358;
                                    if (v361){
                                        v367 = US8_1();
                                    } else {
                                        v367 = US8_0();
                                    }
                                }
                                break;
                            }
                            default: {
                                break;
                            }
                        }
                        v353 = v367;
                        v352++;
                    }
                    bool v368;
                    switch (v353.tag) {
                        case 1: { // Gt
                            v368 = true;
                            break;
                        }
                        default: {
                            v368 = false;
                        }
                    }
                    std::array<Card,5l> v369;
                    if (v368){
                        v369 = v349;
                    } else {
                        v369 = v350;
                    }
                    v374 = US5_1(v369);
                }
            }
        }
    }
    std::array<Card,3l> v375;
    std::array<Card,4l> v376;
    int32_t v377; int32_t v378; int32_t v379; uint8_t v380;
    Tuple6 tmp21 = Tuple6(0l, 0l, 0l, 12u);
    v377 = tmp21.v0; v378 = tmp21.v1; v379 = tmp21.v2; v380 = tmp21.v3;
    while (while_method_6(v377)){
        Card v382;
        v382 = v1[v377];
        bool v383;
        v383 = v379 < 3l;
        int32_t v392; int32_t v393; uint8_t v394;
        if (v383){
            uint8_t v384;
            v384 = v382.rank;
            bool v385;
            v385 = v380 == v384;
            int32_t v386;
            if (v385){
                v386 = v379;
            } else {
                v386 = 0l;
            }
            v375[v386] = v382;
            int32_t v387;
            v387 = v386 + 1l;
            uint8_t v388;
            v388 = v382.rank;
            v392 = v377; v393 = v387; v394 = v388;
        } else {
            break;
        }
        v378 = v392;
        v379 = v393;
        v380 = v394;
        v377++;
    }
    bool v395;
    v395 = v379 == 3l;
    US7 v405;
    if (v395){
        int32_t v396;
        v396 = 0l;
        while (while_method_11(v396)){
            int32_t v398;
            v398 = v378 + -2l;
            bool v399;
            v399 = v396 < v398;
            int32_t v400;
            if (v399){
                v400 = 0l;
            } else {
                v400 = 3l;
            }
            int32_t v401;
            v401 = v400 + v396;
            Card v402;
            v402 = v1[v401];
            v376[v396] = v402;
            v396++;
        }
        v405 = US7_1(v375, v376);
    } else {
        v405 = US7_0();
    }
    US5 v456;
    switch (v405.tag) {
        case 0: { // None
            v456 = US5_0();
            break;
        }
        default: { // Some
            std::array<Card,3l> v406 = v405.v.case1.v0; std::array<Card,4l> v407 = v405.v.case1.v1;
            std::array<Card,2l> v408;
            std::array<Card,2l> v409;
            int32_t v410; int32_t v411; int32_t v412; uint8_t v413;
            Tuple6 tmp22 = Tuple6(0l, 0l, 0l, 12u);
            v410 = tmp22.v0; v411 = tmp22.v1; v412 = tmp22.v2; v413 = tmp22.v3;
            while (while_method_11(v410)){
                Card v415;
                v415 = v407[v410];
                bool v416;
                v416 = v412 < 2l;
                int32_t v425; int32_t v426; uint8_t v427;
                if (v416){
                    uint8_t v417;
                    v417 = v415.rank;
                    bool v418;
                    v418 = v413 == v417;
                    int32_t v419;
                    if (v418){
                        v419 = v412;
                    } else {
                        v419 = 0l;
                    }
                    v408[v419] = v415;
                    int32_t v420;
                    v420 = v419 + 1l;
                    uint8_t v421;
                    v421 = v415.rank;
                    v425 = v410; v426 = v420; v427 = v421;
                } else {
                    break;
                }
                v411 = v425;
                v412 = v426;
                v413 = v427;
                v410++;
            }
            bool v428;
            v428 = v412 == 2l;
            US9 v438;
            if (v428){
                int32_t v429;
                v429 = 0l;
                while (while_method_9(v429)){
                    int32_t v431;
                    v431 = v411 + -1l;
                    bool v432;
                    v432 = v429 < v431;
                    int32_t v433;
                    if (v432){
                        v433 = 0l;
                    } else {
                        v433 = 2l;
                    }
                    int32_t v434;
                    v434 = v433 + v429;
                    Card v435;
                    v435 = v407[v434];
                    v409[v429] = v435;
                    v429++;
                }
                v438 = US9_1(v408, v409);
            } else {
                v438 = US9_0();
            }
            switch (v438.tag) {
                case 0: { // None
                    v456 = US5_0();
                    break;
                }
                default: { // Some
                    std::array<Card,2l> v439 = v438.v.case1.v0; std::array<Card,2l> v440 = v438.v.case1.v1;
                    std::array<Card,0l> v441;
                    std::array<Card,5l> v442;
                    int32_t v443;
                    v443 = 0l;
                    while (while_method_8(v443)){
                        Card v445;
                        v445 = v406[v443];
                        v442[v443] = v445;
                        v443++;
                    }
                    int32_t v446;
                    v446 = 0l;
                    while (while_method_9(v446)){
                        Card v448;
                        v448 = v439[v446];
                        int32_t v449;
                        v449 = 3l + v446;
                        v442[v449] = v448;
                        v446++;
                    }
                    v456 = US5_1(v442);
                }
            }
        }
    }
    std::array<Card,4l> v457;
    std::array<Card,3l> v458;
    int32_t v459; int32_t v460; int32_t v461; uint8_t v462;
    Tuple6 tmp23 = Tuple6(0l, 0l, 0l, 12u);
    v459 = tmp23.v0; v460 = tmp23.v1; v461 = tmp23.v2; v462 = tmp23.v3;
    while (while_method_6(v459)){
        Card v464;
        v464 = v1[v459];
        bool v465;
        v465 = v461 < 4l;
        int32_t v474; int32_t v475; uint8_t v476;
        if (v465){
            uint8_t v466;
            v466 = v464.rank;
            bool v467;
            v467 = v462 == v466;
            int32_t v468;
            if (v467){
                v468 = v461;
            } else {
                v468 = 0l;
            }
            v457[v468] = v464;
            int32_t v469;
            v469 = v468 + 1l;
            uint8_t v470;
            v470 = v464.rank;
            v474 = v459; v475 = v469; v476 = v470;
        } else {
            break;
        }
        v460 = v474;
        v461 = v475;
        v462 = v476;
        v459++;
    }
    bool v477;
    v477 = v461 == 4l;
    US10 v487;
    if (v477){
        int32_t v478;
        v478 = 0l;
        while (while_method_8(v478)){
            int32_t v480;
            v480 = v460 + -3l;
            bool v481;
            v481 = v478 < v480;
            int32_t v482;
            if (v481){
                v482 = 0l;
            } else {
                v482 = 4l;
            }
            int32_t v483;
            v483 = v482 + v478;
            Card v484;
            v484 = v1[v483];
            v458[v478] = v484;
            v478++;
        }
        v487 = US10_1(v457, v458);
    } else {
        v487 = US10_0();
    }
    US5 v506;
    switch (v487.tag) {
        case 0: { // None
            v506 = US5_0();
            break;
        }
        default: { // Some
            std::array<Card,4l> v488 = v487.v.case1.v0; std::array<Card,3l> v489 = v487.v.case1.v1;
            std::array<Card,1l> v490;
            int32_t v491;
            v491 = 0l;
            while (while_method_10(v491)){
                Card v493;
                v493 = v489[v491];
                v490[v491] = v493;
                v491++;
            }
            std::array<Card,0l> v494;
            std::array<Card,5l> v495;
            int32_t v496;
            v496 = 0l;
            while (while_method_11(v496)){
                Card v498;
                v498 = v488[v496];
                v495[v496] = v498;
                v496++;
            }
            int32_t v499;
            v499 = 0l;
            while (while_method_10(v499)){
                Card v501;
                v501 = v490[v499];
                int32_t v502;
                v502 = 4l + v499;
                v495[v502] = v501;
                v499++;
            }
            v506 = US5_1(v495);
        }
    }
    std::array<Card,5l> v507;
    int32_t v508; int32_t v509; uint8_t v510;
    Tuple7 tmp24 = Tuple7(0l, 0l, 12u);
    v508 = tmp24.v0; v509 = tmp24.v1; v510 = tmp24.v2;
    while (while_method_6(v508)){
        Card v512;
        v512 = v1[v508];
        bool v513;
        v513 = v509 < 5l;
        int32_t v526; uint8_t v527;
        if (v513){
            uint8_t v514;
            v514 = v512.suit;
            bool v515;
            v515 = 3u == v514;
            if (v515){
                uint8_t v516;
                v516 = v512.rank;
                bool v517;
                v517 = v510 == v516;
                int32_t v518;
                if (v517){
                    v518 = v509;
                } else {
                    v518 = 0l;
                }
                v507[v518] = v512;
                int32_t v519;
                v519 = v518 + 1l;
                uint8_t v520;
                v520 = v512.rank;
                uint8_t v521;
                v521 = v520 - 1u;
                v526 = v519; v527 = v521;
            } else {
                v526 = v509; v527 = v510;
            }
        } else {
            break;
        }
        v509 = v526;
        v510 = v527;
        v508++;
    }
    bool v528;
    v528 = v509 == 4l;
    bool v563;
    if (v528){
        uint8_t v529;
        v529 = v510 + 1u;
        bool v530;
        v530 = v529 == 0u;
        if (v530){
            Card v531;
            v531 = v1[0l];
            uint8_t v532;
            v532 = v531.suit;
            bool v533;
            v533 = 3u == v532;
            bool v537;
            if (v533){
                uint8_t v534;
                v534 = v531.rank;
                bool v535;
                v535 = v534 == 12u;
                if (v535){
                    v507[4l] = v531;
                    v537 = true;
                } else {
                    v537 = false;
                }
            } else {
                v537 = false;
            }
            if (v537){
                v563 = true;
            } else {
                Card v538;
                v538 = v1[1l];
                uint8_t v539;
                v539 = v538.suit;
                bool v540;
                v540 = 3u == v539;
                bool v544;
                if (v540){
                    uint8_t v541;
                    v541 = v538.rank;
                    bool v542;
                    v542 = v541 == 12u;
                    if (v542){
                        v507[4l] = v538;
                        v544 = true;
                    } else {
                        v544 = false;
                    }
                } else {
                    v544 = false;
                }
                if (v544){
                    v563 = true;
                } else {
                    Card v545;
                    v545 = v1[2l];
                    uint8_t v546;
                    v546 = v545.suit;
                    bool v547;
                    v547 = 3u == v546;
                    bool v551;
                    if (v547){
                        uint8_t v548;
                        v548 = v545.rank;
                        bool v549;
                        v549 = v548 == 12u;
                        if (v549){
                            v507[4l] = v545;
                            v551 = true;
                        } else {
                            v551 = false;
                        }
                    } else {
                        v551 = false;
                    }
                    if (v551){
                        v563 = true;
                    } else {
                        Card v552;
                        v552 = v1[3l];
                        uint8_t v553;
                        v553 = v552.suit;
                        bool v554;
                        v554 = 3u == v553;
                        if (v554){
                            uint8_t v555;
                            v555 = v552.rank;
                            bool v556;
                            v556 = v555 == 12u;
                            if (v556){
                                v507[4l] = v552;
                                v563 = true;
                            } else {
                                v563 = false;
                            }
                        } else {
                            v563 = false;
                        }
                    }
                }
            }
        } else {
            v563 = false;
        }
    } else {
        v563 = false;
    }
    US5 v569;
    if (v563){
        v569 = US5_1(v507);
    } else {
        bool v565;
        v565 = v509 == 5l;
        if (v565){
            v569 = US5_1(v507);
        } else {
            v569 = US5_0();
        }
    }
    std::array<Card,5l> v570;
    int32_t v571; int32_t v572; uint8_t v573;
    Tuple7 tmp25 = Tuple7(0l, 0l, 12u);
    v571 = tmp25.v0; v572 = tmp25.v1; v573 = tmp25.v2;
    while (while_method_6(v571)){
        Card v575;
        v575 = v1[v571];
        bool v576;
        v576 = v572 < 5l;
        int32_t v589; uint8_t v590;
        if (v576){
            uint8_t v577;
            v577 = v575.suit;
            bool v578;
            v578 = 2u == v577;
            if (v578){
                uint8_t v579;
                v579 = v575.rank;
                bool v580;
                v580 = v573 == v579;
                int32_t v581;
                if (v580){
                    v581 = v572;
                } else {
                    v581 = 0l;
                }
                v570[v581] = v575;
                int32_t v582;
                v582 = v581 + 1l;
                uint8_t v583;
                v583 = v575.rank;
                uint8_t v584;
                v584 = v583 - 1u;
                v589 = v582; v590 = v584;
            } else {
                v589 = v572; v590 = v573;
            }
        } else {
            break;
        }
        v572 = v589;
        v573 = v590;
        v571++;
    }
    bool v591;
    v591 = v572 == 4l;
    bool v626;
    if (v591){
        uint8_t v592;
        v592 = v573 + 1u;
        bool v593;
        v593 = v592 == 0u;
        if (v593){
            Card v594;
            v594 = v1[0l];
            uint8_t v595;
            v595 = v594.suit;
            bool v596;
            v596 = 2u == v595;
            bool v600;
            if (v596){
                uint8_t v597;
                v597 = v594.rank;
                bool v598;
                v598 = v597 == 12u;
                if (v598){
                    v570[4l] = v594;
                    v600 = true;
                } else {
                    v600 = false;
                }
            } else {
                v600 = false;
            }
            if (v600){
                v626 = true;
            } else {
                Card v601;
                v601 = v1[1l];
                uint8_t v602;
                v602 = v601.suit;
                bool v603;
                v603 = 2u == v602;
                bool v607;
                if (v603){
                    uint8_t v604;
                    v604 = v601.rank;
                    bool v605;
                    v605 = v604 == 12u;
                    if (v605){
                        v570[4l] = v601;
                        v607 = true;
                    } else {
                        v607 = false;
                    }
                } else {
                    v607 = false;
                }
                if (v607){
                    v626 = true;
                } else {
                    Card v608;
                    v608 = v1[2l];
                    uint8_t v609;
                    v609 = v608.suit;
                    bool v610;
                    v610 = 2u == v609;
                    bool v614;
                    if (v610){
                        uint8_t v611;
                        v611 = v608.rank;
                        bool v612;
                        v612 = v611 == 12u;
                        if (v612){
                            v570[4l] = v608;
                            v614 = true;
                        } else {
                            v614 = false;
                        }
                    } else {
                        v614 = false;
                    }
                    if (v614){
                        v626 = true;
                    } else {
                        Card v615;
                        v615 = v1[3l];
                        uint8_t v616;
                        v616 = v615.suit;
                        bool v617;
                        v617 = 2u == v616;
                        if (v617){
                            uint8_t v618;
                            v618 = v615.rank;
                            bool v619;
                            v619 = v618 == 12u;
                            if (v619){
                                v570[4l] = v615;
                                v626 = true;
                            } else {
                                v626 = false;
                            }
                        } else {
                            v626 = false;
                        }
                    }
                }
            }
        } else {
            v626 = false;
        }
    } else {
        v626 = false;
    }
    US5 v632;
    if (v626){
        v632 = US5_1(v570);
    } else {
        bool v628;
        v628 = v572 == 5l;
        if (v628){
            v632 = US5_1(v570);
        } else {
            v632 = US5_0();
        }
    }
    std::array<Card,5l> v633;
    int32_t v634; int32_t v635; uint8_t v636;
    Tuple7 tmp26 = Tuple7(0l, 0l, 12u);
    v634 = tmp26.v0; v635 = tmp26.v1; v636 = tmp26.v2;
    while (while_method_6(v634)){
        Card v638;
        v638 = v1[v634];
        bool v639;
        v639 = v635 < 5l;
        int32_t v652; uint8_t v653;
        if (v639){
            uint8_t v640;
            v640 = v638.suit;
            bool v641;
            v641 = 1u == v640;
            if (v641){
                uint8_t v642;
                v642 = v638.rank;
                bool v643;
                v643 = v636 == v642;
                int32_t v644;
                if (v643){
                    v644 = v635;
                } else {
                    v644 = 0l;
                }
                v633[v644] = v638;
                int32_t v645;
                v645 = v644 + 1l;
                uint8_t v646;
                v646 = v638.rank;
                uint8_t v647;
                v647 = v646 - 1u;
                v652 = v645; v653 = v647;
            } else {
                v652 = v635; v653 = v636;
            }
        } else {
            break;
        }
        v635 = v652;
        v636 = v653;
        v634++;
    }
    bool v654;
    v654 = v635 == 4l;
    bool v689;
    if (v654){
        uint8_t v655;
        v655 = v636 + 1u;
        bool v656;
        v656 = v655 == 0u;
        if (v656){
            Card v657;
            v657 = v1[0l];
            uint8_t v658;
            v658 = v657.suit;
            bool v659;
            v659 = 1u == v658;
            bool v663;
            if (v659){
                uint8_t v660;
                v660 = v657.rank;
                bool v661;
                v661 = v660 == 12u;
                if (v661){
                    v633[4l] = v657;
                    v663 = true;
                } else {
                    v663 = false;
                }
            } else {
                v663 = false;
            }
            if (v663){
                v689 = true;
            } else {
                Card v664;
                v664 = v1[1l];
                uint8_t v665;
                v665 = v664.suit;
                bool v666;
                v666 = 1u == v665;
                bool v670;
                if (v666){
                    uint8_t v667;
                    v667 = v664.rank;
                    bool v668;
                    v668 = v667 == 12u;
                    if (v668){
                        v633[4l] = v664;
                        v670 = true;
                    } else {
                        v670 = false;
                    }
                } else {
                    v670 = false;
                }
                if (v670){
                    v689 = true;
                } else {
                    Card v671;
                    v671 = v1[2l];
                    uint8_t v672;
                    v672 = v671.suit;
                    bool v673;
                    v673 = 1u == v672;
                    bool v677;
                    if (v673){
                        uint8_t v674;
                        v674 = v671.rank;
                        bool v675;
                        v675 = v674 == 12u;
                        if (v675){
                            v633[4l] = v671;
                            v677 = true;
                        } else {
                            v677 = false;
                        }
                    } else {
                        v677 = false;
                    }
                    if (v677){
                        v689 = true;
                    } else {
                        Card v678;
                        v678 = v1[3l];
                        uint8_t v679;
                        v679 = v678.suit;
                        bool v680;
                        v680 = 1u == v679;
                        if (v680){
                            uint8_t v681;
                            v681 = v678.rank;
                            bool v682;
                            v682 = v681 == 12u;
                            if (v682){
                                v633[4l] = v678;
                                v689 = true;
                            } else {
                                v689 = false;
                            }
                        } else {
                            v689 = false;
                        }
                    }
                }
            }
        } else {
            v689 = false;
        }
    } else {
        v689 = false;
    }
    US5 v695;
    if (v689){
        v695 = US5_1(v633);
    } else {
        bool v691;
        v691 = v635 == 5l;
        if (v691){
            v695 = US5_1(v633);
        } else {
            v695 = US5_0();
        }
    }
    std::array<Card,5l> v696;
    int32_t v697; int32_t v698; uint8_t v699;
    Tuple7 tmp27 = Tuple7(0l, 0l, 12u);
    v697 = tmp27.v0; v698 = tmp27.v1; v699 = tmp27.v2;
    while (while_method_6(v697)){
        Card v701;
        v701 = v1[v697];
        bool v702;
        v702 = v698 < 5l;
        int32_t v715; uint8_t v716;
        if (v702){
            uint8_t v703;
            v703 = v701.suit;
            bool v704;
            v704 = 0u == v703;
            if (v704){
                uint8_t v705;
                v705 = v701.rank;
                bool v706;
                v706 = v699 == v705;
                int32_t v707;
                if (v706){
                    v707 = v698;
                } else {
                    v707 = 0l;
                }
                v696[v707] = v701;
                int32_t v708;
                v708 = v707 + 1l;
                uint8_t v709;
                v709 = v701.rank;
                uint8_t v710;
                v710 = v709 - 1u;
                v715 = v708; v716 = v710;
            } else {
                v715 = v698; v716 = v699;
            }
        } else {
            break;
        }
        v698 = v715;
        v699 = v716;
        v697++;
    }
    bool v717;
    v717 = v698 == 4l;
    bool v752;
    if (v717){
        uint8_t v718;
        v718 = v699 + 1u;
        bool v719;
        v719 = v718 == 0u;
        if (v719){
            Card v720;
            v720 = v1[0l];
            uint8_t v721;
            v721 = v720.suit;
            bool v722;
            v722 = 0u == v721;
            bool v726;
            if (v722){
                uint8_t v723;
                v723 = v720.rank;
                bool v724;
                v724 = v723 == 12u;
                if (v724){
                    v696[4l] = v720;
                    v726 = true;
                } else {
                    v726 = false;
                }
            } else {
                v726 = false;
            }
            if (v726){
                v752 = true;
            } else {
                Card v727;
                v727 = v1[1l];
                uint8_t v728;
                v728 = v727.suit;
                bool v729;
                v729 = 0u == v728;
                bool v733;
                if (v729){
                    uint8_t v730;
                    v730 = v727.rank;
                    bool v731;
                    v731 = v730 == 12u;
                    if (v731){
                        v696[4l] = v727;
                        v733 = true;
                    } else {
                        v733 = false;
                    }
                } else {
                    v733 = false;
                }
                if (v733){
                    v752 = true;
                } else {
                    Card v734;
                    v734 = v1[2l];
                    uint8_t v735;
                    v735 = v734.suit;
                    bool v736;
                    v736 = 0u == v735;
                    bool v740;
                    if (v736){
                        uint8_t v737;
                        v737 = v734.rank;
                        bool v738;
                        v738 = v737 == 12u;
                        if (v738){
                            v696[4l] = v734;
                            v740 = true;
                        } else {
                            v740 = false;
                        }
                    } else {
                        v740 = false;
                    }
                    if (v740){
                        v752 = true;
                    } else {
                        Card v741;
                        v741 = v1[3l];
                        uint8_t v742;
                        v742 = v741.suit;
                        bool v743;
                        v743 = 0u == v742;
                        if (v743){
                            uint8_t v744;
                            v744 = v741.rank;
                            bool v745;
                            v745 = v744 == 12u;
                            if (v745){
                                v696[4l] = v741;
                                v752 = true;
                            } else {
                                v752 = false;
                            }
                        } else {
                            v752 = false;
                        }
                    }
                }
            }
        } else {
            v752 = false;
        }
    } else {
        v752 = false;
    }
    US5 v758;
    if (v752){
        v758 = US5_1(v696);
    } else {
        bool v754;
        v754 = v698 == 5l;
        if (v754){
            v758 = US5_1(v696);
        } else {
            v758 = US5_0();
        }
    }
    US5 v784;
    switch (v758.tag) {
        case 0: { // None
            v784 = v695;
            break;
        }
        default: { // Some
            std::array<Card,5l> v759 = v758.v.case1.v0;
            switch (v695.tag) {
                case 0: { // None
                    v784 = v758;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v760 = v695.v.case1.v0;
                    US8 v761;
                    v761 = US8_0();
                    int32_t v762; US8 v763;
                    Tuple9 tmp28 = Tuple9(0l, v761);
                    v762 = tmp28.v0; v763 = tmp28.v1;
                    while (while_method_7(v762)){
                        Card v765;
                        v765 = v759[v762];
                        Card v766;
                        v766 = v760[v762];
                        US8 v777;
                        switch (v763.tag) {
                            case 0: { // Eq
                                uint8_t v767;
                                v767 = v765.rank;
                                uint8_t v768;
                                v768 = v766.rank;
                                bool v769;
                                v769 = v767 < v768;
                                if (v769){
                                    v777 = US8_2();
                                } else {
                                    bool v771;
                                    v771 = v767 > v768;
                                    if (v771){
                                        v777 = US8_1();
                                    } else {
                                        v777 = US8_0();
                                    }
                                }
                                break;
                            }
                            default: {
                                break;
                            }
                        }
                        v763 = v777;
                        v762++;
                    }
                    bool v778;
                    switch (v763.tag) {
                        case 1: { // Gt
                            v778 = true;
                            break;
                        }
                        default: {
                            v778 = false;
                        }
                    }
                    std::array<Card,5l> v779;
                    if (v778){
                        v779 = v759;
                    } else {
                        v779 = v760;
                    }
                    v784 = US5_1(v779);
                }
            }
        }
    }
    US5 v810;
    switch (v784.tag) {
        case 0: { // None
            v810 = v632;
            break;
        }
        default: { // Some
            std::array<Card,5l> v785 = v784.v.case1.v0;
            switch (v632.tag) {
                case 0: { // None
                    v810 = v784;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v786 = v632.v.case1.v0;
                    US8 v787;
                    v787 = US8_0();
                    int32_t v788; US8 v789;
                    Tuple9 tmp29 = Tuple9(0l, v787);
                    v788 = tmp29.v0; v789 = tmp29.v1;
                    while (while_method_7(v788)){
                        Card v791;
                        v791 = v785[v788];
                        Card v792;
                        v792 = v786[v788];
                        US8 v803;
                        switch (v789.tag) {
                            case 0: { // Eq
                                uint8_t v793;
                                v793 = v791.rank;
                                uint8_t v794;
                                v794 = v792.rank;
                                bool v795;
                                v795 = v793 < v794;
                                if (v795){
                                    v803 = US8_2();
                                } else {
                                    bool v797;
                                    v797 = v793 > v794;
                                    if (v797){
                                        v803 = US8_1();
                                    } else {
                                        v803 = US8_0();
                                    }
                                }
                                break;
                            }
                            default: {
                                break;
                            }
                        }
                        v789 = v803;
                        v788++;
                    }
                    bool v804;
                    switch (v789.tag) {
                        case 1: { // Gt
                            v804 = true;
                            break;
                        }
                        default: {
                            v804 = false;
                        }
                    }
                    std::array<Card,5l> v805;
                    if (v804){
                        v805 = v785;
                    } else {
                        v805 = v786;
                    }
                    v810 = US5_1(v805);
                }
            }
        }
    }
    US5 v836;
    switch (v810.tag) {
        case 0: { // None
            v836 = v569;
            break;
        }
        default: { // Some
            std::array<Card,5l> v811 = v810.v.case1.v0;
            switch (v569.tag) {
                case 0: { // None
                    v836 = v810;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v812 = v569.v.case1.v0;
                    US8 v813;
                    v813 = US8_0();
                    int32_t v814; US8 v815;
                    Tuple9 tmp30 = Tuple9(0l, v813);
                    v814 = tmp30.v0; v815 = tmp30.v1;
                    while (while_method_7(v814)){
                        Card v817;
                        v817 = v811[v814];
                        Card v818;
                        v818 = v812[v814];
                        US8 v829;
                        switch (v815.tag) {
                            case 0: { // Eq
                                uint8_t v819;
                                v819 = v817.rank;
                                uint8_t v820;
                                v820 = v818.rank;
                                bool v821;
                                v821 = v819 < v820;
                                if (v821){
                                    v829 = US8_2();
                                } else {
                                    bool v823;
                                    v823 = v819 > v820;
                                    if (v823){
                                        v829 = US8_1();
                                    } else {
                                        v829 = US8_0();
                                    }
                                }
                                break;
                            }
                            default: {
                                break;
                            }
                        }
                        v815 = v829;
                        v814++;
                    }
                    bool v830;
                    switch (v815.tag) {
                        case 1: { // Gt
                            v830 = true;
                            break;
                        }
                        default: {
                            v830 = false;
                        }
                    }
                    std::array<Card,5l> v831;
                    if (v830){
                        v831 = v811;
                    } else {
                        v831 = v812;
                    }
                    v836 = US5_1(v831);
                }
            }
        }
    }
    US11 v841;
    switch (v59.tag) {
        case 0: { // None
            v841 = US11_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v837 = v59.v.case1.v0;
            v841 = US11_1(v837, 1);
        }
    }
    US11 v846;
    switch (v148.tag) {
        case 0: { // None
            v846 = US11_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v842 = v148.v.case1.v0;
            v846 = US11_1(v842, 2);
        }
    }
    US11 v851;
    switch (v198.tag) {
        case 0: { // None
            v851 = US11_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v847 = v198.v.case1.v0;
            v851 = US11_1(v847, 3);
        }
    }
    US11 v856;
    switch (v236.tag) {
        case 0: { // None
            v856 = US11_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v852 = v236.v.case1.v0;
            v856 = US11_1(v852, 4);
        }
    }
    US11 v861;
    switch (v374.tag) {
        case 0: { // None
            v861 = US11_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v857 = v374.v.case1.v0;
            v861 = US11_1(v857, 5);
        }
    }
    US11 v866;
    switch (v456.tag) {
        case 0: { // None
            v866 = US11_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v862 = v456.v.case1.v0;
            v866 = US11_1(v862, 6);
        }
    }
    US11 v871;
    switch (v506.tag) {
        case 0: { // None
            v871 = US11_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v867 = v506.v.case1.v0;
            v871 = US11_1(v867, 7);
        }
    }
    US11 v876;
    switch (v836.tag) {
        case 0: { // None
            v876 = US11_0();
            break;
        }
        default: { // Some
            std::array<Card,5l> v872 = v836.v.case1.v0;
            v876 = US11_1(v872, 8);
        }
    }
    US11 v878;
    switch (v876.tag) {
        case 0: { // None
            v878 = US11_0();
            break;
        }
        default: {
            v878 = v876;
        }
    }
    US11 v888;
    switch (v878.tag) {
        case 1: { // Some
            std::array<Card,5l> v879 = v878.v.case1.v0; int8_t v880 = v878.v.case1.v1;
            switch (v871.tag) {
                case 0: { // None
                    v888 = v878;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v881 = v871.v.case1.v0; int8_t v882 = v871.v.case1.v1;
                    v888 = US11_1(v879, v880);
                }
            }
            break;
        }
        default: {
            switch (v871.tag) {
                case 0: { // None
                    v888 = v878;
                    break;
                }
                default: {
                    switch (v878.tag) {
                        default: { // None
                            v888 = v871;
                        }
                    }
                }
            }
        }
    }
    US11 v898;
    switch (v888.tag) {
        case 1: { // Some
            std::array<Card,5l> v889 = v888.v.case1.v0; int8_t v890 = v888.v.case1.v1;
            switch (v866.tag) {
                case 0: { // None
                    v898 = v888;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v891 = v866.v.case1.v0; int8_t v892 = v866.v.case1.v1;
                    v898 = US11_1(v889, v890);
                }
            }
            break;
        }
        default: {
            switch (v866.tag) {
                case 0: { // None
                    v898 = v888;
                    break;
                }
                default: {
                    switch (v888.tag) {
                        default: { // None
                            v898 = v866;
                        }
                    }
                }
            }
        }
    }
    US11 v908;
    switch (v898.tag) {
        case 1: { // Some
            std::array<Card,5l> v899 = v898.v.case1.v0; int8_t v900 = v898.v.case1.v1;
            switch (v861.tag) {
                case 0: { // None
                    v908 = v898;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v901 = v861.v.case1.v0; int8_t v902 = v861.v.case1.v1;
                    v908 = US11_1(v899, v900);
                }
            }
            break;
        }
        default: {
            switch (v861.tag) {
                case 0: { // None
                    v908 = v898;
                    break;
                }
                default: {
                    switch (v898.tag) {
                        default: { // None
                            v908 = v861;
                        }
                    }
                }
            }
        }
    }
    US11 v918;
    switch (v908.tag) {
        case 1: { // Some
            std::array<Card,5l> v909 = v908.v.case1.v0; int8_t v910 = v908.v.case1.v1;
            switch (v856.tag) {
                case 0: { // None
                    v918 = v908;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v911 = v856.v.case1.v0; int8_t v912 = v856.v.case1.v1;
                    v918 = US11_1(v909, v910);
                }
            }
            break;
        }
        default: {
            switch (v856.tag) {
                case 0: { // None
                    v918 = v908;
                    break;
                }
                default: {
                    switch (v908.tag) {
                        default: { // None
                            v918 = v856;
                        }
                    }
                }
            }
        }
    }
    US11 v928;
    switch (v918.tag) {
        case 1: { // Some
            std::array<Card,5l> v919 = v918.v.case1.v0; int8_t v920 = v918.v.case1.v1;
            switch (v851.tag) {
                case 0: { // None
                    v928 = v918;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v921 = v851.v.case1.v0; int8_t v922 = v851.v.case1.v1;
                    v928 = US11_1(v919, v920);
                }
            }
            break;
        }
        default: {
            switch (v851.tag) {
                case 0: { // None
                    v928 = v918;
                    break;
                }
                default: {
                    switch (v918.tag) {
                        default: { // None
                            v928 = v851;
                        }
                    }
                }
            }
        }
    }
    US11 v938;
    switch (v928.tag) {
        case 1: { // Some
            std::array<Card,5l> v929 = v928.v.case1.v0; int8_t v930 = v928.v.case1.v1;
            switch (v846.tag) {
                case 0: { // None
                    v938 = v928;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v931 = v846.v.case1.v0; int8_t v932 = v846.v.case1.v1;
                    v938 = US11_1(v929, v930);
                }
            }
            break;
        }
        default: {
            switch (v846.tag) {
                case 0: { // None
                    v938 = v928;
                    break;
                }
                default: {
                    switch (v928.tag) {
                        default: { // None
                            v938 = v846;
                        }
                    }
                }
            }
        }
    }
    US11 v948;
    switch (v938.tag) {
        case 1: { // Some
            std::array<Card,5l> v939 = v938.v.case1.v0; int8_t v940 = v938.v.case1.v1;
            switch (v841.tag) {
                case 0: { // None
                    v948 = v938;
                    break;
                }
                default: { // Some
                    std::array<Card,5l> v941 = v841.v.case1.v0; int8_t v942 = v841.v.case1.v1;
                    v948 = US11_1(v939, v940);
                }
            }
            break;
        }
        default: {
            switch (v841.tag) {
                case 0: { // None
                    v948 = v938;
                    break;
                }
                default: {
                    switch (v938.tag) {
                        default: { // None
                            v948 = v841;
                        }
                    }
                }
            }
        }
    }
    std::array<Card,5l> v953; int8_t v954;
    switch (v948.tag) {
        case 0: { // None
            v953 = v6; v954 = 0;
            break;
        }
        default: { // Some
            std::array<Card,5l> v949 = v948.v.case1.v0; int8_t v950 = v948.v.case1.v1;
            v953 = v949; v954 = v950;
        }
    }
    return Tuple5(v953, v954);
}
Tuple5 score_8(Card v0, Card v1, Card v2, Card v3, Card v4, Card v5, Card v6){
    std::array<Card,7l> v7;
    v7[0l] = v5;
    v7[1l] = v6;
    v7[2l] = v0;
    v7[3l] = v1;
    v7[4l] = v2;
    v7[5l] = v3;
    v7[6l] = v4;
    return score_9(v7);
}
inline bool while_method_12(std::array<Card,5l> v0, US8 v1, int32_t v2){
    bool v3;
    v3 = v2 < 5l;
    return v3;
}
US8 method_10(std::array<Card,5l> v0, std::array<Card,5l> v1){
    US8 v2;
    v2 = US8_0();
    US8 v3; int32_t v4;
    Tuple10 tmp33 = Tuple10(v2, 0l);
    v3 = tmp33.v0; v4 = tmp33.v1;
    while (while_method_12(v0, v3, v4)){
        US8 v20; int32_t v21;
        switch (v3.tag) {
            case 0: { // Eq
                Card v6;
                v6 = v0[v4];
                Card v7;
                v7 = v1[v4];
                uint8_t v8;
                v8 = v6.rank;
                uint8_t v9;
                v9 = v7.rank;
                bool v10;
                v10 = v8 < v9;
                US8 v16;
                if (v10){
                    v16 = US8_2();
                } else {
                    bool v12;
                    v12 = v8 > v9;
                    if (v12){
                        v16 = US8_1();
                    } else {
                        v16 = US8_0();
                    }
                }
                int32_t v17;
                v17 = v4 + 1l;
                v20 = v16; v21 = v17;
                break;
            }
            default: {
                // hello;
                return v3;
            }
        }
        v3 = v20;
        v4 = v21;~
    }
    return v3;
}
int16_t game_1(std::bitset<52l> & v0, std::mt19937 & v1){
    Card v2;
    v2 = draw_card_2(v0, v1);
    Card v3;
    v3 = draw_card_2(v0, v1);
    Card v4;
    v4 = draw_card_2(v0, v1);
    Card v5;
    v5 = draw_card_2(v0, v1);
    US0 v18;
    v18 = US0_1(v2, v3);
    US0 v19;
    v19 = US0_1(v4, v5);
    US1 v20;
    v20 = US1_1(2, 0);
    uint8_t v21; US0 v22; int16_t v23; US0 v24; int16_t v25; US1 v26;
    Tuple2 tmp2 = Tuple2(11u, v18, 2, v19, 1, v20);
    v21 = tmp2.v0; v22 = tmp2.v1; v23 = tmp2.v2; v24 = tmp2.v3; v25 = tmp2.v4; v26 = tmp2.v5;
    while (while_method_2(v21, v22, v23, v24, v25, v26)){
        US0 v120; int16_t v121; US0 v122; int16_t v123; US1 v124;
        switch (v26.tag) {
            case 0: { // Done
                US1 v28;
                v28 = US1_0();
                v120 = v22; v121 = v23; v122 = v24; v123 = v25; v124 = v28;
                break;
            }
            default: { // TurnOf
                int8_t v29 = v26.v.case1.v0; int8_t v30 = v26.v.case1.v1;
                bool v31;
                v31 = v30 == 0;
                US0 v32; int16_t v33; US0 v34; int16_t v35; int8_t v36;
                if (v31){
                    v32 = v22; v33 = v23; v34 = v24; v35 = v25; v36 = 1;
                } else {
                    v32 = v24; v33 = v25; v34 = v22; v35 = v23; v36 = 0;
                }
                bool v37;
                v37 = v33 >= v35;
                int16_t v38;
                if (v37){
                    v38 = v33;
                } else {
                    v38 = v35;
                }
                int16_t v39;
                v39 = v38 + v35;
                bool v40;
                v40 = v33 < v35;
                float v41;
                if (v40){
                    v41 = 1.0f;
                } else {
                    v41 = 0.0f;
                }
                int16_t v42;
                v42 = v39 / 4;
                int16_t v43;
                v43 = v39 + v42;
                int16_t v44;
                v44 = v35 + 2;
                bool v45;
                v45 = v44 <= v43;
                bool v47;
                if (v45){
                    bool v46;
                    v46 = v43 <= 100;
                    v47 = v46;
                } else {
                    v47 = false;
                }
                float v48;
                if (v47){
                    v48 = 0.25f;
                } else {
                    v48 = 0.0f;
                }
                int16_t v49;
                v49 = v39 / 3;
                int16_t v50;
                v50 = v39 + v49;
                bool v51;
                v51 = v44 <= v50;
                bool v53;
                if (v51){
                    bool v52;
                    v52 = v50 <= 100;
                    v53 = v52;
                } else {
                    v53 = false;
                }
                float v54;
                if (v53){
                    v54 = 0.25f;
                } else {
                    v54 = 0.0f;
                }
                int16_t v55;
                v55 = v39 / 2;
                int16_t v56;
                v56 = v39 + v55;
                bool v57;
                v57 = v44 <= v56;
                bool v59;
                if (v57){
                    bool v58;
                    v58 = v56 <= 100;
                    v59 = v58;
                } else {
                    v59 = false;
                }
                float v60;
                if (v59){
                    v60 = 0.25f;
                } else {
                    v60 = 0.0f;
                }
                int16_t v61;
                v61 = v39 + v39;
                bool v62;
                v62 = v44 <= v61;
                bool v64;
                if (v62){
                    bool v63;
                    v63 = v61 <= 100;
                    v64 = v63;
                } else {
                    v64 = false;
                }
                float v65;
                if (v64){
                    v65 = 0.25f;
                } else {
                    v65 = 0.0f;
                }
                int16_t v66;
                v66 = v39 * 3;
                int16_t v67;
                v67 = v66 / 2;
                int16_t v68;
                v68 = v39 + v67;
                bool v69;
                v69 = v44 <= v68;
                bool v71;
                if (v69){
                    bool v70;
                    v70 = v68 <= 100;
                    v71 = v70;
                } else {
                    v71 = false;
                }
                float v72;
                if (v71){
                    v72 = 0.25f;
                } else {
                    v72 = 0.0f;
                }
                bool v73;
                v73 = v33 < 100;
                float v74;
                if (v73){
                    v74 = 0.3f;
                } else {
                    v74 = 0.0f;
                }
                std::array<Tuple3,8l> v75;
                US2 v76;
                v76 = US2_1();
                v75[0l] = Tuple3(v76, v41);
                US2 v77;
                v77 = US2_0();
                v75[1l] = Tuple3(v77, 2.0f);
                US2 v78;
                v78 = US2_2(v43);
                v75[2l] = Tuple3(v78, v48);
                US2 v79;
                v79 = US2_2(v50);
                v75[3l] = Tuple3(v79, v54);
                US2 v80;
                v80 = US2_2(v56);
                v75[4l] = Tuple3(v80, v60);
                US2 v81;
                v81 = US2_2(v61);
                v75[5l] = Tuple3(v81, v65);
                US2 v82;
                v82 = US2_2(v68);
                v75[6l] = Tuple3(v82, v72);
                US2 v83;
                v83 = US2_2(100);
                v75[7l] = Tuple3(v83, v74);
                US2 v84;
                v84 = sample_discrete_5(v75, v1);
                US0 v108; int16_t v109; US1 v110;
                switch (v84.tag) {
                    case 0: { // Call
                        bool v87;
                        v87 = v35 >= v33;
                        int16_t v88;
                        if (v87){
                            v88 = v35;
                        } else {
                            v88 = v33;
                        }
                        bool v89;
                        v89 = 0 < v29;
                        US1 v93;
                        if (v89){
                            int8_t v90;
                            v90 = v29 - 1;
                            v93 = US1_1(v90, v36);
                        } else {
                            v93 = US1_0();
                        }
                        v108 = v32; v109 = v88; v110 = v93;
                        break;
                    }
                    case 1: { // Fold
                        US0 v85;
                        v85 = US0_0();
                        US1 v86;
                        v86 = US1_0();
                        v108 = v85; v109 = v33; v110 = v86;
                        break;
                    }
                    default: { // RaiseTo
                        int16_t v94 = v84.v.case2.v0;
                        bool v95;
                        v95 = v94 >= v33;
                        int16_t v96;
                        if (v95){
                            v96 = v94;
                        } else {
                            v96 = v33;
                        }
                        bool v97;
                        v97 = v44 >= v96;
                        int16_t v98;
                        if (v97){
                            v98 = v44;
                        } else {
                            v98 = v96;
                        }
                        bool v99;
                        v99 = 100 < v98;
                        int16_t v100;
                        if (v99){
                            v100 = 100;
                        } else {
                            v100 = v98;
                        }
                        US1 v101;
                        v101 = US1_1(0, v36);
                        v108 = v32; v109 = v100; v110 = v101;
                    }
                }
                US0 v111; int16_t v112; US0 v113; int16_t v114;
                if (v31){
                    v111 = v108; v112 = v109; v113 = v34; v114 = v35;
                } else {
                    v111 = v34; v112 = v35; v113 = v108; v114 = v109;
                }
                v120 = v111; v121 = v112; v122 = v113; v123 = v114; v124 = v110;
            }
        }
        uint8_t v125;
        v125 = v21 - 1u;
        v21 = v125;
        v22 = v120;
        v23 = v121;
        v24 = v122;
        v25 = v123;
        v26 = v124;
    }
    Card v126;
    v126 = draw_card_2(v0, v1);
    Card v127;
    v127 = draw_card_2(v0, v1);
    Card v128;
    v128 = draw_card_2(v0, v1);
    bool v138;
    v138 = v23 == 2;
    bool v143;
    if (v138){
        bool v141;
        v141 = v25 == 1;
        v143 = v141;
    } else {
        v143 = false;
    }
    int8_t v144;
    if (v143){
        v144 = 2;
    } else {
        v144 = 1;
    }
    US1 v145;
    v145 = US1_1(v144, 0);
    uint8_t v146; US0 v147; int16_t v148; US0 v149; int16_t v150; US1 v151;
    Tuple2 tmp6 = Tuple2(11u, v22, v23, v24, v25, v145);
    v146 = tmp6.v0; v147 = tmp6.v1; v148 = tmp6.v2; v149 = tmp6.v3; v150 = tmp6.v4; v151 = tmp6.v5;
    while (while_method_2(v146, v147, v148, v149, v150, v151)){
        US0 v245; int16_t v246; US0 v247; int16_t v248; US1 v249;
        switch (v151.tag) {
            case 0: { // Done
                US1 v153;
                v153 = US1_0();
                v245 = v147; v246 = v148; v247 = v149; v248 = v150; v249 = v153;
                break;
            }
            default: { // TurnOf
                int8_t v154 = v151.v.case1.v0; int8_t v155 = v151.v.case1.v1;
                bool v156;
                v156 = v155 == 0;
                US0 v157; int16_t v158; US0 v159; int16_t v160; int8_t v161;
                if (v156){
                    v157 = v147; v158 = v148; v159 = v149; v160 = v150; v161 = 1;
                } else {
                    v157 = v149; v158 = v150; v159 = v147; v160 = v148; v161 = 0;
                }
                bool v162;
                v162 = v158 >= v160;
                int16_t v163;
                if (v162){
                    v163 = v158;
                } else {
                    v163 = v160;
                }
                int16_t v164;
                v164 = v163 + v160;
                bool v165;
                v165 = v158 < v160;
                float v166;
                if (v165){
                    v166 = 1.0f;
                } else {
                    v166 = 0.0f;
                }
                int16_t v167;
                v167 = v164 / 4;
                int16_t v168;
                v168 = v164 + v167;
                int16_t v169;
                v169 = v160 + 2;
                bool v170;
                v170 = v169 <= v168;
                bool v172;
                if (v170){
                    bool v171;
                    v171 = v168 <= 100;
                    v172 = v171;
                } else {
                    v172 = false;
                }
                float v173;
                if (v172){
                    v173 = 0.25f;
                } else {
                    v173 = 0.0f;
                }
                int16_t v174;
                v174 = v164 / 3;
                int16_t v175;
                v175 = v164 + v174;
                bool v176;
                v176 = v169 <= v175;
                bool v178;
                if (v176){
                    bool v177;
                    v177 = v175 <= 100;
                    v178 = v177;
                } else {
                    v178 = false;
                }
                float v179;
                if (v178){
                    v179 = 0.25f;
                } else {
                    v179 = 0.0f;
                }
                int16_t v180;
                v180 = v164 / 2;
                int16_t v181;
                v181 = v164 + v180;
                bool v182;
                v182 = v169 <= v181;
                bool v184;
                if (v182){
                    bool v183;
                    v183 = v181 <= 100;
                    v184 = v183;
                } else {
                    v184 = false;
                }
                float v185;
                if (v184){
                    v185 = 0.25f;
                } else {
                    v185 = 0.0f;
                }
                int16_t v186;
                v186 = v164 + v164;
                bool v187;
                v187 = v169 <= v186;
                bool v189;
                if (v187){
                    bool v188;
                    v188 = v186 <= 100;
                    v189 = v188;
                } else {
                    v189 = false;
                }
                float v190;
                if (v189){
                    v190 = 0.25f;
                } else {
                    v190 = 0.0f;
                }
                int16_t v191;
                v191 = v164 * 3;
                int16_t v192;
                v192 = v191 / 2;
                int16_t v193;
                v193 = v164 + v192;
                bool v194;
                v194 = v169 <= v193;
                bool v196;
                if (v194){
                    bool v195;
                    v195 = v193 <= 100;
                    v196 = v195;
                } else {
                    v196 = false;
                }
                float v197;
                if (v196){
                    v197 = 0.25f;
                } else {
                    v197 = 0.0f;
                }
                bool v198;
                v198 = v158 < 100;
                float v199;
                if (v198){
                    v199 = 0.3f;
                } else {
                    v199 = 0.0f;
                }
                std::array<Tuple3,8l> v200;
                US2 v201;
                v201 = US2_1();
                v200[0l] = Tuple3(v201, v166);
                US2 v202;
                v202 = US2_0();
                v200[1l] = Tuple3(v202, 2.0f);
                US2 v203;
                v203 = US2_2(v168);
                v200[2l] = Tuple3(v203, v173);
                US2 v204;
                v204 = US2_2(v175);
                v200[3l] = Tuple3(v204, v179);
                US2 v205;
                v205 = US2_2(v181);
                v200[4l] = Tuple3(v205, v185);
                US2 v206;
                v206 = US2_2(v186);
                v200[5l] = Tuple3(v206, v190);
                US2 v207;
                v207 = US2_2(v193);
                v200[6l] = Tuple3(v207, v197);
                US2 v208;
                v208 = US2_2(100);
                v200[7l] = Tuple3(v208, v199);
                US2 v209;
                v209 = sample_discrete_5(v200, v1);
                US0 v233; int16_t v234; US1 v235;
                switch (v209.tag) {
                    case 0: { // Call
                        bool v212;
                        v212 = v160 >= v158;
                        int16_t v213;
                        if (v212){
                            v213 = v160;
                        } else {
                            v213 = v158;
                        }
                        bool v214;
                        v214 = 0 < v154;
                        US1 v218;
                        if (v214){
                            int8_t v215;
                            v215 = v154 - 1;
                            v218 = US1_1(v215, v161);
                        } else {
                            v218 = US1_0();
                        }
                        v233 = v157; v234 = v213; v235 = v218;
                        break;
                    }
                    case 1: { // Fold
                        US0 v210;
                        v210 = US0_0();
                        US1 v211;
                        v211 = US1_0();
                        v233 = v210; v234 = v158; v235 = v211;
                        break;
                    }
                    default: { // RaiseTo
                        int16_t v219 = v209.v.case2.v0;
                        bool v220;
                        v220 = v219 >= v158;
                        int16_t v221;
                        if (v220){
                            v221 = v219;
                        } else {
                            v221 = v158;
                        }
                        bool v222;
                        v222 = v169 >= v221;
                        int16_t v223;
                        if (v222){
                            v223 = v169;
                        } else {
                            v223 = v221;
                        }
                        bool v224;
                        v224 = 100 < v223;
                        int16_t v225;
                        if (v224){
                            v225 = 100;
                        } else {
                            v225 = v223;
                        }
                        US1 v226;
                        v226 = US1_1(0, v161);
                        v233 = v157; v234 = v225; v235 = v226;
                    }
                }
                US0 v236; int16_t v237; US0 v238; int16_t v239;
                if (v156){
                    v236 = v233; v237 = v234; v238 = v159; v239 = v160;
                } else {
                    v236 = v159; v237 = v160; v238 = v233; v239 = v234;
                }
                v245 = v236; v246 = v237; v247 = v238; v248 = v239; v249 = v235;
            }
        }
        uint8_t v250;
        v250 = v146 - 1u;
        v146 = v250;
        v147 = v245;
        v148 = v246;
        v149 = v247;
        v150 = v248;
        v151 = v249;
    }
    Card v251;
    v251 = draw_card_2(v0, v1);
    bool v261;
    v261 = v148 == 2;
    bool v266;
    if (v261){
        bool v264;
        v264 = v150 == 1;
        v266 = v264;
    } else {
        v266 = false;
    }
    int8_t v267;
    if (v266){
        v267 = 2;
    } else {
        v267 = 1;
    }
    US1 v268;
    v268 = US1_1(v267, 0);
    uint8_t v269; US0 v270; int16_t v271; US0 v272; int16_t v273; US1 v274;
    Tuple2 tmp7 = Tuple2(11u, v147, v148, v149, v150, v268);
    v269 = tmp7.v0; v270 = tmp7.v1; v271 = tmp7.v2; v272 = tmp7.v3; v273 = tmp7.v4; v274 = tmp7.v5;
    while (while_method_2(v269, v270, v271, v272, v273, v274)){
        US0 v368; int16_t v369; US0 v370; int16_t v371; US1 v372;
        switch (v274.tag) {
            case 0: { // Done
                US1 v276;
                v276 = US1_0();
                v368 = v270; v369 = v271; v370 = v272; v371 = v273; v372 = v276;
                break;
            }
            default: { // TurnOf
                int8_t v277 = v274.v.case1.v0; int8_t v278 = v274.v.case1.v1;
                bool v279;
                v279 = v278 == 0;
                US0 v280; int16_t v281; US0 v282; int16_t v283; int8_t v284;
                if (v279){
                    v280 = v270; v281 = v271; v282 = v272; v283 = v273; v284 = 1;
                } else {
                    v280 = v272; v281 = v273; v282 = v270; v283 = v271; v284 = 0;
                }
                bool v285;
                v285 = v281 >= v283;
                int16_t v286;
                if (v285){
                    v286 = v281;
                } else {
                    v286 = v283;
                }
                int16_t v287;
                v287 = v286 + v283;
                bool v288;
                v288 = v281 < v283;
                float v289;
                if (v288){
                    v289 = 1.0f;
                } else {
                    v289 = 0.0f;
                }
                int16_t v290;
                v290 = v287 / 4;
                int16_t v291;
                v291 = v287 + v290;
                int16_t v292;
                v292 = v283 + 2;
                bool v293;
                v293 = v292 <= v291;
                bool v295;
                if (v293){
                    bool v294;
                    v294 = v291 <= 100;
                    v295 = v294;
                } else {
                    v295 = false;
                }
                float v296;
                if (v295){
                    v296 = 0.25f;
                } else {
                    v296 = 0.0f;
                }
                int16_t v297;
                v297 = v287 / 3;
                int16_t v298;
                v298 = v287 + v297;
                bool v299;
                v299 = v292 <= v298;
                bool v301;
                if (v299){
                    bool v300;
                    v300 = v298 <= 100;
                    v301 = v300;
                } else {
                    v301 = false;
                }
                float v302;
                if (v301){
                    v302 = 0.25f;
                } else {
                    v302 = 0.0f;
                }
                int16_t v303;
                v303 = v287 / 2;
                int16_t v304;
                v304 = v287 + v303;
                bool v305;
                v305 = v292 <= v304;
                bool v307;
                if (v305){
                    bool v306;
                    v306 = v304 <= 100;
                    v307 = v306;
                } else {
                    v307 = false;
                }
                float v308;
                if (v307){
                    v308 = 0.25f;
                } else {
                    v308 = 0.0f;
                }
                int16_t v309;
                v309 = v287 + v287;
                bool v310;
                v310 = v292 <= v309;
                bool v312;
                if (v310){
                    bool v311;
                    v311 = v309 <= 100;
                    v312 = v311;
                } else {
                    v312 = false;
                }
                float v313;
                if (v312){
                    v313 = 0.25f;
                } else {
                    v313 = 0.0f;
                }
                int16_t v314;
                v314 = v287 * 3;
                int16_t v315;
                v315 = v314 / 2;
                int16_t v316;
                v316 = v287 + v315;
                bool v317;
                v317 = v292 <= v316;
                bool v319;
                if (v317){
                    bool v318;
                    v318 = v316 <= 100;
                    v319 = v318;
                } else {
                    v319 = false;
                }
                float v320;
                if (v319){
                    v320 = 0.25f;
                } else {
                    v320 = 0.0f;
                }
                bool v321;
                v321 = v281 < 100;
                float v322;
                if (v321){
                    v322 = 0.3f;
                } else {
                    v322 = 0.0f;
                }
                std::array<Tuple3,8l> v323;
                US2 v324;
                v324 = US2_1();
                v323[0l] = Tuple3(v324, v289);
                US2 v325;
                v325 = US2_0();
                v323[1l] = Tuple3(v325, 2.0f);
                US2 v326;
                v326 = US2_2(v291);
                v323[2l] = Tuple3(v326, v296);
                US2 v327;
                v327 = US2_2(v298);
                v323[3l] = Tuple3(v327, v302);
                US2 v328;
                v328 = US2_2(v304);
                v323[4l] = Tuple3(v328, v308);
                US2 v329;
                v329 = US2_2(v309);
                v323[5l] = Tuple3(v329, v313);
                US2 v330;
                v330 = US2_2(v316);
                v323[6l] = Tuple3(v330, v320);
                US2 v331;
                v331 = US2_2(100);
                v323[7l] = Tuple3(v331, v322);
                US2 v332;
                v332 = sample_discrete_5(v323, v1);
                US0 v356; int16_t v357; US1 v358;
                switch (v332.tag) {
                    case 0: { // Call
                        bool v335;
                        v335 = v283 >= v281;
                        int16_t v336;
                        if (v335){
                            v336 = v283;
                        } else {
                            v336 = v281;
                        }
                        bool v337;
                        v337 = 0 < v277;
                        US1 v341;
                        if (v337){
                            int8_t v338;
                            v338 = v277 - 1;
                            v341 = US1_1(v338, v284);
                        } else {
                            v341 = US1_0();
                        }
                        v356 = v280; v357 = v336; v358 = v341;
                        break;
                    }
                    case 1: { // Fold
                        US0 v333;
                        v333 = US0_0();
                        US1 v334;
                        v334 = US1_0();
                        v356 = v333; v357 = v281; v358 = v334;
                        break;
                    }
                    default: { // RaiseTo
                        int16_t v342 = v332.v.case2.v0;
                        bool v343;
                        v343 = v342 >= v281;
                        int16_t v344;
                        if (v343){
                            v344 = v342;
                        } else {
                            v344 = v281;
                        }
                        bool v345;
                        v345 = v292 >= v344;
                        int16_t v346;
                        if (v345){
                            v346 = v292;
                        } else {
                            v346 = v344;
                        }
                        bool v347;
                        v347 = 100 < v346;
                        int16_t v348;
                        if (v347){
                            v348 = 100;
                        } else {
                            v348 = v346;
                        }
                        US1 v349;
                        v349 = US1_1(0, v284);
                        v356 = v280; v357 = v348; v358 = v349;
                    }
                }
                US0 v359; int16_t v360; US0 v361; int16_t v362;
                if (v279){
                    v359 = v356; v360 = v357; v361 = v282; v362 = v283;
                } else {
                    v359 = v282; v360 = v283; v361 = v356; v362 = v357;
                }
                v368 = v359; v369 = v360; v370 = v361; v371 = v362; v372 = v358;
            }
        }
        uint8_t v373;
        v373 = v269 - 1u;
        v269 = v373;
        v270 = v368;
        v271 = v369;
        v272 = v370;
        v273 = v371;
        v274 = v372;
    }
    Card v374;
    v374 = draw_card_2(v0, v1);
    bool v384;
    v384 = v271 == 2;
    bool v389;
    if (v384){
        bool v387;
        v387 = v273 == 1;
        v389 = v387;
    } else {
        v389 = false;
    }
    int8_t v390;
    if (v389){
        v390 = 2;
    } else {
        v390 = 1;
    }
    US1 v391;
    v391 = US1_1(v390, 0);
    uint8_t v392; US0 v393; int16_t v394; US0 v395; int16_t v396; US1 v397;
    Tuple2 tmp8 = Tuple2(11u, v270, v271, v272, v273, v391);
    v392 = tmp8.v0; v393 = tmp8.v1; v394 = tmp8.v2; v395 = tmp8.v3; v396 = tmp8.v4; v397 = tmp8.v5;
    while (while_method_2(v392, v393, v394, v395, v396, v397)){
        US0 v491; int16_t v492; US0 v493; int16_t v494; US1 v495;
        switch (v397.tag) {
            case 0: { // Done
                US1 v399;
                v399 = US1_0();
                v491 = v393; v492 = v394; v493 = v395; v494 = v396; v495 = v399;
                break;
            }
            default: { // TurnOf
                int8_t v400 = v397.v.case1.v0; int8_t v401 = v397.v.case1.v1;
                bool v402;
                v402 = v401 == 0;
                US0 v403; int16_t v404; US0 v405; int16_t v406; int8_t v407;
                if (v402){
                    v403 = v393; v404 = v394; v405 = v395; v406 = v396; v407 = 1;
                } else {
                    v403 = v395; v404 = v396; v405 = v393; v406 = v394; v407 = 0;
                }
                bool v408;
                v408 = v404 >= v406;
                int16_t v409;
                if (v408){
                    v409 = v404;
                } else {
                    v409 = v406;
                }
                int16_t v410;
                v410 = v409 + v406;
                bool v411;
                v411 = v404 < v406;
                float v412;
                if (v411){
                    v412 = 1.0f;
                } else {
                    v412 = 0.0f;
                }
                int16_t v413;
                v413 = v410 / 4;
                int16_t v414;
                v414 = v410 + v413;
                int16_t v415;
                v415 = v406 + 2;
                bool v416;
                v416 = v415 <= v414;
                bool v418;
                if (v416){
                    bool v417;
                    v417 = v414 <= 100;
                    v418 = v417;
                } else {
                    v418 = false;
                }
                float v419;
                if (v418){
                    v419 = 0.25f;
                } else {
                    v419 = 0.0f;
                }
                int16_t v420;
                v420 = v410 / 3;
                int16_t v421;
                v421 = v410 + v420;
                bool v422;
                v422 = v415 <= v421;
                bool v424;
                if (v422){
                    bool v423;
                    v423 = v421 <= 100;
                    v424 = v423;
                } else {
                    v424 = false;
                }
                float v425;
                if (v424){
                    v425 = 0.25f;
                } else {
                    v425 = 0.0f;
                }
                int16_t v426;
                v426 = v410 / 2;
                int16_t v427;
                v427 = v410 + v426;
                bool v428;
                v428 = v415 <= v427;
                bool v430;
                if (v428){
                    bool v429;
                    v429 = v427 <= 100;
                    v430 = v429;
                } else {
                    v430 = false;
                }
                float v431;
                if (v430){
                    v431 = 0.25f;
                } else {
                    v431 = 0.0f;
                }
                int16_t v432;
                v432 = v410 + v410;
                bool v433;
                v433 = v415 <= v432;
                bool v435;
                if (v433){
                    bool v434;
                    v434 = v432 <= 100;
                    v435 = v434;
                } else {
                    v435 = false;
                }
                float v436;
                if (v435){
                    v436 = 0.25f;
                } else {
                    v436 = 0.0f;
                }
                int16_t v437;
                v437 = v410 * 3;
                int16_t v438;
                v438 = v437 / 2;
                int16_t v439;
                v439 = v410 + v438;
                bool v440;
                v440 = v415 <= v439;
                bool v442;
                if (v440){
                    bool v441;
                    v441 = v439 <= 100;
                    v442 = v441;
                } else {
                    v442 = false;
                }
                float v443;
                if (v442){
                    v443 = 0.25f;
                } else {
                    v443 = 0.0f;
                }
                bool v444;
                v444 = v404 < 100;
                float v445;
                if (v444){
                    v445 = 0.3f;
                } else {
                    v445 = 0.0f;
                }
                std::array<Tuple3,8l> v446;
                US2 v447;
                v447 = US2_1();
                v446[0l] = Tuple3(v447, v412);
                US2 v448;
                v448 = US2_0();
                v446[1l] = Tuple3(v448, 2.0f);
                US2 v449;
                v449 = US2_2(v414);
                v446[2l] = Tuple3(v449, v419);
                US2 v450;
                v450 = US2_2(v421);
                v446[3l] = Tuple3(v450, v425);
                US2 v451;
                v451 = US2_2(v427);
                v446[4l] = Tuple3(v451, v431);
                US2 v452;
                v452 = US2_2(v432);
                v446[5l] = Tuple3(v452, v436);
                US2 v453;
                v453 = US2_2(v439);
                v446[6l] = Tuple3(v453, v443);
                US2 v454;
                v454 = US2_2(100);
                v446[7l] = Tuple3(v454, v445);
                US2 v455;
                v455 = sample_discrete_5(v446, v1);
                US0 v479; int16_t v480; US1 v481;
                switch (v455.tag) {
                    case 0: { // Call
                        bool v458;
                        v458 = v406 >= v404;
                        int16_t v459;
                        if (v458){
                            v459 = v406;
                        } else {
                            v459 = v404;
                        }
                        bool v460;
                        v460 = 0 < v400;
                        US1 v464;
                        if (v460){
                            int8_t v461;
                            v461 = v400 - 1;
                            v464 = US1_1(v461, v407);
                        } else {
                            v464 = US1_0();
                        }
                        v479 = v403; v480 = v459; v481 = v464;
                        break;
                    }
                    case 1: { // Fold
                        US0 v456;
                        v456 = US0_0();
                        US1 v457;
                        v457 = US1_0();
                        v479 = v456; v480 = v404; v481 = v457;
                        break;
                    }
                    default: { // RaiseTo
                        int16_t v465 = v455.v.case2.v0;
                        bool v466;
                        v466 = v465 >= v404;
                        int16_t v467;
                        if (v466){
                            v467 = v465;
                        } else {
                            v467 = v404;
                        }
                        bool v468;
                        v468 = v415 >= v467;
                        int16_t v469;
                        if (v468){
                            v469 = v415;
                        } else {
                            v469 = v467;
                        }
                        bool v470;
                        v470 = 100 < v469;
                        int16_t v471;
                        if (v470){
                            v471 = 100;
                        } else {
                            v471 = v469;
                        }
                        US1 v472;
                        v472 = US1_1(0, v407);
                        v479 = v403; v480 = v471; v481 = v472;
                    }
                }
                US0 v482; int16_t v483; US0 v484; int16_t v485;
                if (v402){
                    v482 = v479; v483 = v480; v484 = v405; v485 = v406;
                } else {
                    v482 = v405; v483 = v406; v484 = v479; v485 = v480;
                }
                v491 = v482; v492 = v483; v493 = v484; v494 = v485; v495 = v481;
            }
        }
        uint8_t v496;
        v496 = v392 - 1u;
        v392 = v496;
        v393 = v491;
        v394 = v492;
        v395 = v493;
        v396 = v494;
        v397 = v495;
    }
    switch (v393.tag) {
        case 0: { // None
            switch (v395.tag) {
                case 0: { // None
                    return 0;
                    break;
                }
                default: { // Some
                    Card v520 = v395.v.case1.v0; Card v521 = v395.v.case1.v1;
                    int16_t v522;
                    v522 = -v394;
                    return v522;
                }
            }
            break;
        }
        default: { // Some
            Card v497 = v393.v.case1.v0; Card v498 = v393.v.case1.v1;
            switch (v395.tag) {
                case 0: { // None
                    return v396;
                    break;
                }
                default: { // Some
                    Card v499 = v395.v.case1.v0; Card v500 = v395.v.case1.v1;
                    std::array<Card,5l> v501; int8_t v502;
                    Tuple5 tmp31 = score_8(v126, v127, v128, v251, v374, v497, v498);
                    v501 = tmp31.v0; v502 = tmp31.v1;
                    std::array<Card,5l> v503; int8_t v504;
                    Tuple5 tmp32 = score_8(v126, v127, v128, v251, v374, v499, v500);
                    v503 = tmp32.v0; v504 = tmp32.v1;
                    bool v505;
                    v505 = v502 < v504;
                    US8 v511;
                    if (v505){
                        v511 = US8_2();
                    } else {
                        bool v507;
                        v507 = v502 > v504;
                        if (v507){
                            v511 = US8_1();
                        } else {
                            v511 = US8_0();
                        }
                    }
                    US8 v513;
                    switch (v511.tag) {
                        case 0: { // Eq
                            v513 = method_10(v501, v503);
                            break;
                        }
                        default: {
                            v513 = v511;
                        }
                    }
                    switch (v513.tag) {
                        case 0: { // Eq
                            return 0;
                            break;
                        }
                        case 1: { // Gt
                            return v396;
                            break;
                        }
                        default: { // Lt
                            int16_t v514;
                            v514 = -v394;
                            return v514;
                        }
                    }
                }
            }
        }
    }
}
int16_t game_loop_0(){
    std::random_device v0;
    std::mt19937 v1(v0());
    std::mt19937 & v2 = v1;
    int32_t v3; int16_t v4;
    Tuple0 tmp0 = Tuple0(0l, 0);
    v3 = tmp0.v0; v4 = tmp0.v1;
    while (while_method_0(v3)){
        std::bitset<52l> v6;
        std::bitset<52l> & v7 = v6;
        int16_t v8;
        v8 = game_1(v7, v2);
        int32_t v9;
        v9 = v3 % 10000l;
        bool v10;
        v10 = v9 == 0l;
        if (v10){
            std::cout << v3 << std::endl;
        } else {
        }
        int16_t v11;
        v11 = v4 + v8;
        v4 = v11;
        v3++;
    }
    return v4;
}
int32_t main() {
    int16_t v0;
    v0 = game_loop_0();
    std::cout << v0 << std::endl;
    return 0l;
}
